#include "hip/hip_runtime.h"
#include "core/detail/multi_array_utils.hpp"
#include "cuda/constant_mem.h"
#include "cuda/core/cu_sim_data.h"
#include "cuda/core/ptc_updater_helper.cuh"
#include "cuda/core/sim_environment_dev.h"
#include "cuda/cudaUtility.h"
#include "cuda/kernels.h"
#include "cuda/ptr_util.h"
#include "cuda/utils/interpolation.cuh"
#include "ptc_updater_logsph.h"
#include "utils/logger.h"
#include "utils/util_functions.h"

#define DEPOSIT_EPS 1.0e-10f

namespace Aperture {

__constant__ PtcUpdaterDev::fields_data dev_fields;

namespace Kernels {

HD_INLINE void
cart2logsph(Scalar &v1, Scalar &v2, Scalar &v3, Scalar x1, Scalar x2,
            Scalar x3) {
  Scalar v1n = v1, v2n = v2, v3n = v3;
  v1 =
      v1n * sin(x2) * cos(x3) + v2n * sin(x2) * sin(x3) + v3n * cos(x2);
  v2 =
      v1n * cos(x2) * cos(x3) + v2n * cos(x2) * sin(x3) - v3n * sin(x2);
  v3 = -v1n * sin(x3) + v2n * cos(x3);
}

HD_INLINE void
logsph2cart(Scalar &v1, Scalar &v2, Scalar &v3, Scalar x1, Scalar x2,
            Scalar x3) {
  Scalar v1n = v1, v2n = v2, v3n = v3;
  v1 =
      v1n * sin(x2) * cos(x3) + v2n * cos(x2) * cos(x3) - v3n * sin(x3);
  v2 =
      v1n * sin(x2) * sin(x3) + v2n * cos(x2) * sin(x3) + v3n * cos(x3);
  v3 = v1n * cos(x2) - v2n * sin(x2);
}

__global__ void
vay_push_2d(particle_data ptc, size_t num,
            PtcUpdaterDev::fields_data fields, Scalar dt) {
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = ptc.cell[idx];
    // Skip empty particles
    if (c == MAX_CELL || idx >= num) continue;

    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    if (!dev_mesh.is_in_bulk(c1, c2)) {
      ptc.cell[idx] = MAX_CELL;
      continue;
    }
    // Load particle quantities
    Interpolator2D<spline_t> interp;
    auto flag = ptc.flag[idx];
    int sp = get_ptc_type(flag);
    auto old_x1 = ptc.x1[idx], old_x2 = ptc.x2[idx];
    auto p1 = ptc.p1[idx], p2 = ptc.p2[idx], p3 = ptc.p3[idx],
         gamma = ptc.E[idx];
    Scalar q_over_m = dt * 0.5f * dev_charges[sp] / dev_masses[sp];
    if (p1 != p1 || p2 != p2 || p3 != p3) {
      // printf("NaN detected! p is %f, E1 is %f, E2 is %f, E3 is %f,
      // B1 is %f, B2 is %f, B3 is %f\n", p,
      //        E1, E2, E3, B1, B2, B3);
      printf(
          "NaN detected in push! p1 is %f, p2 is %f, p3 is %f, gamma "
          "is %f\n",
          p1, p2, p3, gamma);
      asm("trap;");
    }
    // step 0: Grab E & M fields at the particle position
    gamma = std::sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);
    if (!check_bit(flag, ParticleFlag::ignore_EM)) {
      Scalar E1 =
          (interp(fields.E1, old_x1, old_x2, c1, c2, Stagger(0b110))) *
          // interp(dev_bg_fields.E1, old_x1, old_x2, c1, c2,
          //        Stagger(0b110))) *
          q_over_m;
      Scalar E2 =
          (interp(fields.E2, old_x1, old_x2, c1, c2, Stagger(0b101))) *
          // interp(dev_bg_fields.E2, old_x1, old_x2, c1, c2,
          //        Stagger(0b101))) *
          q_over_m;
      Scalar E3 =
          (interp(fields.E3, old_x1, old_x2, c1, c2, Stagger(0b011))) *
          // interp(dev_bg_fields.E3, old_x1, old_x2, c1, c2,
          //        Stagger(0b011))) *
          q_over_m;
      Scalar B1 =
          (interp(fields.B1, old_x1, old_x2, c1, c2, Stagger(0b001)) +
           interp(dev_bg_fields.B1, old_x1, old_x2, c1, c2,
                  Stagger(0b001))) *
          q_over_m;
      Scalar B2 =
          (interp(fields.B2, old_x1, old_x2, c1, c2, Stagger(0b010)) +
           interp(dev_bg_fields.B2, old_x1, old_x2, c1, c2,
                  Stagger(0b010))) *
          q_over_m;
      Scalar B3 =
          (interp(fields.B3, old_x1, old_x2, c1, c2, Stagger(0b100))) *
          // interp(dev_bg_fields.B3, old_x1, old_x2, c1, c2,
          //        Stagger(0b100))) *
          q_over_m;

      // printf("B1 = %f, B2 = %f, B3 = %f\n", B1, B2, B3);
      // printf("E1 = %f, E2 = %f, E3 = %f\n", E1, E2, E3);
      // printf("B cell is %f\n", *ptrAddr(fields.B1, c1*sizeof(Scalar)
      // + c2*fields.B1.pitch)); printf("q over m is %f\n", q_over_m);
      // printf("gamma before is %f\n", gamma);
      // printf("p is (%f, %f, %f), gamma is %f\n", p1, p2, p3, gamma);

      // step 1: Update particle momentum using vay pusher
      Scalar up1 = p1 + 2.0f * E1 + (p2 * B3 - p3 * B2) / gamma;
      Scalar up2 = p2 + 2.0f * E2 + (p3 * B1 - p1 * B3) / gamma;
      Scalar up3 = p3 + 2.0f * E3 + (p1 * B2 - p2 * B1) / gamma;
      // printf("p prime is (%f, %f, %f), gamma is %f\n", up1, up2, up3,
      // gamma);
      Scalar tt = B1 * B1 + B2 * B2 + B3 * B3;
      Scalar ut = up1 * B1 + up2 * B2 + up3 * B3;

      Scalar sigma = 1.0f + up1 * up1 + up2 * up2 + up3 * up3 - tt;
      Scalar inv_gamma2 =
          2.0f /
          (sigma + std::sqrt(sigma * sigma + 4.0f * (tt + ut * ut)));
      Scalar s = 1.0f / (1.0f + inv_gamma2 * tt);
      gamma = 1.0f / std::sqrt(inv_gamma2);

      p1 =
          (up1 + B1 * ut * inv_gamma2 + (up2 * B3 - up3 * B2) / gamma) *
          s;
      p2 =
          (up2 + B2 * ut * inv_gamma2 + (up3 * B1 - up1 * B3) / gamma) *
          s;
      p3 =
          (up3 + B3 * ut * inv_gamma2 + (up1 * B2 - up2 * B1) / gamma) *
          s;

      // printf("p after is (%f, %f, %f), gamma is %f, inv_gamma2 is %f,
      // %d\n", p1, p2, p3,
      //        gamma, inv_gamma2, dev_params.gravity_on);
      // Add an artificial gravity
      if (dev_params.gravity_on) {
        Scalar r = exp(dev_mesh.pos(0, c1, old_x1));
        p1 -= dt * dev_params.gravity / (r * r);
        gamma = sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);
        if (gamma != gamma) {
          // printf("NaN detected! p is %f, E1 is %f, E2 is %f, E3 is
          // %f, B1 is %f, B2 is %f, B3 is %f\n", p,
          //        E1, E2, E3, B1, B2, B3);
          printf(
              "NaN detected after gravity! p1 is %f, p2 is %f, p3 is "
              "%f, gamma is "
              "%f\n",
              p1, p2, p3, gamma);
          asm("trap;");
        }
      }

      Scalar p = sqrt(p1 * p1 + p2 * p2 + p3 * p3);
      // if (dev_params.rad_cooling_on && sp != (int)ParticleType::ion)
      // {
      if (dev_params.rad_cooling_on) {
        Scalar res = dt * sqrt(tt / q_over_m / q_over_m) / gamma;
        // if ()
        // int substeps = ceil(res);
        // Scalar ds = 1.0f / substeps;
        // for (int step = 0; step < substeps; step++) {
        // Scalar pdotB = p1 * B1 + p2 * B2 + p3 * B3;
        // Scalar pp1 = p1 - B1 * pdotB / tt - gamma * (E2 * B3 - E3 *
        // B2) / tt; Scalar pp2 = p2 - B2 * pdotB / tt - gamma * (E3 *
        // B1 - E1 * B3) / tt; Scalar pp3 = p3 - B3 * pdotB / tt - gamma
        // * (E1 * B2 - E2 * B1) / tt; Scalar pp = sqrt(pp1 * pp1 + pp2
        // * pp2 + pp3 * pp3);
        // // Scalar p = sqrt(p1 * p1 + p2 * p2 + p3 * p3);

        // p1 -= pp1 * dev_params.rad_cooling_coef * tt /
        // square(dev_params.B0 * q_over_m); p2 -= pp2 *
        // dev_params.rad_cooling_coef * tt / square(dev_params.B0 *
        // q_over_m); p3 -= pp3 * dev_params.rad_cooling_coef * tt /
        // square(dev_params.B0 * q_over_m);
        Scalar tmp1 = (E1 + (p2 * B3 - p3 * B2) / gamma) / q_over_m;
        Scalar tmp2 = (E2 + (p3 * B1 - p1 * B3) / gamma) / q_over_m;
        Scalar tmp3 = (E3 + (p1 * B2 - p2 * B1) / gamma) / q_over_m;
        Scalar tmp_sq = tmp1 * tmp1 + tmp2 * tmp2 + tmp3 * tmp3;
        Scalar bE = (p1 * E1 + p2 * E2 + p3 * E3) / (gamma * q_over_m);

        Scalar delta_p1 =
            dev_params.rad_cooling_coef *
            (((tmp2 * B3 - tmp3 * B2) + bE * E1) / q_over_m -
             gamma * p1 * (tmp_sq - bE * bE)) /
            square(dev_params.B0);
        Scalar delta_p2 =
            dev_params.rad_cooling_coef *
            (((tmp3 * B1 - tmp1 * B3) + bE * E2) / q_over_m -
             gamma * p2 * (tmp_sq - bE * bE)) /
            square(dev_params.B0);
        Scalar delta_p3 =
            dev_params.rad_cooling_coef *
            (((tmp1 * B2 - tmp2 * B1) + bE * E3) / q_over_m -
             gamma * p3 * (tmp_sq - bE * bE)) /
            square(dev_params.B0);
        Scalar dp = sqrt(delta_p1 * delta_p1 + delta_p2 * delta_p2 +
                         delta_p3 * delta_p3);
        // if (dp < p) {
        p1 +=
            (dp < p || dp < 1e-5 ? delta_p1 : 0.5 * p * delta_p1 / dp);
        p2 +=
            (dp < p || dp < 1e-5 ? delta_p2 : 0.5 * p * delta_p2 / dp);
        p3 +=
            (dp < p || dp < 1e-5 ? delta_p3 : 0.5 * p * delta_p3 / dp);
        gamma = sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);
        // }
        // }
      }
    }

    // printf("gamma after is %f\n", gamma);
    // printf("p before is (%f, %f, %f)\n", ptc.p1[idx], ptc.p2[idx],
    // ptc.p3[idx]);
    ptc.p1[idx] = p1;
    ptc.p2[idx] = p2;
    ptc.p3[idx] = p3;
    ptc.E[idx] = gamma;
  }
}

__global__ void
move_photons(photon_data photons, size_t num, Scalar dt) {
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = photons.cell[idx];
    // Skip empty particles
    if (c == MAX_CELL) continue;
    // Load particle quantities
    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    auto v1 = photons.p1[idx], v2 = photons.p2[idx],
         v3 = photons.p3[idx];
    Scalar E = std::sqrt(v1 * v1 + v2 * v2 + v3 * v3);
    v1 = v1 / E;
    v2 = v2 / E;
    v3 = v3 / E;

    auto old_x1 = photons.x1[idx], old_x2 = photons.x2[idx],
         old_x3 = photons.x3[idx];

    // Compute the actual movement
    Scalar r1 = dev_mesh.pos(0, c1, old_x1);
    Scalar exp_r1 = std::exp(r1);

    // Censor photons already outside the conversion radius
    if (exp_r1 > dev_params.r_cutoff || exp_r1 < 1.02) {
      photons.cell[idx] = MAX_CELL;
      continue;
    }

    Scalar r2 = dev_mesh.pos(1, c2, old_x2);
    Scalar x = exp_r1 * std::sin(r2) * std::cos(old_x3);
    Scalar y = exp_r1 * std::sin(r2) * std::sin(old_x3);
    Scalar z = exp_r1 * std::cos(r2);

    logsph2cart(v1, v2, v3, r1, r2, old_x3);
    x += v1 * dt;
    y += v2 * dt;
    z += v3 * dt;
    Scalar r1p = sqrt(x * x + y * y + z * z);
    Scalar r2p = acos(z / r1p);
    r1p = log(r1p);
    Scalar r3p = atan(y / x);
    if (x < 0.0f) v1 *= -1.0f;

    cart2logsph(v1, v2, v3, r1p, r2p, r3p);
    photons.p1[idx] = v1 * E;
    photons.p2[idx] = v2 * E;
    photons.p3[idx] = v3 * E;

    Pos_t new_x1 = old_x1 + (r1p - r1) / dev_mesh.delta[0];
    Pos_t new_x2 = old_x2 + (r2p - r2) / dev_mesh.delta[1];
    // printf("new_x1 is %f, new_x2 is %f, old_x1 is %f, old_x2 is
    // %f\n", new_x1, new_x2, old_x1, old_x2);
    int dc1 = floor(new_x1);
    int dc2 = floor(new_x2);
    photons.cell[idx] = dev_mesh.get_idx(c1 + dc1, c2 + dc2);
    new_x1 -= (Pos_t)dc1;
    new_x2 -= (Pos_t)dc2;
    // printf("new_x1 is %f, new_x2 is %f, dc2 = %d\n", new_x1, new_x2,
    // dc2);
    photons.x1[idx] = new_x1;
    photons.x2[idx] = new_x2;
    photons.x3[idx] = r3p;
    photons.path_left[idx] -= dt;
  }
}

__global__ void
__launch_bounds__(512, 4)
    deposit_current_2d_log_sph(particle_data ptc, size_t num,
                               // PtcUpdaterDev::fields_data fields_a,
                               hipPitchedPtr J1, hipPitchedPtr J2,
                               hipPitchedPtr J3, hipPitchedPtr *Rho,
                               Grid_LogSph_dev::mesh_ptrs mesh_ptrs,
                               Scalar dt, uint32_t step) {
  // if (threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 &&
  // blockIdx.y == 0)
  //   printf("J3 pitch: %lu, sizeof: %d\n", J3.pitch,
  //   sizeof(hipPitchedPtr));
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = ptc.cell[idx];
    // Skip empty particles
    if (c == MAX_CELL || idx >= num) continue;

    // Load particle quantities
    Interpolator2D<spline_t> interp;
    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    if (!dev_mesh.is_in_bulk(c1, c2)) {
      ptc.cell[idx] = MAX_CELL;
      continue;
    }
    auto v1 = ptc.p1[idx], v2 = ptc.p2[idx], v3 = ptc.p3[idx];
    Scalar gamma = ptc.E[idx];
    // printf("gamma is %f\n", gamma);
    // printf("velocity before is (%f, %f, %f)\n", v1, v2, v3);

    auto flag = ptc.flag[idx];
    int sp = get_ptc_type(flag);
    auto w = ptc.weight[idx];
    auto old_x1 = ptc.x1[idx], old_x2 = ptc.x2[idx],
         old_x3 = ptc.x3[idx];

    v1 = v1 / gamma;
    v2 = v2 / gamma;
    v3 = v3 / gamma;

    // step 1: Compute particle movement and update position
    Scalar r1 = dev_mesh.pos(0, c1, old_x1);
    Scalar exp_r1 = std::exp(r1);
    Scalar r2 = dev_mesh.pos(1, c2, old_x2);
    Scalar x = exp_r1 * std::sin(r2) * std::cos(old_x3);
    Scalar y = exp_r1 * std::sin(r2) * std::sin(old_x3);
    Scalar z = exp_r1 * std::cos(r2);
    // printf("cart position is (%f, %f, %f)\n", x, y, z);

    logsph2cart(v1, v2, v3, r1, r2, old_x3);
    // printf("cart velocity is (%f, %f, %f)\n", v1, v2, v3);
    x += v1 * dt;
    y += v2 * dt;
    z += v3 * dt;
    // printf("new cart position is (%f, %f, %f)\n", x, y, z);
    Scalar r1p = sqrt(x * x + y * y + z * z);
    Scalar r2p = acos(z / r1p);
    r1p = log(r1p);
    Scalar r3p = atan(y / x);
    if (x < 0.0f) v1 *= -1.0f;

    // printf("new position is (%f, %f, %f)\n", exp(r1p), r2p, r3p);

    cart2logsph(v1, v2, v3, r1p, r2p, r3p);
    ptc.p1[idx] = v1 * gamma;
    ptc.p2[idx] = v2 * gamma;
    ptc.p3[idx] = v3 * gamma;
    if (v1 != v1) {
      // printf("NaN detected! p is %f, E1 is %f, E2 is %f, E3 is %f, B1
      // is %f, B2 is %f, B3 is %f\n", p,
      //        E1, E2, E3, B1, B2, B3);
      printf(
          "NaN detected in deposit! p1 is %f, p2 is %f, p3 is %f, "
          "gamma is %f\n",
          ptc.p1[idx], ptc.p2[idx], ptc.p3[idx], gamma);
      asm("trap;");
    }

    // Scalar old_pos3 =
    Pos_t new_x1 = old_x1 + (r1p - r1) / dev_mesh.delta[0];
    Pos_t new_x2 = old_x2 + (r2p - r2) / dev_mesh.delta[1];
    // printf("new_x1 is %f, new_x2 is %f, old_x1 is %f, old_x2 is
    // %f\n", new_x1, new_x2, old_x1, old_x2);
    int dc1 = floor(new_x1);
    int dc2 = floor(new_x2);
#ifndef NDEBUG
    if (dc1 > 1 || dc1 < -1 || dc2 > 1 || dc2 < -1)
      printf("----------------- Error: moved more than 1 cell!");
#endif
    new_x1 -= (Pos_t)dc1;
    new_x2 -= (Pos_t)dc2;
    // reflect around the axis
    if (c2 + dc2 < dev_mesh.guard[1]) {
      dc2 += 1;
      new_x2 = 1.0f - new_x2;
    } else if (c2 + dc2 >= dev_mesh.dims[1] - dev_mesh.guard[1]) {
      dc2 -= 1;
      new_x2 = 1.0f - new_x2;
    }
    ptc.cell[idx] = dev_mesh.get_idx(c1 + dc1, c2 + dc2);
    // printf("new_x1 is %f, new_x2 is %f, dc2 = %d\n", new_x1, new_x2,
    // dc2);
    ptc.x1[idx] = new_x1;
    ptc.x2[idx] = new_x2;
    ptc.x3[idx] = r3p;

    // step 2: Deposit current
    if (check_bit(flag, ParticleFlag::ignore_current)) continue;
    // Scalar djz[spline_t::support + 1][spline_t::support + 1] =
    // {0.0f};
    Scalar weight = -dev_charges[sp] * w;

    int j_0 = (dc2 == -1 ? -2 : -1);
    int j_1 = (dc2 == 1 ? 1 : 0);
    int i_0 = (dc1 == -1 ? -2 : -1);
    int i_1 = (dc1 == 1 ? 1 : 0);
    Scalar djy[3] = {0.0f};
    for (int j = j_0; j <= j_1; j++) {
      Scalar sy0 = interp.interpolate(-old_x2 + j + 1);
      Scalar sy1 = interp.interpolate(-new_x2 + (j + 1 - dc2));

      size_t j_offset = (j + c2) * J1.pitch;
      Scalar djx = 0.0f;
      for (int i = i_0; i <= i_1; i++) {
        Scalar sx0 = interp.interpolate(-old_x1 + i + 1);
        Scalar sx1 = interp.interpolate(-new_x1 + (i + 1 - dc1));

        // j1 is movement in r
        int offset = j_offset + (i + c1) * sizeof(Scalar);
        Scalar val0 = movement2d(sy0, sy1, sx0, sx1);
        djx += val0;
        atomicAdd(ptrAddr(J1, offset + sizeof(Scalar)), weight * djx);

        // j2 is movement in theta
        Scalar val1 = movement2d(sx0, sx1, sy0, sy1);
        djy[i - i_0] += val1;
        atomicAdd(ptrAddr(J2, offset + J2.pitch),
                  weight * djy[i - i_0]);

        // j3 is simply v3 times rho at volume average
        // printf("J1 pitch: %d, xsize: %d, ysize: %d\n",
        // J1.pitch, J1.xsize,
        // J1.ysize);
        Scalar val2 = center2d(sx0, sx1, sy0, sy1);
        atomicAdd(ptrAddr(J3, offset),
                  -weight * v3 * val2 / *ptrAddr(mesh_ptrs.dV, offset));

        // rho is deposited at the final position, only do this if we
        // are going to output data next step
        // if ((step + 1) % dev_params.data_interval == 0) {
        Scalar s1 = sx1 * sy1;
        atomicAdd(ptrAddr(Rho[sp], offset), -weight * s1);
        // }
      }
    }
  }
}

__global__ void
convert_j(hipPitchedPtr j1, hipPitchedPtr j2,
          PtcUpdaterDev::fields_data fields) {
  for (int j = blockIdx.y * blockDim.y + threadIdx.y;
       j < dev_mesh.dims[1]; j += blockDim.y * gridDim.y) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
      size_t offset_f = j * dev_fields.J1.pitch + i * sizeof(Scalar);
      size_t offset_d = j * j1.pitch + i * sizeof(double);
      (*ptrAddr(dev_fields.J1, offset_f)) =
          (*(float2 *)((char *)j1.ptr + offset_d)).x;
      (*ptrAddr(dev_fields.J2, offset_f)) =
          (*(float2 *)((char *)j2.ptr + offset_d)).x;
    }
  }
}

__global__ void
process_j(PtcUpdaterDev::fields_data fields,
          Grid_LogSph_dev::mesh_ptrs mesh_ptrs, Scalar dt) {
  for (int j = blockIdx.y * blockDim.y + threadIdx.y;
       j < dev_mesh.dims[1]; j += blockDim.y * gridDim.y) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
      size_t offset = j * dev_fields.J1.pitch + i * sizeof(Scalar);
      Scalar w = dev_mesh.delta[0] * dev_mesh.delta[1] / dt;
      (*ptrAddr(dev_fields.J1, offset)) *=
          w / *ptrAddr(mesh_ptrs.A1_e, offset);
      (*ptrAddr(dev_fields.J2, offset)) *=
          w / *ptrAddr(mesh_ptrs.A2_e, offset);
      for (int n = 0; n < dev_params.num_species; n++) {
        (*ptrAddr(dev_fields.Rho[n], offset)) /=
            *ptrAddr(mesh_ptrs.dV, offset);
      }
    }
  }
}

__global__ void
inject_ptc(particle_data ptc, size_t num, int inj_per_cell, Scalar p1,
           Scalar p2, Scalar p3, Scalar w, hipPitchedPtr rho0,
           hipPitchedPtr rho1, hiprandState *states, Scalar omega) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState localState = states[id];
  for (int i = dev_mesh.guard[1] + 1 + id;
       // i = dev_mesh.dims[1] - dev_mesh.guard[1] - 3 + id;
       i < dev_mesh.dims[1] - dev_mesh.guard[1] - 1;
       i += blockDim.x * gridDim.x) {
    size_t offset = num + i * inj_per_cell * 2;
    Scalar r = exp(dev_mesh.pos(0, dev_mesh.guard[0] + 2, 0.5f));
    Scalar dens = max(-*ptrAddr(rho0, dev_mesh.guard[0] + 2, i),
                      *ptrAddr(rho1, dev_mesh.guard[0] + 2, i));
    if (dens > 0.2 * square(dev_mesh.dims[1] / 3.14f)) continue;
    for (int n = 0; n < inj_per_cell; n++) {
      Pos_t x2 = hiprand_uniform(&localState);
      Scalar theta = dev_mesh.pos(1, i, x2);
      Scalar vphi = omega * r * sin(theta);
      // Scalar vphi = 0.0f;
      ptc.x1[offset + n * 2] = 0.5f;
      ptc.x2[offset + n * 2] = x2;
      ptc.x3[offset + n * 2] = 0.0f;
      ptc.p1[offset + n * 2] = p1;
      ptc.p2[offset + n * 2] = p2;
      ptc.p3[offset + n * 2] = vphi;
      ptc.E[offset + n * 2] =
          sqrt(1.0f + p1 * p1 + p2 * p2 + vphi * vphi);
      // printf("inject E is %f\n", ptc.E[offset + n * 2]);
      // ptc.p3[offset + n * 2] = p3;
      ptc.cell[offset + n * 2] =
          dev_mesh.get_idx(dev_mesh.guard[0] + 2, i);
      ptc.weight[offset + n * 2] = w * sin(theta);
      ptc.flag[offset + n * 2] = set_ptc_type_flag(
          bit_or(ParticleFlag::primary), ParticleType::electron);

      ptc.x1[offset + n * 2 + 1] = 0.5f;
      ptc.x2[offset + n * 2 + 1] = x2;
      ptc.x3[offset + n * 2 + 1] = 0.0f;
      ptc.p1[offset + n * 2 + 1] = p1;
      ptc.p2[offset + n * 2 + 1] = p2;
      ptc.p3[offset + n * 2 + 1] = vphi;
      ptc.E[offset + n * 2 + 1] =
          sqrt(1.0f + p1 * p1 + p2 * p2 + vphi * vphi);
      // printf("inject E is %f\n", ptc.E[offset + n * 2 + 1]);
      // ptc.p3[offset + n * 2 + 1] = p3;
      ptc.cell[offset + n * 2 + 1] =
          dev_mesh.get_idx(dev_mesh.guard[0] + 2, i);
      ptc.weight[offset + n * 2 + 1] = w * sin(theta);
      ptc.flag[offset + n * 2 + 1] = set_ptc_type_flag(
          bit_or(ParticleFlag::primary), ParticleType::ion);
    }
  }
  states[id] = localState;
}

__global__ void
boundary_rho(PtcUpdaterDev::fields_data fields,
             Grid_LogSph_dev::mesh_ptrs mesh_ptrs) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
    size_t offset_0 = i * sizeof(Scalar) +
                      dev_mesh.guard[1] * dev_fields.Rho[0].pitch;
    size_t offset_pi = i * sizeof(Scalar) +
                       (dev_mesh.dims[1] - dev_mesh.guard[1] - 2) *
                           dev_fields.Rho[0].pitch;
    for (int n = 0; n < dev_params.num_species; n++) {
      // (*ptrAddr(dev_fields.Rho[n], offset_0)) +=
      //     *ptrAddr(dev_fields.Rho[n], offset_0 - 2 *
      //     dev_fields.Rho[n].pitch)
      //     * *ptrAddr(mesh_ptrs.dV, offset_0 - 2 *
      //     dev_fields.Rho[n].pitch) / *ptrAddr(mesh_ptrs.dV,
      //     offset_0);
      // (*ptrAddr(dev_fields.Rho[n], offset_pi)) +=
      //     *ptrAddr(dev_fields.Rho[n], offset_pi + 2 *
      //     dev_fields.Rho[n].pitch) * *ptrAddr(mesh_ptrs.dV, offset_pi
      //     + 2
      //     * dev_fields.Rho[n].pitch) / *ptrAddr(mesh_ptrs.dV,
      //     offset_pi);

      // (*ptrAddr(dev_fields.Rho[n], offset_0 - 2 *
      // dev_fields.Rho[0].pitch)) =
      //     0.0f;
      // (*ptrAddr(dev_fields.Rho[n], offset_pi + 2 *
      // dev_fields.Rho[0].pitch))
      // =
      //     0.0f;
    }
    // (*ptrAddr(dev_fields.J1, offset_0)) -=
    //     *ptrAddr(dev_fields.J1, offset_0 - 2 * dev_fields.J1.pitch);
    // (*ptrAddr(dev_fields.J1, offset_pi)) +=
    //     *ptrAddr(dev_fields.J1, offset_pi + 2 * dev_fields.J1.pitch)
    //     * *ptrAddr(mesh_ptrs.A1_e, offset_pi + 2 *
    //     dev_fields.J1.pitch) / *ptrAddr(mesh_ptrs.A1_e, offset_pi);

    // *ptrAddr(dev_fields.J1, offset_0 - 2 * dev_fields.J1.pitch) =
    // 0.0f; *ptrAddr(dev_fields.J1, offset_pi + 2 *
    // dev_fields.J1.pitch) = 0.0f;

    // (*ptrAddr(dev_fields.J2, offset_0)) -=
    //     *ptrAddr(dev_fields.J2, offset_0 - dev_fields.J2.pitch);
    (*ptrAddr(dev_fields.J2, offset_pi + dev_fields.J2.pitch)) -=
        *ptrAddr(dev_fields.J2, offset_pi + 2 * dev_fields.J2.pitch);

    (*ptrAddr(dev_fields.J3, offset_0 - dev_fields.J3.pitch)) = 0.0f;
    (*ptrAddr(dev_fields.J3, offset_pi + dev_fields.J3.pitch)) = 0.0f;
    // (*ptrAddr(dev_fields.J2, offset_0 - dev_fields.J2.pitch)) = 0.0f;
    // (*ptrAddr(dev_fields.J2, offset_pi)) = 0.0f;
    // (*ptrAddr(dev_fields.J2, offset_pi - dev_fields.J2.pitch)) -=
    //     *ptrAddr(dev_fields.J2, offset_pi + dev_fields.J2.pitch);
  }
}

__global__ void
annihilate_pairs(particle_data ptc, size_t num, hipPitchedPtr j1,
                 hipPitchedPtr j2, hipPitchedPtr j3,
                 Grid_LogSph_dev::mesh_ptrs mesh_ptrs, Scalar dt) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    // First do a deposit before annihilation
    auto c = ptc.cell[idx];
    auto flag = ptc.flag[idx];
    // Skip empty particles
    if (c == MAX_CELL) continue;

    if (check_bit(flag, ParticleFlag::annihilate)) {
      // Load particle quantities
      Interpolator2D<spline_t> interp;
      int c1 = dev_mesh.get_c1(c);
      int c2 = dev_mesh.get_c2(c);
      int sp = get_ptc_type(flag);
      auto w = ptc.weight[idx];
      auto old_x1 = ptc.x1[idx], old_x2 = ptc.x2[idx];

      Pos_t new_x1 = 0.5f;
      Pos_t new_x2 = 0.5f;

      // Move the particles to be annihilated to the center of the cell
      ptc.x1[idx] = new_x1;
      ptc.x2[idx] = new_x2;

      // Deposit extra current due to this movement
      if (!check_bit(flag, ParticleFlag::ignore_current)) {
        // Scalar djz[spline_t::support + 1][spline_t::support + 1] =
        // {0.0f};
        Scalar weight = -dev_charges[sp] * w;

        Scalar djy[3] = {0.0f};
        for (int j = -1; j <= 0; j++) {
          Scalar sy0 = interp.interpolate(-old_x2 + j + 1);
          Scalar sy1 = interp.interpolate(-new_x2 + j + 1);

          size_t j_offset = (j + c2) * j1.pitch;
          Scalar djx = 0.0f;
          for (int i = -1; i <= 0; i++) {
            Scalar sx0 = interp.interpolate(-old_x1 + i + 1);
            Scalar sx1 = interp.interpolate(-new_x1 + i + 1);

            // j1 is movement in r
            int offset = j_offset + (i + c1) * sizeof(Scalar);
            Scalar val0 = movement2d(sy0, sy1, sx0, sx1);
            djx += val0;
            atomicAdd(ptrAddr(j1, offset + sizeof(Scalar)),
                      weight * djx * dev_mesh.delta[0] *
                          dev_mesh.delta[1] /
                          (dt * *ptrAddr(mesh_ptrs.A1_e,
                                         offset + sizeof(Scalar))));

            // j2 is movement in theta
            Scalar val1 = movement2d(sx0, sx1, sy0, sy1);
            djy[i + 1] += val1;
            atomicAdd(
                ptrAddr(j2, offset + j2.pitch),
                weight * djy[i + 1] * dev_mesh.delta[0] *
                    dev_mesh.delta[1] /
                    (dt * *ptrAddr(mesh_ptrs.A2_e, offset + j2.pitch)));
          }
        }
      }

      // Actually kill the particle
      ptc.cell[idx] = MAX_CELL;
      ptc.flag[idx] = 0;
    }
  }
}

__global__ void
flag_annihilation(particle_data ptc, size_t num, hipPitchedPtr dens_e,
                  hipPitchedPtr dens_p, hipPitchedPtr balance,
                  hipPitchedPtr annihilate) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    auto c = ptc.cell[i];
    // Skip empty particles
    if (c == MAX_CELL) continue;

    auto flag = ptc.flag[i];
    if (get_ptc_type(flag) > 1) continue;  // ignore ions
    auto w = ptc.weight[i];

    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    Scalar sin_t = std::sin(dev_mesh.pos(1, c2, 0.5f));
    int ann = *(int *)((char *)annihilate.ptr + c1 * sizeof(int) +
                       c2 * annihilate.pitch);
    Scalar n_min = 0.2 * square(dev_mesh.inv_delta[0]) * sin_t;
    Scalar n;
    if (ann != 0) {
      if (get_ptc_type(flag) == (int)ParticleType::electron)
        n = atomicAdd(ptrAddr(dens_e, c1, c2), w);
      else  // if (get_ptc_type(flag) == (int)ParticleType::positron)
        n = atomicAdd(ptrAddr(dens_p, c1, c2), w);
      if (n < n_min) {
        set_bit(ptc.flag[i], ParticleFlag::annihilate);
        atomicAdd(ptrAddr(balance, c1, c2),
                  w * (get_ptc_type(flag) == (int)ParticleType::electron
                           ? -1.0f
                           : 1.0f));
      }
    }
    // Scalar sin_t = std::sin(dev_mesh.pos(1, c2, 0.5f));
    // // size_t offset = c1 * sizeof(Scalar) + c2 * dens.pitch;

    // Scalar n_e = *ptrAddr(dens_e, c1, c2);
    // Scalar n_p = *ptrAddr(dens_p, c1, c2);
    // if (get_ptc_type(flag) == (int)ParticleType::electron)
    //   n_e = atomicAdd(ptrAddr(dens_e, c1, c2), w);
    // else if (get_ptc_type(flag) == (int)ParticleType::positron)
    //   n_p = atomicAdd(ptrAddr(dens_p, c1, c2), w);
    // Scalar r = std::exp(dev_mesh.pos(0, c1, 0.5f));
    // Scalar n_min = 0.2 * square(dev_mesh.inv_delta[0]) * sin_t;
    // // TODO: implement the proper condition
    // if (n_e > n_min && n_p > n_min) {
    //   set_bit(ptc.flag[i], ParticleFlag::annihilate);
    //   atomicAdd(ptrAddr(balance, c1, c2),
    //             w * (get_ptc_type(flag) ==
    //             (int)ParticleType::electron
    //                      ? -1.0f
    //                      : 1.0f));
    // }
  }
  // After this operation, the balance array will contain how much
  // imbalance is there in the annihilated part. We will add this
  // imbalance back in as an extra particle
}

__global__ void
check_annihilation(particle_data ptc, size_t num, hipPitchedPtr dens_e,
                   hipPitchedPtr dens_p, hipPitchedPtr annihilate) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    auto c = ptc.cell[i];
    // Skip empty particles
    if (c == MAX_CELL) continue;

    auto flag = ptc.flag[i];
    if (get_ptc_type(flag) > 1) continue;  // ignore ions
    auto w = ptc.weight[i];

    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    size_t offset = c1 * sizeof(int) + c2 * annihilate.pitch;
    int b = *(int *)((char *)annihilate.ptr + offset);
    Scalar sin_t = std::sin(dev_mesh.pos(1, c2, 0.5f));
    // size_t offset = c1 * sizeof(Scalar) + c2 * dens.pitch;

    Scalar n_e = *ptrAddr(dens_e, c1, c2);
    Scalar n_p = *ptrAddr(dens_p, c1, c2);
    if (get_ptc_type(flag) == (int)ParticleType::electron)
      n_e = atomicAdd(ptrAddr(dens_e, c1, c2), w);
    else if (get_ptc_type(flag) == (int)ParticleType::positron)
      n_p = atomicAdd(ptrAddr(dens_p, c1, c2), w);
    Scalar n_min = 0.2 * square(dev_mesh.inv_delta[0]) * sin_t;

    if (n_e > n_min && n_p > n_min && b == 0) {
      atomicExch((int *)((char *)annihilate.ptr + offset), 1);
    }
  }
}

__global__ void
add_extra_particles(particle_data ptc, size_t num,
                    hipPitchedPtr balance) {
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  int num_offset = n2 * dev_mesh.dims[0] + n1;
  Scalar w = *ptrAddr(balance, n1, n2);

  if (std::abs(w) > EPS) {
    ptc.cell[num + num_offset] = num_offset;
    ptc.x1[num + num_offset] = 0.5f;
    ptc.x2[num + num_offset] = 0.5f;
    ptc.x3[num + num_offset] = 0.0f;
    ptc.p1[num + num_offset] = 0.0f;
    ptc.p2[num + num_offset] = 0.0f;
    ptc.p3[num + num_offset] = 0.0f;
    ptc.E[num + num_offset] = 1.0f;
    ptc.weight[num + num_offset] = std::abs(w);
    if (w > 0)
      ptc.flag[num + num_offset] =
          set_ptc_type_flag(0, ParticleType::positron);
    else
      ptc.flag[num + num_offset] =
          set_ptc_type_flag(0, ParticleType::electron);
  }
}

__global__ void
filter_current(hipPitchedPtr j, hipPitchedPtr j_tmp,
               hipPitchedPtr A) {
  // Load position parameters
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  size_t globalOffset = n2 * j.pitch + n1 * sizeof(Scalar);

  // Do the actual computation here
  (*ptrAddr(j_tmp, globalOffset)) =
      0.25f * *ptrAddr(j, globalOffset) * *ptrAddr(A, globalOffset);
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.125f * *ptrAddr(j, globalOffset + sizeof(Scalar)) *
      *ptrAddr(A, globalOffset + sizeof(Scalar));
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.125f * *ptrAddr(j, globalOffset - sizeof(Scalar)) *
      *ptrAddr(A, globalOffset - sizeof(Scalar));
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.125f * *ptrAddr(j, globalOffset + j.pitch) *
      *ptrAddr(A, globalOffset + A.pitch);
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.125f * *ptrAddr(j, globalOffset - j.pitch) *
      *ptrAddr(A, globalOffset - A.pitch);
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.0625f * *ptrAddr(j, globalOffset + sizeof(Scalar) + j.pitch) *
      *ptrAddr(A, globalOffset + sizeof(Scalar) + j.pitch);
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.0625f * *ptrAddr(j, globalOffset - sizeof(Scalar) + j.pitch) *
      *ptrAddr(A, globalOffset - sizeof(Scalar) + j.pitch);
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.0625f * *ptrAddr(j, globalOffset + sizeof(Scalar) - j.pitch) *
      *ptrAddr(A, globalOffset + sizeof(Scalar) - A.pitch);
  (*ptrAddr(j_tmp, globalOffset)) +=
      0.0625f * *ptrAddr(j, globalOffset - sizeof(Scalar) - j.pitch) *
      *ptrAddr(A, globalOffset - sizeof(Scalar) - A.pitch);
  (*ptrAddr(j_tmp, globalOffset)) /= *ptrAddr(A, globalOffset);
}

}  // namespace Kernels

PtcUpdaterLogSph::PtcUpdaterLogSph(const cu_sim_environment &env)
    : PtcUpdaterDev(env),
      d_rand_states(nullptr),
      m_threadsPerBlock(256),
      m_blocksPerGrid(128),
      m_dens_e(env.local_grid()),
      m_dens_p(env.local_grid()),
      m_balance(env.local_grid()),
      m_annihilate(env.local_grid().extent()) {
  const Grid_LogSph_dev &grid =
      dynamic_cast<const Grid_LogSph_dev &>(env.grid());
  // TODO: Check error!!
  m_mesh_ptrs = grid.get_mesh_ptrs();

  int seed = m_env.params().random_seed;
  CudaSafeCall(hipMalloc(
      &d_rand_states,
      m_threadsPerBlock * m_blocksPerGrid * sizeof(hiprandState)));
  init_rand_states((hiprandState *)d_rand_states, seed,
                   m_threadsPerBlock, m_blocksPerGrid);

  // m_J1.initialize();
  // m_J2.initialize();
}

PtcUpdaterLogSph::~PtcUpdaterLogSph() {
  hipFree((hiprandState *)d_rand_states);
}

void
PtcUpdaterLogSph::update_particles(cu_sim_data &data, double dt,
                                   uint32_t step) {
  initialize_dev_fields(data);

  if (m_env.grid().dim() == 2) {
    auto &mesh = m_env.grid().mesh();
    // Skip empty particle array
    if (data.particles.number() > 0) {
      Logger::print_info(
          "Updating {} particles in log spherical coordinates",
          data.particles.number());
      Kernels::vay_push_2d<<<256, 512>>>(data.particles.data(),
                                         data.particles.number(),
                                         m_dev_fields, dt);
      hipDeviceSynchronize();
      CudaCheckError();
      data.J.initialize();
      for (auto &rho : data.Rho) {
        rho.initialize();
      }
      // m_J1.initialize();
      // m_J2.initialize();
      // Logger::print_info(
      //     "right before deposit, m_dev_fields.J3 ptr: {}, pitch: {},
      //     " "xsize: {}, ysize: {}", m_dev_fields.J3.ptr,
      //     m_dev_fields.J3.pitch, m_dev_fields.J3.xsize,
      //     m_dev_fields.J3.ysize);
      Kernels::deposit_current_2d_log_sph<<<256, 512>>>(
          data.particles.data(), data.particles.number(), data.J.ptr(0),
          data.J.ptr(1), data.J.ptr(2), m_dev_fields.Rho, m_mesh_ptrs,
          dt, step);
      hipDeviceSynchronize();
      CudaCheckError();
      Kernels::process_j<<<dim3(32, 32), dim3(32, 32)>>>(
          m_dev_fields, m_mesh_ptrs, dt);
      hipDeviceSynchronize();
      CudaCheckError();

      // Kernels::convert_j<<<dim3(32, 32), dim3(32, 32)>>>(
      //     m_J1.ptr(), m_J2.ptr(), m_dev_fields);
      // CudaCheckError();
      dim3 blockSize(32, 16);
      dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 16);
      for (int n = 0; n < m_env.params().current_smoothing; n++) {
        Kernels::filter_current<<<gridSize, blockSize>>>(
            data.J.ptr(0), m_dens_e.ptr(), m_mesh_ptrs.A1_e);
        data.J.data(0).copy_from(m_dens_e.data());
        Kernels::filter_current<<<gridSize, blockSize>>>(
            data.J.ptr(1), m_dens_e.ptr(), m_mesh_ptrs.A2_e);
        data.J.data(1).copy_from(m_dens_e.data());
        CudaCheckError();
      }
    }
    // Skip empty particle array
    if (data.photons.number() > 0) {
      Logger::print_info(
          "Updating {} photons in log spherical coordinates",
          data.photons.number());
      Kernels::move_photons<<<256, 512>>>(data.photons.data(),
                                          data.photons.number(), dt);
      CudaCheckError();
    }
  }
  hipDeviceSynchronize();
}

void
PtcUpdaterLogSph::handle_boundary(cu_sim_data &data) {
  data.particles.clear_guard_cells();
  data.photons.clear_guard_cells();

  Kernels::boundary_rho<<<32, 512>>>(m_dev_fields, m_mesh_ptrs);
  CudaCheckError();
  hipDeviceSynchronize();
}

void
PtcUpdaterLogSph::inject_ptc(cu_sim_data &data, int inj_per_cell,
                             Scalar p1, Scalar p2, Scalar p3, Scalar w,
                             Scalar omega) {
  Kernels::inject_ptc<<<m_blocksPerGrid, m_threadsPerBlock>>>(
      data.particles.data(), data.particles.number(), inj_per_cell, p1,
      p2, p3, w, data.Rho[0].ptr(), data.Rho[2].ptr(),
      (hiprandState *)d_rand_states, omega);
  CudaCheckError();

  data.particles.set_num(data.particles.number() +
                         2 * inj_per_cell *
                             data.E.grid().mesh().reduced_dim(1));
}

void
PtcUpdaterLogSph::initialize_dev_fields(cu_sim_data &data) {
  if (!m_fields_initialized) {
    m_dev_fields.E1 = data.E.ptr(0);
    m_dev_fields.E2 = data.E.ptr(1);
    m_dev_fields.E3 = data.E.ptr(2);
    m_dev_fields.B1 = data.B.ptr(0);
    m_dev_fields.B2 = data.B.ptr(1);
    m_dev_fields.B3 = data.B.ptr(2);
    m_dev_fields.J1 = data.J.ptr(0);
    m_dev_fields.J2 = data.J.ptr(1);
    m_dev_fields.J3 = data.J.ptr(2);
    // Logger::print_info(
    //     "m_dev_fields.J3 pitch: {}, xsize: {}, ysize: {}",
    //     m_dev_fields.J3.pitch, m_dev_fields.J3.xsize,
    //     m_dev_fields.J3.ysize);
    for (int i = 0; i < data.num_species; i++) {
      m_dev_fields.Rho[i] = data.Rho[i].ptr();
    }
    CudaSafeCall(
        hipMemcpyToSymbol(HIP_SYMBOL(dev_fields), (void *)&m_dev_fields,
                           sizeof(PtcUpdaterDev::fields_data)));
    m_fields_initialized = true;
  }
}

void
PtcUpdaterLogSph::annihilate_extra_pairs(cu_sim_data &data, double dt) {
  m_dens_e.data().assign_dev(0.0);
  m_dens_p.data().assign_dev(0.0);
  m_annihilate.assign_dev(0);

  Kernels::check_annihilation<<<256, 512>>>(
      data.particles.data(), data.particles.number(), m_dens_e.ptr(),
      m_dens_p.ptr(), m_annihilate.data_d());
  CudaCheckError();

  m_dens_e.data().assign_dev(0.0);
  m_dens_p.data().assign_dev(0.0);
  m_balance.data().assign_dev(0.0);
  Kernels::flag_annihilation<<<256, 512>>>(
      data.particles.data(), data.particles.number(), m_dens_e.ptr(),
      m_dens_p.ptr(), m_balance.ptr(), m_annihilate.data_d());
  CudaCheckError();

  Kernels::annihilate_pairs<<<256, 512>>>(
      data.particles.data(), data.particles.number(), data.J.ptr(0),
      data.J.ptr(1), data.J.ptr(2), m_mesh_ptrs, dt);
  CudaCheckError();

  auto &mesh = data.E.grid().mesh();
  dim3 blockSize(32, 16);
  dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 16);

  Kernels::add_extra_particles<<<gridSize, blockSize>>>(
      data.particles.data(), data.particles.number(), m_balance.ptr());
  CudaCheckError();

  hipDeviceSynchronize();
  data.particles.set_num(data.particles.number() +
                         mesh.reduced_dim(0) * mesh.reduced_dim(1));
}

}  // namespace Aperture