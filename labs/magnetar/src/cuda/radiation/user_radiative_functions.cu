#include "hip/hip_runtime.h"
#ifndef _USER_RADIATIVE_FUNCTIONS_H_
#define _USER_RADIATIVE_FUNCTIONS_H_

#include "cuda/constant_mem.h"
#include "cuda/cudarng.h"
#include "cuda/data_ptrs.h"
#include "cuda/grids/grid_log_sph_ptrs.h"
#include "cuda/utils/interpolation.cuh"
#include "grids/grid_log_sph.h"
#include "sim_environment.h"

namespace Aperture {

namespace Kernels {

__device__ mesh_ptrs_log_sph dev_mesh_ptrs_log_sph;

__device__ Scalar rho_c(Scalar r, Scalar theta) {
  Scalar c = std::cos(theta);
  Scalar c2 = c * c;
  Scalar s = std::sin(theta);
  if (std::abs(s) < 1.0e-5) return 1.0e10;
  return r * std::pow(1.0f + 3.0f * c2, 1.5) / (3.0f * s * (1.0f + c2));
}

__device__ bool
check_emit_photon(data_ptrs& data, uint32_t tid, CudaRng& rng) {
  auto& ptc = data.particles;
  bool emit = check_bit(ptc.flag[tid], ParticleFlag::emit_photon);

  if (emit) {
    ptc.flag[tid] &= ~bit_or(ParticleFlag::emit_photon);
  }
  return emit;
}

__device__ void
emit_photon(data_ptrs& data, uint32_t tid, int offset, CudaRng& rng) {
  auto& ptc = data.particles;
  auto& photons = data.photons;

  auto c = ptc.cell[tid];
  Scalar p1 = ptc.p1[tid];
  Scalar p2 = ptc.p2[tid];
  Scalar p3 = ptc.p3[tid];
  auto x1 = ptc.x1[tid];
  auto x2 = ptc.x2[tid];
  auto c1 = dev_mesh.get_c1(c);
  auto c2 = dev_mesh.get_c2(c);
  Scalar r = exp(dev_mesh.pos(0, c1, x1));
  Scalar theta = dev_mesh.pos(1, c2, x2);
  Scalar gamma = ptc.E[tid];
  Scalar pi = std::sqrt(p1 * p1 + p2 * p2 + p3 * p3);
  // Scalar u = rng();

  Interpolator2D<Spline::spline_t<1>> interp;
  Scalar B1 = interp(data.B1, x1, x2, c1, c2, Stagger(0b001));
  Scalar B2 = interp(data.B2, x1, x2, c1, c2, Stagger(0b010));
  Scalar B3 = interp(data.B3, x1, x2, c1, c2, Stagger(0b100));
  Scalar B = sqrt(B1 * B1 + B2 * B2 + B3 * B3);
  Scalar pdotB = (p1 * B1 + p2 * B2 + p3 * B3);
  Scalar p_mag_signed = sgn(pdotB) * sgn(B1) * std::abs(pdotB) / B;
  Scalar g = sqrt(1.0f + p_mag_signed * p_mag_signed);
  Scalar mu = std::abs(B1 / B);
  Scalar y = (B / dev_params.BQ) /
             (dev_params.star_kT * (g - p_mag_signed * mu));
  if (y < 20.0f && y > 0.0f) {
    Scalar coef = dev_params.res_drag_coef * y * y * y /
                  (r * r * (std::exp(y) - 1.0f));
    Scalar Ndot = std::abs(coef * (1.0f - p_mag_signed * mu / g));
    float theta_p = CONST_PI * rng();
    float u = cos(theta_p);

    // The abs is not necessary?
    Scalar Eph = std::abs(
        gamma * (g - std::abs(p_mag_signed) * u) *
        (1.0f - 1.0f / std::sqrt(1.0f + 2.0f * B / dev_params.BQ)));
    if (Eph < 2.1f) {
      // Treat this as hard X-ray emission
        Scalar angle =
            acos(sgn(pdotB) * (B1 * cos(theta) - B2 * sin(theta)) / B);
        float phi_p = 2.0f * CONST_PI * rng();
        Scalar cos_angle =
            std::cos(angle) * std::cos(theta_p) +
            std::sin(angle) * std::sin(theta_p) * std::cos(phi_p);
        angle = std::acos(cos_angle);
        auto& ph_flux = data.ph_flux;
        if (p1 > 0.0f && gamma > 1.5f) {
          Eph = std::log(std::abs(Eph)) / std::log(10.0f);
          if (Eph < -6.0f) Eph = -6.0f;
          int n0 = ((Eph + 6.0f) / 8.1f * (ph_flux.p.xsize - 1));
          if (n0 < 0) n0 = 0;
          if (n0 >= ph_flux.p.xsize) n0 = ph_flux.p.xsize - 1;
          int n1 = (std::abs(angle) / (CONST_PI + 1.0e-5)) *
                   (ph_flux.p.ysize - 1);
          if (n1 < 0) n1 = 0;
          if (n1 >= ph_flux.p.ysize) n1 = ph_flux.p.ysize - 1;
          auto w = ptc.weight[tid];
          atomicAdd(&ph_flux(n0, n1), Ndot * dev_params.delta_t * w);
          // printf("n0 is %d, n1 is %d, Ndot is %f, ph_flux is %f\n",
          // n0,
          //        n1, Ndot, ph_flux(n0, n1));
        }
      Scalar pf = std::sqrt(square(max(gamma - Eph * Ndot, 1.1f)) - 1.0f);
      ptc.p1[tid] = p1 * pf / pi;
      ptc.p2[tid] = p2 * pf / pi;
      ptc.p3[tid] = p3 * pf / pi;
      ptc.E[tid] = std::sqrt(1.0 + ptc.p1[tid] * ptc.p1[tid] +
                             ptc.p2[tid] * ptc.p2[tid] +
                             ptc.p3[tid] * ptc.p3[tid]);
    } else {
      // Treat this as a discrete photon emission
      if (Eph > gamma - 1.0f) Eph = gamma - 1.1f;
      float v = rng();
      if (v < Ndot * dev_params.delta_t) {
        Scalar pf = std::sqrt(square(gamma - Eph) - 1.0f);
        // gamma = (gamma - std::abs(Eph));
        ptc.p1[tid] = p1 * pf / pi;
        ptc.p2[tid] = p2 * pf / pi;
        ptc.p3[tid] = p3 * pf / pi;
        ptc.E[tid] = std::sqrt(1.0 + ptc.p1[tid] * ptc.p1[tid] +
                               ptc.p2[tid] * ptc.p2[tid] +
                               ptc.p3[tid] * ptc.p3[tid]);
        if (ptc.E[tid] != ptc.E[tid]) {
          printf(
              "NaN detected in photon emission! p1 is %f, p2 is %f, p3 "
              "is "
              "%f, gamma "
              "is %f\n",
              p1, p2, p3, gamma);
          asm("trap;");
          // p1 = p2 = p3 = 0.0f;
        }

        photons.x1[offset] = ptc.x1[tid];
        photons.x2[offset] = ptc.x2[tid];
        photons.x3[offset] = ptc.x3[tid];
        photons.p1[offset] = Eph * p1 / pi;
        photons.p2[offset] = Eph * p2 / pi;
        photons.p3[offset] = Eph * p3 / pi;
        photons.E[offset] = Eph;
        photons.weight[offset] = ptc.weight[tid];
        photons.path_left[offset] = dev_params.photon_path;
        photons.cell[offset] = ptc.cell[tid];
      }
    }
  }
}

__device__ bool
check_produce_pair(data_ptrs& data, uint32_t tid, CudaRng& rng) {
  auto& photons = data.photons;
  uint32_t cell = photons.cell[tid];
  int c1 = dev_mesh.get_c1(cell);
  int c2 = dev_mesh.get_c2(cell);
  // auto x1 = data.photons.x1[tid];
  auto x2 = data.photons.x2[tid];
  // auto p1 = data.photons.p1[tid];
  // auto p2 = data.photons.p2[tid];
  // auto p3 = data.photons.p3[tid];
  // auto Eph = data.photons.E[tid];
  Scalar theta = dev_mesh.pos(1, c2, x2);
  Scalar r = exp(dev_mesh.pos(0, c1, x1));
  // Do not care about photons in the first and last theta cell
  if (theta < dev_mesh.delta[1] ||
      theta > CONST_PI - dev_mesh.delta[1]) {
    photons.cell[tid] = MAX_CELL;
    return false;
  }

  // Scalar rho = max(
  //     std::abs(data.Rho[0](c1, c2) + data.Rho[1](c1, c2)),
  //     0.0001f);
  Scalar N = dev_params.q_e * std::abs(data.Rho[0](c1, c2)) + std::abs(data.Rho[1](c1, c2));
  // Scalar multiplicity = N / rho;
  // if (multiplicity > 100.0f) {
  if (N > 2.0f * square(1.0f / dev_mesh.delta[1] / r) * sin(theta)) {
    // Multiplicity already too high, kill photon but do not make a pair
    photons.cell[tid] = MAX_CELL;
    return false;
  }
  return (photons.path_left[tid] <= 0.0f);
}

__device__ void
produce_pair(data_ptrs& data, uint32_t tid, uint32_t offset,
             CudaRng& rng) {
  auto& ptc = data.particles;
  auto& photons = data.photons;

  Scalar p1 = photons.p1[tid];
  Scalar p2 = photons.p2[tid];
  Scalar p3 = photons.p3[tid];
  Scalar E_ph2 = p1 * p1 + p2 * p2 + p3 * p3;
  if (E_ph2 <= 4.01f) E_ph2 = 4.01f;

  Scalar ratio = std::sqrt(0.25f - 1.0f / E_ph2);
  Scalar gamma = sqrt(1.0f + ratio * ratio * E_ph2);

  if (gamma != gamma) {
    photons.cell[tid] = MAX_CELL;
    return;
  }
  // Add the two new particles
  int offset_e = offset;
  int offset_p = offset + 1;
  // int offset_p = ptc_num + start_pos + pos_in_block +
  // pair_count[blockIdx.x];

  ptc.x1[offset_e] = ptc.x1[offset_p] = photons.x1[tid];
  ptc.x2[offset_e] = ptc.x2[offset_p] = photons.x2[tid];
  ptc.x3[offset_e] = ptc.x3[offset_p] = photons.x3[tid];
  // printf("x1 = %f, x2 = %f, x3 = %f\n", ptc.x1[offset_e],
  // ptc.x2[offset_e], ptc.x3[offset_e]);

  ptc.p1[offset_e] = ptc.p1[offset_p] = ratio * p1;
  ptc.p2[offset_e] = ptc.p2[offset_p] = ratio * p2;
  ptc.p3[offset_e] = ptc.p3[offset_p] = ratio * p3;
  ptc.E[offset_e] = ptc.E[offset_p] = gamma;

#ifndef NDEBUG
  assert(ptc.cell[offset_e] == MAX_CELL);
  assert(ptc.cell[offset_p] == MAX_CELL);
#endif

  // float u = rng();
  ptc.weight[offset_e] = ptc.weight[offset_p] = photons.weight[tid];
  ptc.cell[offset_e] = ptc.cell[offset_p] = photons.cell[tid];
  ptc.flag[offset_e] = set_ptc_type_flag(
           bit_or(ParticleFlag::secondary), ParticleType::electron);
  ptc.flag[offset_p] = set_ptc_type_flag(
           bit_or(ParticleFlag::secondary), ParticleType::positron);

  // Set this photon to be empty
  photons.cell[tid] = MAX_CELL;
}

}  // namespace Kernels

void
user_rt_init(sim_environment& env) {
  // Copy the mesh pointer to device memory
  Grid_LogSph* grid = dynamic_cast<Grid_LogSph*>(&env.local_grid());
  auto ptrs = get_mesh_ptrs(*grid);
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Kernels::dev_mesh_ptrs_log_sph),
                                  (void*)&ptrs,
                                  sizeof(mesh_ptrs_log_sph)));
}

}  // namespace Aperture

#endif  // _USER_RADIATIVE_FUNCTIONS_H_
