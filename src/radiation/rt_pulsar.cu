#include "hip/hip_runtime.h"
#include "cuda/constant_mem.h"
#include "cuda/cudaUtility.h"
#include "cuda/cuda_control.h"
#include "cuda/cudarng.h"
#include "cuda/kernels.h"
#include "data/detail/multi_array_utils.hpp"
#include "data/particles.h"
#include "data/particles_1d.h"
#include "data/photons.h"
#include "data/photons_1d.h"
#include "radiation/curvature_instant.h"
#include "radiation/rt_pulsar.h"
#include "sim_environment.h"
#include "utils/logger.h"
#include "utils/util_functions.h"
#include <thrust/device_ptr.h>
#include <thrust/scan.h>

namespace Aperture {

namespace Kernels {

template <typename PtcData>
__global__ void
count_photon_produced(PtcData ptc, size_t number, int* ph_count,
                      int* phPos, hiprandState* states,
                      hipPitchedPtr ph_events) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  // CudaRng rng(&states[id]);
  // auto inv_comp = make_inverse_compton_PL(dev_params.spectral_alpha,
  // dev_params.e_s,
  //                                         dev_params.e_min,
  //                                         dev_params.photon_path,
  //                                         rng);
  // CurvatureInstant<Kernels::CudaRng> rad_model(dev_params, rng);
  // auto inv_comp = make_inverse_compton_dummy(10.0, )
  __shared__ int photonProduced;
  if (threadIdx.x == 0) photonProduced = 0;

  __syncthreads();

  for (uint32_t tid = id; tid < number; tid += blockDim.x * gridDim.x) {
    uint32_t cell = ptc.cell[tid];
    // Skip empty particles
    if (cell == MAX_CELL) continue;
    int c1 = dev_mesh.get_c1(cell);

    // Skip photon emission when outside given radius
    Scalar r = std::exp(dev_mesh.pos(0, c1, ptc.x1[tid]));
    Scalar gamma = ptc.E[tid];

    // if (rad_model.emit_photon(gamma)) {
    if (gamma > dev_params.gamma_thr && r < dev_params.r_cutoff && r > 1.02f) {
      phPos[tid] = atomicAdd(&photonProduced, 1) + 1;
      int c2 = dev_mesh.get_c2(cell);
      atomicAdd(ptrAddr(ph_events,
                        c2 * ph_events.pitch + c1 * sizeof(Scalar)),
                1.0f);
    }
  }

  __syncthreads();

  // Record the number of photons produced this block to global array
  if (threadIdx.x == 0) {
    ph_count[blockIdx.x] = photonProduced;
  }
}

template <typename PtcData, typename PhotonData>
__global__ void
produce_photons(PtcData ptc, size_t ptc_num, PhotonData photons,
                size_t ph_num, int* phPos, int* ph_count, int* ph_cum,
                hiprandState* states) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  CudaRng rng(&states[id]);
  // auto inv_comp = make_inverse_compton_PL(dev_params.spectral_alpha,
  // dev_params.e_s,
  //                                         dev_params.e_min,
  //                                         dev_params.photon_path,
  //                                         rng);
  // CurvatureInstant<Kernels::CudaRng> rad_model(dev_params, rng);
  for (uint32_t tid = id; tid < ptc_num;
       tid += blockDim.x * gridDim.x) {
    int pos_in_block = phPos[tid] - 1;
    if (pos_in_block > -1) {
      int start_pos = ph_cum[blockIdx.x];

      // TODO: Compute gamma
      Scalar p1 = ptc.p1[tid];
      Scalar p2 = ptc.p2[tid];
      Scalar p3 = ptc.p3[tid];
      // Scalar gamma = sqrt(1.0f + p1 * p1 + p2 * p2 + p3 * p3);
      Scalar gamma = ptc.E[tid];
      Scalar pi = std::sqrt(gamma * gamma - 1.0f);
      // Scalar Eph = rad_model.draw_photon_energy(gamma, p);
      Scalar Eph = dev_params.E_secondary * 2.0f;
      Scalar pf = std::sqrt(square(gamma - Eph) - 1.0f);
      // gamma = (gamma - std::abs(Eph));
      ptc.p1[tid] = p1 * pf / pi;
      ptc.p2[tid] = p2 * pf / pi;
      ptc.p3[tid] = p3 * pf / pi;

      // If photon energy is too low, do not track it, but still
      // subtract its energy as done above
      // if (std::abs(Eph) < dev_params.E_ph_min) continue;

      // Add the new photon
      // Scalar path = rad_model.draw_photon_freepath(Eph);
      Scalar u = rng();
      Scalar path =
          dev_params.photon_path * std::sqrt(-2.0f * std::log(u));
      // Scalar path = dev_params.photon_path;
      // if (path > dev_params.lph_cutoff) continue;
      // if (true) continue;
      // printf("Eph is %f, path is %f\n", Eph, path);
      int offset = ph_num + start_pos + pos_in_block;
      photons.x1[offset] = ptc.x1[tid];
      photons.x2[offset] = ptc.x2[tid];
      photons.x3[offset] = ptc.x3[tid];
      photons.p1[offset] = Eph * p1 / pi;
      photons.p2[offset] = Eph * p2 / pi;
      photons.p3[offset] = Eph * p3 / pi;
      photons.weight[offset] = ptc.weight[tid];
      photons.path_left[offset] = path;
      photons.cell[offset] = ptc.cell[tid];
    }
  }
}

template <typename PhotonData>
__global__ void
count_pairs_produced(PhotonData photons, size_t number, int* pair_count,
                     int* pair_pos, hiprandState* states,
                     hipPitchedPtr pair_events) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  // CudaRng rng(&states[id]);
  // auto inv_comp = make_inverse_compton_PL(dev_params.spectral_alpha,
  // dev_params.e_s,
  //                                         dev_params.e_min,
  //                                         dev_params.photon_path,
  //                                         rng);
  __shared__ int pairsProduced;
  if (threadIdx.x == 0) pairsProduced = 0;

  __syncthreads();

  for (uint32_t tid = id; tid < number; tid += blockDim.x * gridDim.x) {
    // if (tid >= number) continue;
    uint32_t cell = photons.cell[tid];
    // Skip empty photons
    if (cell == MAX_CELL) continue;

    if (photons.path_left[tid] <= 0.0f) {
      pair_pos[tid] = atomicAdd(&pairsProduced, 1) + 1;
      int c1 = dev_mesh.get_c1(cell);
      int c2 = dev_mesh.get_c2(cell);

      atomicAdd(ptrAddr(pair_events,
                        c2 * pair_events.pitch + c1 * sizeof(Scalar)),
                1.0f);
    }
  }

  __syncthreads();

  // Record the number of pairs produced this block to global array
  if (threadIdx.x == 0) {
    pair_count[blockIdx.x] = pairsProduced;
  }
}

template <typename PtcData, typename PhotonData>
__global__ void
produce_pairs(PhotonData photons, size_t ph_num, PtcData ptc,
              size_t ptc_num, int* pair_pos, int* pair_count,
              int* pair_cum, hiprandState* states) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  // CudaRng rng(&states[id]);
  // auto inv_comp = make_inverse_compton_PL1D(dev_params,
  // dev_params.photon_path, rng); RadModel rad_model(dev_params, rng);

  for (uint32_t tid = id; tid < ph_num; tid += blockDim.x * gridDim.x) {
    int pos_in_block = pair_pos[tid] - 1;
    if (pos_in_block > -1 && photons.cell[tid] != MAX_CELL) {
      int start_pos = pair_cum[blockIdx.x] * 2;

      // Split the photon energy evenly between the pairs
      Scalar p1 = photons.p1[tid];
      Scalar p2 = photons.p2[tid];
      Scalar p3 = photons.p3[tid];
      Scalar E_ph2 = p1 * p1 + p2 * p2 + p3 * p3;
      // Scalar new_p = std::sqrt(max(0.25f * E_ph * E_ph, 1.0f)
      // - 1.0f);
      Scalar ratio = std::sqrt(0.25f - 1.0f / E_ph2);

      // Add the two new particles
      int offset = ptc_num + start_pos + pos_in_block * 2;
      ptc.x1[offset] = ptc.x1[offset + 1] = photons.x1[tid];
      ptc.x2[offset] = ptc.x2[offset + 1] = photons.x2[tid];
      ptc.x3[offset] = ptc.x3[offset + 1] = photons.x3[tid];

      ptc.p1[offset] = ptc.p1[offset + 1] = ratio * p1;
      ptc.p2[offset] = ptc.p2[offset + 1] = ratio * p2;
      ptc.p3[offset] = ptc.p3[offset + 1] = ratio * p3;

      ptc.weight[offset] = ptc.weight[offset + 1] = photons.weight[tid];
      ptc.cell[offset] = ptc.cell[offset + 1] = photons.cell[tid];
      ptc.flag[offset] = set_ptc_type_flag(
          bit_or(ParticleFlag::secondary), ParticleType::electron);
      ptc.flag[offset + 1] = set_ptc_type_flag(
          bit_or(ParticleFlag::secondary), ParticleType::positron);

      // Set this photon to be empty
      photons.cell[tid] = MAX_CELL;
    }
  }
}

}  // namespace Kernels

RadiationTransferPulsar::RadiationTransferPulsar(const Environment& env)
    : m_env(env),
      d_rand_states(nullptr),
      m_threadsPerBlock(256),
      m_blocksPerGrid(512),
      m_numPerBlock(m_blocksPerGrid),
      m_cumNumPerBlock(m_blocksPerGrid),
      m_posInBlock(env.params().max_ptc_number),
      m_pair_events(env.local_grid()),
      m_ph_events(env.local_grid()) {
  int seed = m_env.params().random_seed;

  CudaSafeCall(hipMalloc(
      &d_rand_states,
      m_threadsPerBlock * m_blocksPerGrid * sizeof(hiprandState)));
  init_rand_states((hiprandState*)d_rand_states, seed, m_threadsPerBlock,
                   m_blocksPerGrid);

  m_pair_events.initialize();
  m_ph_events.initialize();
  // Kernels::init_rand_states<<<m_blocksPerGrid, m_threadsPerBlock>>>(
  //     (hiprandState*)d_rand_states, seed);
  // CudaCheckError();

  // Allocate auxiliary arrays for pair creation
  // CudaSafeCall(hipMalloc(&m_numPerBlock, m_blocksPerGrid *
  // sizeof(uint32_t))); CudaSafeCall(hipMalloc(&m_cumNumPerBlock,
  // m_blocksPerGrid * sizeof(uint32_t)));
  // CudaSafeCall(hipMalloc(&m_posInBlock,
  // m_env.params().max_ptc_number));
}

RadiationTransferPulsar::~RadiationTransferPulsar() {
  hipFree((hiprandState*)d_rand_states);
}

void
RadiationTransferPulsar::emit_photons(Photons& photons,
                                      Particles& ptc) {
  m_posInBlock.assign_dev(0, ptc.number());
  m_numPerBlock.assign_dev(0);
  m_cumNumPerBlock.assign_dev(0);

  hipDeviceSynchronize();
  // Logger::print_debug("Initialize finished");

  Kernels::count_photon_produced<particle_data>
      <<<m_blocksPerGrid, m_threadsPerBlock>>>(
          ptc.data(), ptc.number(), m_numPerBlock.data_d(),
          m_posInBlock.data_d(), (hiprandState*)d_rand_states,
          m_ph_events.ptr());
  CudaCheckError();

  thrust::device_ptr<int> ptrNumPerBlock(m_numPerBlock.data_d());
  thrust::device_ptr<int> ptrCumNum(m_cumNumPerBlock.data_d());

  hipDeviceSynchronize();
  // Logger::print_debug("Count finished");
  // Scan the number of photons produced per block. The last one will be
  // the total
  thrust::exclusive_scan(ptrNumPerBlock,
                         ptrNumPerBlock + m_blocksPerGrid, ptrCumNum);
  CudaCheckError();
  // Logger::print_debug("Scan finished");
  m_cumNumPerBlock.sync_to_host();
  m_numPerBlock.sync_to_host();
  int new_photons = m_cumNumPerBlock[m_blocksPerGrid - 1] +
                    m_numPerBlock[m_blocksPerGrid - 1];
  Logger::print_info("{} photons are produced!", new_photons);

  Kernels::produce_photons<particle_data, photon_data>
      <<<m_blocksPerGrid, m_threadsPerBlock>>>(
          ptc.data(), ptc.number(), photons.data(), photons.number(),
          m_posInBlock.data_d(), m_numPerBlock.data_d(),
          m_cumNumPerBlock.data_d(), (hiprandState*)d_rand_states);
  CudaCheckError();

  photons.set_num(photons.number() + new_photons);
  // Logger::print_info("There are {} photons in the pool",
  //                    photons.number());
}

void
RadiationTransferPulsar::produce_pairs(Particles& ptc,
                                       Photons& photons) {
  m_posInBlock.assign_dev(0, ptc.number());
  m_numPerBlock.assign_dev(0);
  m_cumNumPerBlock.assign_dev(0);

  Kernels::count_pairs_produced<photon_data>
      <<<m_blocksPerGrid, m_threadsPerBlock>>>(
          photons.data(), photons.number(), m_numPerBlock.data_d(),
          m_posInBlock.data_d(), (hiprandState*)d_rand_states,
          m_pair_events.ptr());
  CudaCheckError();

  thrust::device_ptr<int> ptrNumPerBlock(m_numPerBlock.data_d());
  thrust::device_ptr<int> ptrCumNum(m_cumNumPerBlock.data_d());

  // Scan the number of photons produced per block. The last one will be
  // the total
  thrust::exclusive_scan(ptrNumPerBlock,
                         ptrNumPerBlock + m_blocksPerGrid, ptrCumNum);
  m_cumNumPerBlock.sync_to_host();
  m_numPerBlock.sync_to_host();
  int new_pairs = (m_cumNumPerBlock[m_blocksPerGrid - 1] +
                   m_numPerBlock[m_blocksPerGrid - 1]);
  // Logger::print_info("{} electron-positron pairs are produced!",
  //                    new_pairs);

  Kernels::produce_pairs<particle_data, photon_data>
      <<<m_blocksPerGrid, m_threadsPerBlock>>>(
          photons.data(), photons.number(), ptc.data(), ptc.number(),
          m_posInBlock.data_d(), m_numPerBlock.data_d(),
          m_cumNumPerBlock.data_d(), (hiprandState*)d_rand_states);
  CudaCheckError();

  ptc.set_num(ptc.number() + new_pairs * 2);
  // Logger::print_info("There are {} particles in the pool",
  //                    ptc.number());
}

}  // namespace Aperture