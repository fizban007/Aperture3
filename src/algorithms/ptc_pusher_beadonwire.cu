#include "hip/hip_runtime.h"
#include "algorithms/ptc_pusher_beadonwire.h"
#include "sim_environment.h"
#include <array>
#include <cmath>
#include <fmt/ostream.h>
#include "utils/logger.h"
#include "utils/util_functions.h"
#include "cuda/cuda_control.h"
#include "cuda/cudaUtility.h"
#include "cuda/constant_mem.h"
#include "algorithms/functions.h"

namespace Aperture {

HD_INLINE double gamma(double beta_phi, double p) {
  double b2 = beta_phi * beta_phi;
  // if (beta_phi < 0) p = -p;

  // if (b2 > 1.0 && p*p/(1.0 + b2) + (1.0 - b2) < 0) {
  //   Logger::print_info("b2 is {}, p is {}, sqrt is {}, {}", b2, p, p*p/(1.0 + b2), (1.0 - b2));
  // }
  // double result = -p * b2 / std::sqrt(1.0 + b2) + std::sqrt(p*p/(1.0 + b2) + (1.0 - b2));
  // result *= 1.0 / (1.0 - b2);

  return std::sqrt(1.0 + p*p + b2);
}


namespace Kernels {

// TODO: consider fusing these kernels?

__global__
void lorentz_push(particle_data ptc, const Scalar* E, double dt, uint32_t num) {
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
       i < num;
       i += blockDim.x * gridDim.x) {
    if (!check_bit(ptc.flag[i], ParticleFlag::ignore_EM)) {
      auto c = ptc.cell[i];
      auto rel_x = ptc.x1[i];
      auto p1 = ptc.p1[i];
      int sp = get_ptc_type(ptc.flag[i]);
      Scalar E1 = E[c] * rel_x + E[c - 1] * (1.0 - rel_x);

      p1 += dev_charges[sp] * E1 * dt / dev_masses[sp];
      ptc.p1[i] = p1;
    }
  }
}

__global__
void move_ptc(particle_data ptc, double dt, uint32_t num) {
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
       i < num;
       i += blockDim.x * gridDim.x) {
    auto c = ptc.cell[i];
    auto p = ptc.p1[i];
    Scalar gamma = sqrt(1.0 + p*p);

    Scalar dx = p * dt / (gamma * dev_mesh.delta[0]);
    Scalar new_x1 = ptc.x1[i] + dx;
    int delta_c = floor(new_x1);
    c += delta_c;

    ptc.dx1[i] = dx;
    ptc.cell[i] = c;
    ptc.x1[i] = new_x1 - (Pos_t)delta_c;
  }
}

}


ParticlePusher_BeadOnWire::ParticlePusher_BeadOnWire(const Environment& env) :
  m_params(env.params()) {}

ParticlePusher_BeadOnWire::~ParticlePusher_BeadOnWire() {}

void
ParticlePusher_BeadOnWire::push(SimData& data, double dt) {
  Logger::print_info("In particle pusher");
  auto& grid = data.E.grid();
  auto& mesh = grid.mesh();
}

void
ParticlePusher_BeadOnWire::move_ptc(Particles& particles, double x,
                                    const Grid& grid, double dt) {
  auto& ptc = particles.data();
  auto& mesh = grid.mesh();
  if (mesh.dim() == 1) {
    Kernels::move_ptc<<<512, 512>>>(ptc, dt, particles.number());
    CudaCheckError();
  }
}

void
ParticlePusher_BeadOnWire::lorentz_push(Particles& particles, double x,
                                      const VectorField<Scalar>& E,
                                      const VectorField<Scalar>& B, double dt) {
  auto& ptc = particles.data();
  if (E.grid().dim() == 1) {
    Kernels::lorentz_push<<<512, 512>>>(ptc, E.ptr(0), dt, particles.number());
    CudaCheckError();
  }
}

void
ParticlePusher_BeadOnWire::handle_boundary(SimData &data) {
  auto& mesh = data.E.grid().mesh();
  auto& ptc = data.particles;
  if (ptc.number() > 0) {
    if (m_params.periodic_boundary[0] == false) {
      ptc.clear_guard_cells();
    }
  }
  // auto& photon = data.photons;
  // if (photon.number() > 0) {
  //   if (m_params.periodic_boundary[0] == false) {
  //     photon.clear_guard_cells();
  //   }
  // }

}

void
ParticlePusher_BeadOnWire::extra_force(Particles &particles, Index_t idx, double x, const Grid &grid, double dt) {
  auto& ptc = particles.data();
}

}  // namespace Aperture
