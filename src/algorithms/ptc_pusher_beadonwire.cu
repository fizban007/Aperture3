#include "hip/hip_runtime.h"
#include "algorithms/ptc_pusher_beadonwire.h"
#include "sim_environment.h"
#include <array>
#include <cmath>
#include <fmt/ostream.h>
#include "utils/logger.h"
#include "utils/util_functions.h"
#include "cuda/cuda_control.h"
#include "cuda/cudaUtility.h"
#include "cuda/constant_mem.h"
#include "algorithms/functions.h"

namespace Aperture {

HD_INLINE double gamma(double beta_phi, double p) {
  double b2 = beta_phi * beta_phi;
  // if (beta_phi < 0) p = -p;

  // if (b2 > 1.0 && p*p/(1.0 + b2) + (1.0 - b2) < 0) {
  //   Logger::print_info("b2 is {}, p is {}, sqrt is {}, {}", b2, p, p*p/(1.0 + b2), (1.0 - b2));
  // }
  // double result = -p * b2 / std::sqrt(1.0 + b2) + std::sqrt(p*p/(1.0 + b2) + (1.0 - b2));
  // result *= 1.0 / (1.0 - b2);

  return std::sqrt(1.0 + p*p + b2);
}


namespace Kernels {

__global__
void lorentz_push(Scalar* p, const Pos_t* x, const uint32_t* cell, const uint32_t* flag,
                  const Scalar* E, double dt, uint32_t num) {
  for (uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
       i < num;
       i += blockDim.x * gridDim.x) {
    if (!check_bit(flag[i], ParticleFlag::ignore_EM)) {
      auto c = cell[i];
      auto rel_x = x[i];
      auto p1 = p[i];
      int sp = get_ptc_type(flag[i]);
      Scalar E1 = E[c] * rel_x + E[c - 1] * (1.0 - rel_x);

      p1 += dev_charges[sp] * E1 * dt / dev_masses[sp];
      p[i] = p1;
    }
  }
}

__global__
void move_ptc(Pos_t* x, uint32_t* cell, const Scalar* p, double dt, uint32_t num) {
  
}

}


ParticlePusher_BeadOnWire::ParticlePusher_BeadOnWire(const Environment& env) :
  m_params(env.params()) {}

ParticlePusher_BeadOnWire::~ParticlePusher_BeadOnWire() {}

void
ParticlePusher_BeadOnWire::push(SimData& data, double dt) {
  Logger::print_info("In particle pusher");
  auto& grid = data.E.grid();
  auto& mesh = grid.mesh();
}

void
ParticlePusher_BeadOnWire::move_ptc(Particles& particles, double x,
                                    const Grid& grid, double dt) {
  auto& ptc = particles.data();
  auto& mesh = grid.mesh();
  if (mesh.dim() == 1) {
  }
}

void
ParticlePusher_BeadOnWire::lorentz_push(Particles& particles, double x,
                                      const VectorField<Scalar>& E,
                                      const VectorField<Scalar>& B, double dt) {
  auto& ptc = particles.data();
  if (E.grid().dim() == 1) {
    Kernels::lorentz_push<<<512, 512>>>(ptc.p1, ptc.x1, ptc.cell, ptc.flag,
                                        E.ptr(0), dt, particles.number());
    CudaCheckError();
  }
}

void
ParticlePusher_BeadOnWire::handle_boundary(SimData &data) {
  auto& mesh = data.E.grid().mesh();
  auto& ptc = data.particles;
  if (ptc.number() > 0) {
    if (m_params.periodic_boundary[0] == false) {
      ptc.clear_guard_cells();
    }
  }
  // auto& photon = data.photons;
  // if (photon.number() > 0) {
  //   if (m_params.periodic_boundary[0] == false) {
  //     photon.clear_guard_cells();
  //   }
  // }

}

void
ParticlePusher_BeadOnWire::extra_force(Particles &particles, Index_t idx, double x, const Grid &grid, double dt) {
  auto& ptc = particles.data();
}

}  // namespace Aperture
