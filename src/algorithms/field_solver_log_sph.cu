#include "hip/hip_runtime.h"
#include "algorithms/field_solver_helper.cuh"
#include "algorithms/field_solver_log_sph.h"
#include "algorithms/finite_diff.h"
#include "cuda/constant_mem.h"
#include "cuda/cudaUtility.h"
#include "data/detail/multi_array_utils.hpp"
#include "data/field_data.h"
#include "data/fields_utils.h"
#include "utils/timer.h"

namespace Aperture {

namespace Kernels {

template <int DIM1, int DIM2>
__global__ void
compute_e_update(hipPitchedPtr e1, hipPitchedPtr e2,
                 hipPitchedPtr e3, hipPitchedPtr b1,
                 hipPitchedPtr b2, hipPitchedPtr b3,
                 hipPitchedPtr j1, hipPitchedPtr j2,
                 hipPitchedPtr j3, Grid_LogSph::mesh_ptrs mesh_ptrs,
                 Scalar dt) {
  // Load position parameters
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * DIM1 + c1;
  int n2 = dev_mesh.guard[1] + t2 * DIM2 + c2;
  size_t globalOffset = n2 * e1.pitch + n1 * sizeof(Scalar);

  // Do the actual computation here
  // (Curl u)_1 = d2u3 - d3u2
  (*ptrAddr(e1, globalOffset)) +=
      // -dt * *ptrAddr(j1, globalOffset);
      dt * ((*ptrAddr(b3, globalOffset + b3.pitch) *
                 *ptrAddr(mesh_ptrs.l3_b, globalOffset + b3.pitch) -
             *ptrAddr(b3, globalOffset) *
                 *ptrAddr(mesh_ptrs.l3_b, globalOffset)) /
                *ptrAddr(mesh_ptrs.A1_e, globalOffset) -
            *ptrAddr(j1, globalOffset));

  // (Curl u)_2 = d3u1 - d1u3
  (*ptrAddr(e2, globalOffset)) +=
      // -dt * *ptrAddr(j2, globalOffset);
      dt *
      ((*ptrAddr(b3, globalOffset) *
            *ptrAddr(mesh_ptrs.l3_b, globalOffset) -
        *ptrAddr(b3, globalOffset + sizeof(Scalar)) *
            *ptrAddr(mesh_ptrs.l3_b, globalOffset + sizeof(Scalar))) /
           *ptrAddr(mesh_ptrs.A2_e, globalOffset) -
       *ptrAddr(j2, globalOffset));

  // (Curl u)_3 = d1u2 - d2u1
  (*ptrAddr(e3, globalOffset)) +=
      // -dt * *ptrAddr(j3, globalOffset);
      dt *
      ((*ptrAddr(b2, globalOffset + sizeof(Scalar)) *
            *ptrAddr(mesh_ptrs.l2_b, globalOffset + sizeof(Scalar)) -
        *ptrAddr(b2, globalOffset) *
            *ptrAddr(mesh_ptrs.l2_b, globalOffset) +
        *ptrAddr(b1, globalOffset) *
            *ptrAddr(mesh_ptrs.l1_b, globalOffset) -
        *ptrAddr(b1, globalOffset + b1.pitch) *
            *ptrAddr(mesh_ptrs.l1_b, globalOffset + b1.pitch)) /
           *ptrAddr(mesh_ptrs.A3_e, globalOffset) -
       *ptrAddr(j3, globalOffset));

  // Extra work for the axis
  if (threadIdx.y == 0 && blockIdx.y == 0) {
    n2 = dev_mesh.guard[1] - 1;
    globalOffset = n2 * e1.pitch + n1 * sizeof(Scalar);

    // (*ptrAddr(e2, globalOffset)) = 0.0f;
    (*ptrAddr(e3, globalOffset)) = 0.0f;

    (*ptrAddr(e1, globalOffset)) +=
        dt * ((*ptrAddr(b3, globalOffset + b3.pitch) *
                   *ptrAddr(mesh_ptrs.l3_b, globalOffset + b3.pitch) -
               *ptrAddr(b3, globalOffset) *
                   *ptrAddr(mesh_ptrs.l3_b, globalOffset)) /
                  *ptrAddr(mesh_ptrs.A1_e, globalOffset) -
              *ptrAddr(j1, globalOffset));
    // if (n1 == 4) {
    //   printf("E1 is %f, %f\n", *ptrAddr(e1, globalOffset), *ptrAddr(e1, globalOffset + b3.pitch));
    //   printf("E2 is %f, %f\n", *ptrAddr(e2, globalOffset), *ptrAddr(e2, globalOffset + b3.pitch));
    //   printf("B3 is %f, %f\n", *ptrAddr(b3, globalOffset), *ptrAddr(b3, globalOffset + b3.pitch));
    // }
  }
}

template <int DIM1, int DIM2>
__global__ void
compute_b_update(hipPitchedPtr e1, hipPitchedPtr e2,
                 hipPitchedPtr e3, hipPitchedPtr b1,
                 hipPitchedPtr b2, hipPitchedPtr b3,
                 Grid_LogSph::mesh_ptrs mesh_ptrs, Scalar dt) {
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * DIM1 + c1;
  int n2 = dev_mesh.guard[1] + t2 * DIM2 + c2;
  size_t globalOffset = n2 * e1.pitch + n1 * sizeof(Scalar);

  // Do the actual computation here
  // (Curl u)_1 = d2u3 - d3u2
  (*ptrAddr(b1, globalOffset)) +=
      -dt *
      (*ptrAddr(e3, globalOffset) *
           *ptrAddr(mesh_ptrs.l3_e, globalOffset) -
       *ptrAddr(e3, globalOffset - e3.pitch) *
           *ptrAddr(mesh_ptrs.l3_e, globalOffset - e3.pitch)) /
      *ptrAddr(mesh_ptrs.A1_b, globalOffset);

  // (Curl u)_2 = d3u1 - d1u3
  (*ptrAddr(b2, globalOffset)) +=
      -dt *
      (*ptrAddr(e3, globalOffset - sizeof(Scalar)) *
           *ptrAddr(mesh_ptrs.l3_e, globalOffset - sizeof(Scalar)) -
       *ptrAddr(e3, globalOffset) *
           *ptrAddr(mesh_ptrs.l3_e, globalOffset)) /
      *ptrAddr(mesh_ptrs.A2_b, globalOffset);

  // (Curl u)_3 = d1u2 - d2u1
  (*ptrAddr(b3, globalOffset)) +=
      -dt *
      ((*ptrAddr(e2, globalOffset) *
            *ptrAddr(mesh_ptrs.l2_e, globalOffset) -
        *ptrAddr(e2, globalOffset - sizeof(Scalar)) *
            *ptrAddr(mesh_ptrs.l2_e, globalOffset - sizeof(Scalar)) +
        *ptrAddr(e1, globalOffset - e1.pitch) *
            *ptrAddr(mesh_ptrs.l1_e, globalOffset - e1.pitch) -
        *ptrAddr(e1, globalOffset) *
            *ptrAddr(mesh_ptrs.l1_e, globalOffset)) /
       *ptrAddr(mesh_ptrs.A3_b, globalOffset));

  // Extra work for the axis at theta = 0
  if (threadIdx.y == 0 && blockIdx.y == 0) {
    n2 = dev_mesh.guard[1] - 1;
    globalOffset = n2 * b1.pitch + n1 * sizeof(Scalar);

    // (*ptrAddr(b1, globalOffset)) +=
    //     -dt *
    //     (*ptrAddr(e3, globalOffset + e1.pitch) *
    //          *ptrAddr(mesh_ptrs.l3_e, globalOffset + e1.pitch) -
    //      *ptrAddr(e3, globalOffset) *
    //          *ptrAddr(mesh_ptrs.l3_e, globalOffset)) /
    //     *ptrAddr(mesh_ptrs.A1_b, globalOffset);
    // (*ptrAddr(b1, globalOffset)) +=
    //     -dt * *ptrAddr(e3, globalOffset + e3.pitch) *
    //     *ptrAddr(mesh_ptrs.l3_e, globalOffset + e3.pitch) /
    //     *ptrAddr(mesh_ptrs.A1_b, globalOffset);

    (*ptrAddr(b2, globalOffset)) = 0.0f;
  }
}

template <int DIM1, int DIM2>
__global__ void
compute_divs(hipPitchedPtr e1, hipPitchedPtr e2, hipPitchedPtr e3,
             hipPitchedPtr b1, hipPitchedPtr b2, hipPitchedPtr b3,
             hipPitchedPtr divE, hipPitchedPtr divB,
             Grid_LogSph::mesh_ptrs mesh_ptrs) {
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * DIM1 + c1;
  int n2 = dev_mesh.guard[1] + t2 * DIM2 + c2;
  size_t globalOffset = n2 * divE.pitch + n1 * sizeof(Scalar);

  if (n1 > dev_mesh.guard[0]) {
    (*ptrAddr(divE, globalOffset)) =
        (*ptrAddr(e1, globalOffset + sizeof(Scalar)) *
             *ptrAddr(mesh_ptrs.A1_e, globalOffset + sizeof(Scalar)) -
         *ptrAddr(e1, globalOffset) *
             *ptrAddr(mesh_ptrs.A1_e, globalOffset) +
         *ptrAddr(e2, globalOffset + e2.pitch) *
             *ptrAddr(mesh_ptrs.A2_e, globalOffset + e2.pitch) -
         *ptrAddr(e2, globalOffset) *
             *ptrAddr(mesh_ptrs.A2_e, globalOffset)) /
        (*ptrAddr(mesh_ptrs.dV, globalOffset) * dev_mesh.delta[0] *
         dev_mesh.delta[1]);
  }
  (*ptrAddr(divB, globalOffset)) =
      (*ptrAddr(b1, globalOffset) *
           *ptrAddr(mesh_ptrs.A1_b, globalOffset) -
       *ptrAddr(b1, globalOffset - sizeof(Scalar)) *
           *ptrAddr(mesh_ptrs.A1_b, globalOffset - sizeof(Scalar)) +
       *ptrAddr(b2, globalOffset) *
           *ptrAddr(mesh_ptrs.A2_b, globalOffset) -
       *ptrAddr(b2, globalOffset - b2.pitch) *
           *ptrAddr(mesh_ptrs.A2_b, globalOffset - b2.pitch)) /
      (*ptrAddr(mesh_ptrs.dV, globalOffset) * dev_mesh.delta[0] *
       dev_mesh.delta[1]);

  // if (threadIdx.y == 0 && blockIdx.y == 0) {
  //   n2 = dev_mesh.guard[1] - 1;
  //   globalOffset = n2 * e1.pitch + n1 * sizeof(Scalar);

  //   (*ptrAddr(divE, globalOffset)) =
  //       (*ptrAddr(e1, globalOffset + sizeof(Scalar)) *
  //            *ptrAddr(mesh_ptrs.A1_e, globalOffset + sizeof(Scalar)) -
  //        *ptrAddr(e1, globalOffset) *
  //        *ptrAddr(mesh_ptrs.A1_e, globalOffset) +
  //        *ptrAddr(e2, globalOffset + e2.pitch) *
  //        *ptrAddr(mesh_ptrs.A2_e, globalOffset + e2.pitch)) /
  //       (*ptrAddr(mesh_ptrs.dV, globalOffset) * dev_mesh.delta[0] *
  //        dev_mesh.delta[1]);
  // }
}

template <int DIM2>
__global__ void
stellar_boundary(hipPitchedPtr e1, hipPitchedPtr e2,
                 hipPitchedPtr e3, hipPitchedPtr b1,
                 hipPitchedPtr b2, hipPitchedPtr b3, Scalar omega) {
  for (int j = blockIdx.x * blockDim.x + threadIdx.x;
       j < dev_mesh.dims[1]; j += blockDim.x * gridDim.x) {
    Scalar* row_e2 = ptrAddr(e2, j * e2.pitch);
    Scalar* row_b1 = ptrAddr(dev_bg_fields.B1, j * b1.pitch);
    Scalar* row_e1 = ptrAddr(e1, j * e1.pitch);
    Scalar* row_b2 = ptrAddr(dev_bg_fields.B2, j * b2.pitch);
    Scalar theta_s = dev_mesh.pos(1, j, true);
    Scalar theta = dev_mesh.pos(1, j, false);
    for (int i = 0; i <= dev_mesh.guard[0]; i++) {
      Scalar r_s = std::exp(dev_mesh.pos(0, i, true));
      Scalar r = std::exp(dev_mesh.pos(0, i, false));
      row_e2[i] = -omega * std::sin(theta_s) * r * row_b1[i];
      // Do not impose right on the surface
      row_e1[i] = omega * std::sin(theta) * r_s * row_b2[i];
      (*ptrAddr(b1, j * b1.pitch + i * sizeof(Scalar))) = 0.0f;
      (*ptrAddr(b2, j * b2.pitch + i * sizeof(Scalar))) = 0.0f;
      (*ptrAddr(b3, j * b3.pitch + i * sizeof(Scalar))) = 0.0f;
      (*ptrAddr(e3, j * e3.pitch + i * sizeof(Scalar))) = 0.0f;
    }
  }
}

template <int DIM1>
__global__ void
axis_boundary(hipPitchedPtr e1, hipPitchedPtr e2, hipPitchedPtr e3,
              hipPitchedPtr b1, hipPitchedPtr b2, hipPitchedPtr b3) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
    (*ptrAddr(e3, (dev_mesh.guard[1] - 1) * e3.pitch +
                      i * sizeof(Scalar))) = 0.0f;
    // (*ptrAddr(b3, (dev_mesh.guard[1] - 1) * b3.pitch +
    //                   i * sizeof(Scalar))) = 0.0f;
    // (*ptrAddr(b1, (dev_mesh.guard[1] - 1) * b1.pitch +
    //                   i * sizeof(Scalar))) = 0.0f;
    (*ptrAddr(
        b3, (dev_mesh.guard[1] - 1) * b3.pitch + i * sizeof(Scalar))) =
        *ptrAddr(b3, dev_mesh.guard[1] * b3.pitch + i * sizeof(Scalar));
    // (*ptrAddr(e3, dev_mesh.guard[1] * e3.pitch + i * sizeof(Scalar)))
    // =
    //     0.0f;

    (*ptrAddr(e3,
              (dev_mesh.dims[1] - dev_mesh.guard[1] - 1) * e3.pitch +
                  i * sizeof(Scalar))) = 0.0f;
    // (*ptrAddr(b3,
    //           (dev_mesh.dims[1] - dev_mesh.guard[1] - 1) * b3.pitch +
    //               i * sizeof(Scalar))) = 0.0f;
    // (*ptrAddr(b1,
    //           (dev_mesh.dims[1] - dev_mesh.guard[1] - 1) * b1.pitch +
    //               i * sizeof(Scalar))) = 0.0f;
    (*ptrAddr(b3, (dev_mesh.dims[1] - dev_mesh.guard[1]) * b3.pitch +
                      i * sizeof(Scalar))) =
        *ptrAddr(b3,
                 (dev_mesh.dims[1] - dev_mesh.guard[1] - 1) * b3.pitch +
                     i * sizeof(Scalar));
  }
}

template <int DIM2>
__global__ void
outflow_boundary(hipPitchedPtr e1, hipPitchedPtr e2,
                 hipPitchedPtr e3, hipPitchedPtr b1,
                 hipPitchedPtr b2, hipPitchedPtr b3) {
  for (int j = blockIdx.x * blockDim.x + threadIdx.x;
       j < dev_mesh.dims[1]; j += blockDim.x * gridDim.x) {
    for (int i = 0; i < dev_params.damping_length; i++) {
      int n1 = dev_mesh.dims[0] - dev_params.damping_length + i;
      size_t offset = j * e1.pitch + n1 * sizeof(Scalar);
      Scalar lambda =
          1.0f - dev_params.damping_coef *
                     square((Scalar)i / dev_params.damping_length);
      (*ptrAddr(e1, offset)) *= lambda;
      (*ptrAddr(e2, offset)) *= lambda;
      (*ptrAddr(e3, offset)) *= lambda;
      // (*ptrAddr(b1, offset)) *= lambda;
      (*ptrAddr(b2, offset)) *= lambda;
      (*ptrAddr(b3, offset)) *= lambda;
    }
  }
}

}  // namespace Kernels

FieldSolver_LogSph::FieldSolver_LogSph(const Grid_LogSph& g)
    : m_grid(g), m_divE(g), m_divB(g) {
  m_divB.set_stagger(0b000);
}

FieldSolver_LogSph::~FieldSolver_LogSph() {}

void
FieldSolver_LogSph::update_fields(SimData& data, double dt,
                                  double time) {
  update_fields(data.E, data.B, data.J, dt, time);
}

void
FieldSolver_LogSph::update_fields(vfield_t& E, vfield_t& B,
                                  const vfield_t& J, double dt,
                                  double time) {
  Logger::print_info("Updating fields");
  auto mesh_ptrs = m_grid.get_mesh_ptrs();
  auto& mesh = m_grid.mesh();

  if (m_grid.dim() == 2) {
    // We only implemented 2d at the moment
    dim3 blockSize(32, 16);
    dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 16);
    // Update B
    Kernels::compute_b_update<32, 16><<<gridSize, blockSize>>>(
        E.ptr(0), E.ptr(1), E.ptr(2), B.ptr(0), B.ptr(1), B.ptr(2),
        mesh_ptrs, dt);
    CudaCheckError();
    hipDeviceSynchronize();

    // Update E
    Kernels::compute_e_update<32, 16><<<gridSize, blockSize>>>(
        E.ptr(0), E.ptr(1), E.ptr(2), B.ptr(0), B.ptr(1), B.ptr(2),
        J.ptr(0), J.ptr(1), J.ptr(2), mesh_ptrs, dt);
    CudaCheckError();
    // hipDeviceSynchronize();

    if (m_comm_callback_vfield != nullptr) {
      m_comm_callback_vfield(E);
      m_comm_callback_vfield(B);
    }

    // Compute divE
    Kernels::compute_divs<32, 16><<<gridSize, blockSize>>>(
        E.ptr(0), E.ptr(1), E.ptr(2), B.ptr(0), B.ptr(1), B.ptr(2),
        m_divE.ptr(), m_divB.ptr(), mesh_ptrs);
    CudaCheckError();
  }
}

void
FieldSolver_LogSph::set_background_j(const vfield_t& J) {}

void
FieldSolver_LogSph::boundary_conditions(SimData& data, double omega) {
  // Logger::print_info("omega is {}", omega);
  Kernels::stellar_boundary<256>
      <<<32, 256>>>(data.E.ptr(0), data.E.ptr(1), data.E.ptr(2),
                    data.B.ptr(0), data.B.ptr(1), data.B.ptr(2), omega);
  CudaCheckError();

  Kernels::axis_boundary<256>
      <<<32, 256>>>(data.E.ptr(0), data.E.ptr(1), data.E.ptr(2),
                    data.B.ptr(0), data.B.ptr(1), data.B.ptr(2));
  CudaCheckError();

  Kernels::outflow_boundary<256>
      <<<32, 256>>>(data.E.ptr(0), data.E.ptr(1), data.E.ptr(2),
                    data.B.ptr(0), data.B.ptr(1), data.B.ptr(2));
  CudaCheckError();
}

}  // namespace Aperture