#include "hip/hip_runtime.h"
#include "algorithms/field_solver_force_free.h"
#include "algorithms/finite_diff.h"
#include "algorithms/field_solver_helper.cuh"
#include "data/fields_utils.h"
#include "cuda/cudaUtility.h"
#include "cuda/constant_mem.h"
#include "utils/timer.h"

namespace Aperture {

namespace Kernels {

template <int DIM1, int DIM2, int DIM3>
__global__
void compute_FFE_EdotB(hipPitchedPtr eb,
                       hipPitchedPtr e1, hipPitchedPtr e2, hipPitchedPtr e3,
                       hipPitchedPtr b1, hipPitchedPtr b2, hipPitchedPtr b3, Scalar q) {
  // Declare cache array in shared memory
  __shared__ Scalar s_e1[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_e2[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_e3[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b1[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b2[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b3[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];

  // Load shared memory
  int t1 = blockIdx.x, t2 = blockIdx.y, t3 = blockIdx.z;
  int c1 = threadIdx.x + Pad<2>::val,
      c2 = threadIdx.y + Pad<2>::val,
      c3 = threadIdx.z + Pad<2>::val;
  size_t globalOffset =  (dev_mesh.guard[2] + t3 * DIM3 + c3 - Pad<2>::val) * e1.pitch * e1.ysize +
                         (dev_mesh.guard[1] + t2 * DIM2 + c2 - Pad<2>::val) * e1.pitch +
                         (dev_mesh.guard[0] + t1 * DIM1 + c1 - Pad<2>::val) * sizeof(Scalar);

  init_shared_memory<2, DIM1, DIM2, DIM3>(s_e1, s_e2, s_e3, e1, e2, e3,
                                          globalOffset, c1, c2, c3);
  init_shared_memory<2, DIM1, DIM2, DIM3>(s_b1, s_b2, s_b3, b1, b2, b3,
                                          globalOffset, c1, c2, c3);
  __syncthreads();

  Scalar vecE1 = 0.5f * (s_e1[c3][c2][c1] + s_e1[c3][c2][c1 - 1]);
  Scalar vecE2 = 0.5f * (s_e2[c3][c2][c1] + s_e2[c3][c2 - 1][c1]);
  Scalar vecE3 = 0.5f * (s_e3[c3][c2][c1] + s_e3[c3 - 1][c2][c1]);
  Scalar vecB1 = 0.25f * (s_b1[c3][c2][c1] + s_b1[c3 - 1][c2][c1] +
                          s_b1[c3][c2 - 1][c1] + s_b1[c3 - 1][c2 - 1][c1]);
  Scalar vecB2 = 0.25f * (s_b2[c3][c2][c1] + s_b2[c3 - 1][c2][c1] +
                          s_b2[c3][c2][c1 - 1] + s_b2[c3 - 1][c2][c1 - 1]);
  Scalar vecB3 = 0.25f * (s_b3[c3][c2][c1] + s_b3[c3][c2][c1 - 1] +
                          s_b3[c3][c2 - 1][c1] + s_b3[c3][c2 - 1][c1 - 1]);
  Scalar EdotB = vecE1 * vecB1 + vecE2 * vecB2 + vecE3 * vecB3;

  // Do the actual computation here
  (*(Scalar*)((char*)eb.ptr + globalOffset)) += q * EdotB;
}

template <int DIM1, int DIM2, int DIM3>
__global__
void compute_FFE_J(hipPitchedPtr j1, hipPitchedPtr j2, hipPitchedPtr j3,
                   hipPitchedPtr e1, hipPitchedPtr e2, hipPitchedPtr e3,
                   hipPitchedPtr b1, hipPitchedPtr b2, hipPitchedPtr b3,
                   hipPitchedPtr f, Scalar q) {
  // Declare cache array in shared memory
  __shared__ Scalar s_e1[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_e2[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_e3[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b1[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b2[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b3[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_f[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];

  // Load shared memory
  int t1 = blockIdx.x, t2 = blockIdx.y, t3 = blockIdx.z;
  int c1 = threadIdx.x + Pad<2>::val,
      c2 = threadIdx.y + Pad<2>::val,
      c3 = threadIdx.z + Pad<2>::val;
  size_t globalOffset =  (dev_mesh.guard[2] + t3 * DIM3 + c3 - Pad<2>::val) * e1.pitch * e1.ysize +
                         (dev_mesh.guard[1] + t2 * DIM2 + c2 - Pad<2>::val) * e1.pitch +
                         (dev_mesh.guard[0] + t1 * DIM1 + c1 - Pad<2>::val) * sizeof(Scalar);

  init_shared_memory<2, DIM1, DIM2, DIM3>(s_e1, s_e2, s_e3, e1, e2, e3,
                                          globalOffset, c1, c2, c3);
  init_shared_memory<2, DIM1, DIM2, DIM3>(s_b1, s_b2, s_b3, b1, b2, b3,
                                          globalOffset, c1, c2, c3);
  init_shared_memory<2, DIM1, DIM2, DIM3>(s_f, f, globalOffset, c1, c2, c3);
  __syncthreads();

  Scalar vecE1 = 0.5f * (s_e1[c3][c2][c1] + s_e1[c3][c2][c1 - 1]);
  Scalar vecE2 = 0.5f * (s_e2[c3][c2][c1] + s_e2[c3][c2 - 1][c1]);
  Scalar vecE3 = 0.5f * (s_e3[c3][c2][c1] + s_e3[c3 - 1][c2][c1]);
  Scalar vecB1 = 0.25f * (s_b1[c3][c2][c1] + s_b1[c3 - 1][c2][c1] +
                          s_b1[c3][c2 - 1][c1] + s_b1[c3 - 1][c2 - 1][c1]);
  Scalar vecB2 = 0.25f * (s_b2[c3][c2][c1] + s_b2[c3 - 1][c2][c1] +
                          s_b2[c3][c2][c1 - 1] + s_b2[c3 - 1][c2][c1 - 1]);
  Scalar vecB3 = 0.25f * (s_b3[c3][c2][c1] + s_b3[c3][c2][c1 - 1] +
                          s_b3[c3][c2 - 1][c1] + s_b3[c3][c2 - 1][c1 - 1]);
  Scalar inv_B_sqr = 1.0f / (vecB1 * vecB1 + vecB2 * vecB2 + vecB3 * vecB3);
  Scalar divE = (s_e1[c3][c2][c1] - s_e1[c3][c2][c1 - 1]) / dev_mesh.delta[0] +
                (s_e2[c3][c2][c1] - s_e2[c3][c2 - 1][c1]) / dev_mesh.delta[1] +
                (s_e3[c3][c2][c1] - s_e3[c3 - 1][c2][c1]) / dev_mesh.delta[2];
  Scalar EcrossB1 = vecE2 * vecB3 - vecE3 * vecB2;
  Scalar EcrossB2 = vecE3 * vecB1 - vecE1 * vecB3;
  Scalar EcrossB3 = vecE1 * vecB2 - vecE2 * vecB1;
  // Scalar EdotB = vecE1 * vecB1 + vecE2 * vecB2 + vecE3 * vecB3;

  // Do the actual computation here
  (*(Scalar*)((char*)j1.ptr + globalOffset)) = q * (s_f[c3][c2][c1] * vecB1 + divE * EcrossB1) * inv_B_sqr;
  (*(Scalar*)((char*)j2.ptr + globalOffset)) = q * (s_f[c3][c2][c1] * vecB2 + divE * EcrossB2) * inv_B_sqr;
  (*(Scalar*)((char*)j3.ptr + globalOffset)) = q * (s_f[c3][c2][c1] * vecB3 + divE * EcrossB3) * inv_B_sqr;
}


template <int DIM1, int DIM2, int DIM3>
__global__
void compute_FFE_dE(hipPitchedPtr e1out, hipPitchedPtr e2out, hipPitchedPtr e3out,
                    hipPitchedPtr j1, hipPitchedPtr j2, hipPitchedPtr j3,
                    hipPitchedPtr e1, hipPitchedPtr e2, hipPitchedPtr e3,
                    hipPitchedPtr b1, hipPitchedPtr b2, hipPitchedPtr b3,
                    Scalar dt) {
  // Declare cache array in shared memory
  __shared__ Scalar s_e1[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_e2[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_e3[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b1[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b2[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];
  __shared__ Scalar s_b3[DIM3 + 2*Pad<2>::val]
      [DIM2 + 2*Pad<2>::val][DIM1 + 2*Pad<2>::val];

  // Load shared memory
  int c1 = threadIdx.x + Pad<2>::val,
      c2 = threadIdx.y + Pad<2>::val,
      c3 = threadIdx.z + Pad<2>::val;
  size_t globalOffset =  (dev_mesh.guard[2] + blockIdx.z * DIM3 + c3 - Pad<2>::val) * e1.pitch * e1.ysize +
                         (dev_mesh.guard[1] + blockIdx.y * DIM2 + c2 - Pad<2>::val) * e1.pitch +
                         (dev_mesh.guard[0] + blockIdx.x * DIM1 + c1 - Pad<2>::val) * sizeof(Scalar);
  init_shared_memory<2, DIM1, DIM2, DIM3>(s_e1, s_e2, s_e3, e1, e2, e3,
                                          globalOffset, c1, c2, c3);
  init_shared_memory<2, DIM1, DIM2, DIM3>(s_b1, s_b2, s_b3, b1, b2, b3,
                                          globalOffset, c1, c2, c3);
  __syncthreads();
  Scalar vecE1 = 0.5f * (s_e1[c3][c2][c1] + s_e1[c3][c2][c1 - 1]);
  Scalar vecE2 = 0.5f * (s_e2[c3][c2][c1] + s_e2[c3][c2 - 1][c1]);
  Scalar vecE3 = 0.5f * (s_e3[c3][c2][c1] + s_e3[c3 - 1][c2][c1]);
  Scalar vecB1 = 0.25f * (s_b1[c3][c2][c1] + s_b1[c3 - 1][c2][c1] +
                          s_b1[c3][c2 - 1][c1] + s_b1[c3 - 1][c2 - 1][c1]);
  Scalar vecB2 = 0.25f * (s_b2[c3][c2][c1] + s_b2[c3 - 1][c2][c1] +
                          s_b2[c3][c2][c1 - 1] + s_b2[c3 - 1][c2][c1 - 1]);
  Scalar vecB3 = 0.25f * (s_b3[c3][c2][c1] + s_b3[c3][c2][c1 - 1] +
                          s_b3[c3][c2 - 1][c1] + s_b3[c3][c2 - 1][c1 - 1]);
  Scalar EcrossB1 = vecE2 * vecB3 - vecE3 * vecB2;
  Scalar EcrossB2 = vecE3 * vecB1 - vecE1 * vecB3;
  Scalar EcrossB3 = vecE1 * vecB2 - vecE2 * vecB1;
  Scalar divE = (s_e1[c3][c2][c1] - s_e1[c3][c2][c1 - 1]) / dev_mesh.delta[0] +
                (s_e2[c3][c2][c1] - s_e2[c3][c2 - 1][c1]) / dev_mesh.delta[1] +
                (s_e3[c3][c2][c1] - s_e3[c3 - 1][c2][c1]) / dev_mesh.delta[2];
  Scalar inv_B_sqr = 1.0f / (vecB1 * vecB1 + vecB2 * vecB2 + vecB3 * vecB3);

  // Compute the second part of the current
  (*(Scalar*)((char*)j1.ptr + globalOffset)) = divE * EcrossB1 * inv_B_sqr;
  (*(Scalar*)((char*)j2.ptr + globalOffset)) = divE * EcrossB2 * inv_B_sqr;
  (*(Scalar*)((char*)j3.ptr + globalOffset)) = divE * EcrossB3 * inv_B_sqr;

  // Reuse EcrossB1, 2, 3 to compute B\dot(curl B)
  EcrossB1 = vecB1 * 0.5f * ((s_b3[c3][c2][c1] - s_b3[c3][c2 - 1][c1] +
                              s_b3[c3][c2][c1 - 1] - s_b3[c3][c2 - 1][c1 - 1]) * dev_mesh.inv_delta[1] -
                             (s_b2[c3][c2][c1] - s_b2[c3 - 1][c2][c1] +
                              s_b2[c3][c2][c1 - 1] - s_b2[c3 - 1][c2][c1 - 1]) * dev_mesh.inv_delta[2]);
  EcrossB2 = vecB2 * 0.5f * ((s_b1[c3][c2][c1] - s_b1[c3 - 1][c2][c1] +
                              s_b1[c3][c2 - 1][c1] - s_b1[c3 - 1][c2 - 1][c1]) * dev_mesh.inv_delta[2] -
                             (s_b3[c3][c2][c1] - s_b3[c3][c2][c1 - 1] +
                              s_b3[c3][c2 - 1][c1] - s_b3[c3][c2 - 1][c1 - 1]) * dev_mesh.inv_delta[0]);
  EcrossB3 = vecB3 * 0.5f * ((s_b2[c3][c2][c1] - s_b2[c3][c2][c1 - 1] +
                              s_b2[c3 - 1][c2][c1] - s_b2[c3 - 1][c2][c1 - 1]) * dev_mesh.inv_delta[0] -
                             (s_b1[c3][c2][c1] - s_b1[c3][c2 - 1][c1] +
                              s_b1[c3 - 1][c2][c1] - s_b1[c3 - 1][c2 - 1][c1]) * dev_mesh.inv_delta[1]);
  // Now use EcrossB1, 2, 3 to compute E\dot(curl E)
  EcrossB1 -= vecE1 * 0.25f * ((s_e3[c3][c2 + 1][c1] - s_e3[c3][c2 - 1][c1] +
                                s_e3[c3 - 1][c2 + 1][c1] - s_e3[c3 - 1][c2 - 1][c1]) * dev_mesh.inv_delta[1] -
                               (s_e2[c3 + 1][c2][c1] - s_e2[c3 - 1][c2][c1] +
                                s_e2[c3 + 1][c2 - 1][c1] - s_e2[c3 - 1][c2 - 1][c1]) * dev_mesh.inv_delta[2]);
  EcrossB2 -= vecE2 * 0.25f * ((s_e1[c3 + 1][c2][c1] - s_e1[c3 - 1][c2][c1] +
                                s_e1[c3 + 1][c2][c1 - 1] - s_e1[c3 - 1][c2][c1 - 1]) * dev_mesh.inv_delta[2] -
                               (s_e3[c3][c2][c1 + 1] - s_e3[c3][c2][c1 - 1] +
                                s_e3[c3 - 1][c2][c1 + 1] - s_e3[c3 - 1][c2][c1 - 1]) * dev_mesh.inv_delta[0]);
  EcrossB3 -= vecE3 * 0.25f * ((s_e2[c3][c2][c1 + 1] - s_e2[c3][c2][c1 - 1] +
                                s_e2[c3][c2 - 1][c1 + 1] - s_e2[c3][c2 - 1][c1 - 1]) * dev_mesh.inv_delta[0] -
                               (s_e1[c3][c2 + 1][c1] - s_e1[c3][c2 - 1][c1] +
                                s_e1[c3][c2 + 1][c1 - 1] - s_e1[c3][c2 - 1][c1 - 1]) * dev_mesh.inv_delta[1]);
  EcrossB1 = EcrossB1 + EcrossB2 + EcrossB3;

  // Compute the first term of the FFE current
  (*(Scalar*)((char*)j1.ptr + globalOffset)) += EcrossB1 * vecB1 * inv_B_sqr;
  (*(Scalar*)((char*)j2.ptr + globalOffset)) += EcrossB1 * vecB2 * inv_B_sqr;
  (*(Scalar*)((char*)j3.ptr + globalOffset)) += EcrossB1 * vecB3 * inv_B_sqr;

  // Now use EcrossB1 to compute curl B
  EcrossB1 = (s_b3[c3][c2][c1] - s_b3[c3][c2 - 1][c1]) * dev_mesh.inv_delta[1] -
             (s_b2[c3][c2][c1] - s_b2[c3 - 1][c2][c1]) * dev_mesh.inv_delta[2];
  EcrossB2 = (s_b1[c3][c2][c1] - s_b1[c3 - 1][c2][c1]) * dev_mesh.inv_delta[2] -
             (s_b3[c3][c2][c1] - s_b3[c3][c2][c1 - 1]) * dev_mesh.inv_delta[0];
  EcrossB3 = (s_b2[c3][c2][c1] - s_b2[c3][c2][c1 - 1]) * dev_mesh.inv_delta[0] -
             (s_b1[c3][c2][c1] - s_b1[c3][c2 - 1][c1]) * dev_mesh.inv_delta[1];

  // Compute the update of E, sans J
  (*(Scalar*)((char*)e1out.ptr + globalOffset)) += dt * EcrossB1;
  (*(Scalar*)((char*)e2out.ptr + globalOffset)) += dt * EcrossB2;
  (*(Scalar*)((char*)e3out.ptr + globalOffset)) += dt * EcrossB3;
}

}

FieldSolver_FFE::FieldSolver_FFE(const Grid& g) :
    m_Etmp(g), m_Btmp(g)
    // , m_tmp2(g),
    // m_e1(g), m_e2(g), m_e3(g), m_e4(g),
    // m_b1(g), m_b2(g), m_b3(g), m_b4(g)
{
  // m_j1(g), m_j2(g), m_j3(g), m_j4(g) {
  // m_b1.set_field_type(FieldType::B);
  // m_b2.set_field_type(FieldType::B);
  // m_b3.set_field_type(FieldType::B);
  // m_b4.set_field_type(FieldType::B);
}

FieldSolver_FFE::~FieldSolver_FFE() {}

void
FieldSolver_FFE::update_fields(SimData &data, double dt, double time) {
  
}

void
FieldSolver_FFE::compute_J(vfield_t &J, const vfield_t &E, const vfield_t &B) {
  
}

void
FieldSolver_FFE::update_field_substep(vfield_t &E_out, vfield_t &B_out, vfield_t &J_out,
                                      const vfield_t &E_in, const vfield_t &B_in, Scalar dt) {
  // Initialize all tmp fields to zero on the device
  // m_tmp.initialize();
  // m_tmp2.initialize();
  m_Etmp.initialize();
  m_Etmp.set_field_type(FieldType::E);

  timer::stamp();
  // Compute the curl of E_in and add it to B_out
  curl_add(B_out, E_in, dt);
  hipDeviceSynchronize();
  timer::show_duration_since_stamp("First curl and add", "ms");

  // Compute both dE and J together, put the result of J into Etmp
  timer::stamp();
  ffe_dE(E_out, m_Etmp, E_in, B_in, dt);
  hipDeviceSynchronize();
  timer::show_duration_since_stamp("Computing FFE J", "ms");
  // interpolate J back to staggered position, multiply by dt, and add to E_out
  timer::stamp();
  m_Etmp.interpolate_from_center_add(E_out, dt);
  hipDeviceSynchronize();
  timer::show_duration_since_stamp("Interpolate and add", "ms");

  // TODO: Figure out how to best handle removal of the parallel delta_E
}

void
FieldSolver_FFE::ffe_edotb(ScalarField<Scalar>& result, const VectorField<Scalar>& E,
                           const VectorField<Scalar>& B, Scalar q) {
  auto& grid = E.grid();
  auto& mesh = grid.mesh();

  dim3 blockSize(16, 8, 8);
  dim3 gridSize(mesh.reduced_dim(0) / 16, mesh.reduced_dim(1) / 8,
                mesh.reduced_dim(2) / 8);
  Kernels::compute_FFE_EdotB<16, 8, 8><<<gridSize, blockSize>>>
      (result.ptr(), E.ptr(0), E.ptr(1), E.ptr(2),
       B.ptr(0), B.ptr(1), B.ptr(2), q);
  CudaCheckError();
}

void
FieldSolver_FFE::ffe_j(VectorField<Scalar>& result, const ScalarField<Scalar>& tmp_f,
                       const VectorField<Scalar>& E, const VectorField<Scalar>& B,
                       Scalar q) {
  auto& grid = E.grid();
  auto& mesh = grid.mesh();

  dim3 blockSize(16, 8, 4);
  dim3 gridSize(mesh.reduced_dim(0) / 16, mesh.reduced_dim(1) / 8,
                mesh.reduced_dim(2) / 4);

  Kernels::compute_FFE_J<16, 8, 4><<<gridSize, blockSize>>>
      (result.ptr(0), result.ptr(1), result.ptr(2),
       E.ptr(0), E.ptr(1), E.ptr(2),
       B.ptr(0), B.ptr(1), B.ptr(2),
       tmp_f.ptr(), q);
  CudaCheckError();
}

void
FieldSolver_FFE::ffe_dE(VectorField<Scalar>& Eout, VectorField<Scalar>& J,
                        const VectorField<Scalar>& E, const VectorField<Scalar>& B,
                        Scalar dt) {
  auto& grid = E.grid();
  auto& mesh = grid.mesh();

  dim3 blockSize(32, 4, 4);
  dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 4,
                mesh.reduced_dim(2) / 4);

  Kernels::compute_FFE_dE<32, 4, 4><<<gridSize, blockSize>>>
      (Eout.ptr(0), Eout.ptr(1), Eout.ptr(2),
       J.ptr(0), J.ptr(1), J.ptr(2),
       E.ptr(0), E.ptr(1), E.ptr(2),
       B.ptr(0), B.ptr(1), B.ptr(2),
       dt);
  CudaCheckError();
}


}
