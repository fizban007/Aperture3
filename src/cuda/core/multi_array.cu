#include "core/multi_array_impl.hpp"
#include "cuda/cudaUtility.h"
#include "cuda/data/multi_array_utils.cuh"
#include "cuda/utils/pitchptr.cuh"
#include "utils/logger.h"
#include <algorithm>
#include <cstring>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>

namespace Aperture {

template <typename T>
inline hipExtent
cuda_ext(const Extent& ext, const T& t) {
  return make_hipExtent(ext.x * sizeof(T), ext.y, ext.z);
}

template <typename T>
void
multi_array<T>::copy_from(const self_type& other) {
  if (m_size != other.m_size) {
    throw std::range_error(
        "Trying to copy from a multi_array of different size!");
  }
  // memcpy(m_data_h, other.m_data_h, m_size * sizeof(T));
  assert(m_extent == other.m_extent);
  hipMemcpy3DParms myParms = {};

  myParms.srcPtr = make_hipPitchedPtr(other.m_data_d, other.m_pitch,
                                       other.m_extent.width(),
                                       other.m_extent.height());
  myParms.srcPos = make_hipPos(0, 0, 0);
  myParms.dstPtr = make_hipPitchedPtr(
      m_data_d, m_pitch, m_extent.width(), m_extent.height());
  myParms.dstPos = make_hipPos(0, 0, 0);
  myParms.extent = cuda_ext(m_extent, T{});
  myParms.kind = hipMemcpyDeviceToDevice;

  CudaSafeCall(hipMemcpy3D(&myParms));
  // sync_to_host();
}

template <typename T>
void
multi_array<T>::alloc_mem(const Extent& ext) {
  if (m_data_h != nullptr || m_data_d != nullptr) free_mem();
  auto size = ext.size();
  m_data_h = new T[size];

  auto extent = cuda_ext(ext, T{});
  hipPitchedPtr ptr;
  CudaSafeCall(hipMalloc3D(&ptr, extent));
  m_data_d = ptr.ptr;
  m_pitch = ptr.pitch;
  Logger::print_info("pitch is {}, x is {}, y is {}", m_pitch,
                     ptr.xsize, ptr.ysize);
}

template <typename T>
void
multi_array<T>::free_mem() {
  if (m_data_h != nullptr) {
    delete[] m_data_h;
    m_data_h = nullptr;
  }
  if (m_data_d != nullptr) {
    CudaSafeCall(hipFree(m_data_d));
    m_data_d = nullptr;
  }
}

template <typename T>
void
multi_array<T>::assign_dev(const T& value) {
  hipPitchedPtr p = get_cudaPitchedPtr(*this);
  if (m_extent.depth() > 1) {
    // Logger::print_info("assign_dev 3d version");
    dim3 blockSize(8, 8, 8);
    // dim3 gridSize(8, 8, 8);
    dim3 gridSize((this->m_extent.x + 7) / 8,
                  (this->m_extent.y + 7) / 8,
                  (this->m_extent.z + 7) / 8);
    Kernels::map_array_unary_op<T><<<gridSize, blockSize>>>(
        p, this->m_extent, detail::Op_AssignConst<T>(value));
    CudaCheckError();
  } else if (m_extent.height() > 1) {
    // Logger::print_info("assign_dev 2d version");
    dim3 blockSize(32, 16);
    dim3 gridSize((this->m_extent.x + 31) / 32,
                  (this->m_extent.y + 15) / 16);
    Kernels::map_array_unary_op_2d<T><<<gridSize, blockSize>>>(
        p, this->m_extent, detail::Op_AssignConst<T>(value));
    CudaCheckError();
  } else if (m_extent.width() > 1) {
    Kernels::map_array_unary_op_1d<T><<<64, 128>>>(
        p, this->m_extent, detail::Op_AssignConst<T>(value));
    CudaCheckError();
  }
}

template <typename T>
void
multi_array<T>::sync_to_host() {
  hipMemcpy3DParms myParms = {};
  myParms.srcPtr = make_hipPitchedPtr(
      m_data_d, m_pitch, m_extent.width(), m_extent.height());
  myParms.srcPos = make_hipPos(0, 0, 0);
  myParms.dstPtr =
      make_hipPitchedPtr(m_data_h, sizeof(T) * m_extent.width(),
                          m_extent.width(), m_extent.height());
  myParms.dstPos = make_hipPos(0, 0, 0);
  myParms.extent = cuda_ext(m_extent, T{});
  myParms.kind = hipMemcpyDeviceToHost;

  Logger::print_info("dev pitch {}, host pitch {}", m_pitch,
                     sizeof(T) * m_extent.width());
  CudaSafeCall(hipMemcpy3D(&myParms));
}

template <typename T>
void
multi_array<T>::sync_to_device() {
  hipMemcpy3DParms myParms = {};
  myParms.srcPtr =
      make_hipPitchedPtr(m_data_h, sizeof(T) * m_extent.width(),
                          m_extent.width(), m_extent.height());
  myParms.srcPos = make_hipPos(0, 0, 0);
  myParms.dstPtr = make_hipPitchedPtr(
      m_data_d, m_pitch, m_extent.width(), m_extent.height());
  myParms.dstPos = make_hipPos(0, 0, 0);
  myParms.extent = cuda_ext(m_extent, T{});
  myParms.kind = hipMemcpyHostToDevice;

  CudaSafeCall(hipMemcpy3D(&myParms));
}

/////////////////////////////////////////////////////////////////
// Explicitly instantiate the classes we will use
/////////////////////////////////////////////////////////////////
template class multi_array<long long>;
template class multi_array<long>;
template class multi_array<int>;
template class multi_array<short>;
template class multi_array<char>;
template class multi_array<unsigned int>;
template class multi_array<unsigned long>;
template class multi_array<unsigned long long>;
template class multi_array<float>;
template class multi_array<double>;
template class multi_array<long double>;

}  // namespace Aperture
