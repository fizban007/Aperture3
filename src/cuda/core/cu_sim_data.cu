#include "hip/hip_runtime.h"
#include "cu_sim_data.h"
#include "cuda/constant_mem.h"
#include "cuda/constant_mem_func.h"
#include "cuda/cudaUtility.h"
#include "cuda/grids/grid_1dgr_dev.h"
#include "cuda/grids/grid_log_sph_dev.h"

namespace Aperture {

namespace Kernels {

__global__ void
fill_particles(particle_data ptc, Scalar weight, int multiplicity) {
  for (int j =
           blockIdx.y * blockDim.y + threadIdx.y + dev_mesh.guard[1];
       j < dev_mesh.dims[1] - dev_mesh.guard[1];
       j += blockDim.y * gridDim.y) {
    for (int i =
             blockIdx.x * blockDim.x + threadIdx.x + dev_mesh.guard[0];
         i < dev_mesh.dims[0] - dev_mesh.guard[0];
         i += blockDim.x * gridDim.x) {
      uint32_t cell = i + j * dev_mesh.dims[0];
      Scalar theta = dev_mesh.pos(1, j, 0.5f);
      // int Np = 3;
      for (int n = 0; n < multiplicity; n++) {
        size_t idx = cell * multiplicity * 2 + n * 2;
        ptc.x1[idx] = ptc.x1[idx + 1] = 0.5f;
        ptc.x2[idx] = ptc.x2[idx + 1] = 0.5f;
        ptc.x3[idx] = ptc.x3[idx + 1] = 0.0f;
        ptc.p1[idx] = ptc.p1[idx + 1] = 0.0f;
        ptc.p2[idx] = ptc.p2[idx + 1] = 0.0f;
        ptc.p3[idx] = ptc.p3[idx + 1] = 0.0f;
        ptc.E[idx] = ptc.E[idx + 1] = 1.0f;
        ptc.cell[idx] = ptc.cell[idx + 1] = cell;
        ptc.weight[idx] = ptc.weight[idx + 1] = weight * sin(theta);
        ptc.flag[idx] = set_ptc_type_flag(0, ParticleType::electron);
        ptc.flag[idx + 1] =
            set_ptc_type_flag(0, ParticleType::positron);
      }
    }
  }
}

}  // namespace Kernels

cu_sim_data::cu_sim_data(const cu_sim_environment& e)
    : env(e), dev_map(e.dev_map()) {
  num_species = env.params().num_species;
  Rho.resize(dev_map.size());
  initialize(e);
}

cu_sim_data::~cu_sim_data() {}

void
cu_sim_data::initialize(const cu_sim_environment& env) {
  init_grid(env);
  for (int n = 0; n < dev_map.size(); n++) {
    // Loop over the devices on the node to initialize each data
    // structure
    int dev_id = dev_map[n];
    CudaSafeCall(hipSetDevice(dev_id));
    E.emplace_back(*grid[n]);
    E[n].initialize();
    B.emplace_back(*grid[n]);
    B[n].set_field_type(FieldType::B);
    B[n].initialize();
    Ebg.emplace_back(*grid[n]);
    Ebg[n].initialize();
    Bbg.emplace_back(*grid[n]);
    Bbg[n].initialize();
    J.emplace_back(*grid[n]);
    J[n].initialize();
    flux.emplace_back(*grid[n]);
    flux[n].initialize();

    for (int i = 0; i < num_species; i++) {
      Rho[n].emplace_back(*grid[n]);
      Rho[n][i].initialize();
      Rho[n][i].sync_to_host();
    }

    init_dev_bg_fields(Ebg[n], Bbg[n]);
  }

  for (int n = 0; n < dev_map.size(); n++) {
    int dev_id = dev_map[n];
    CudaSafeCall(hipSetDevice(dev_id));
    hipDeviceSynchronize();
  }
}

void
cu_sim_data::fill_multiplicity(Scalar weight, int multiplicity) {
  for (int n = 0; n < dev_map.size(); n++) {
    int dev_id = dev_map[n];
    CudaSafeCall(hipSetDevice(dev_id));
    Kernels::fill_particles<<<dim3(16, 16), dim3(32, 32)>>>(
        particles[n].data(), weight, multiplicity);
    // hipDeviceSynchronize();
    CudaCheckError();

    auto& mesh = grid[n]->mesh();
    particles[n].set_num(mesh.reduced_dim(0) * mesh.reduced_dim(1) *
                         multiplicity);
  }
}

void
cu_sim_data::init_grid(const cu_sim_environment& env) {
  grid.resize(dev_map.size());
  for (int n = 0; n < dev_map.size(); n++) {
    int dev_id = dev_map[n];
    CudaSafeCall(hipSetDevice(dev_id));

    // Setup the grids
    if (env.params().coord_system == "Cartesian") {
      grid[n].reset(new Grid());
    } else if (env.params().coord_system == "LogSpherical") {
      grid[n].reset(new Grid_LogSph_dev());
    } else if (env.params().coord_system == "1DGR" &&
               grid[n]->dim() == 1) {
      grid[n].reset(new Grid_1dGR_dev());
    } else {
      grid[n].reset(new Grid());
    }
    grid[n]->init(env.sub_params(n));
    auto& mesh = grid[n]->mesh();
    Logger::print_debug("Grid dimension for dev {} is {}x{}x{}", dev_id,
                        mesh.dims[0], mesh.dims[1], mesh.dims[2]);
    Logger::print_debug("Grid lower are {}, {}, {}", mesh.lower[0],
                        mesh.lower[1], mesh.lower[2]);
    if (grid[n]->mesh().delta[0] < env.params().delta_t) {
      std::cerr
          << "Grid spacing should be larger than delta_t! Aborting!"
          << std::endl;
      abort();
    }
    init_dev_mesh(grid[n]->mesh());
  }
}




}  // namespace Aperture
