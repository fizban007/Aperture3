#include "cuda/constant_mem.h"
#include "cuda/constant_mem_func.h"
#include "cuda/cudaUtility.h"
#include "cuda/utils/pitchptr.cuh"

namespace Aperture {

__constant__ SimParamsBase dev_params;
__constant__ Quadmesh dev_mesh;
__constant__ float dev_charges[8];
__constant__ float dev_masses[8];
__constant__ FieldData dev_bg_fields;
__device__ uint32_t dev_rank;
__device__ uint32_t dev_ptc_id = 0;

void
init_dev_params(const SimParams& params) {
  const SimParamsBase* p = &params;
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_params), (void*)p,
                                  sizeof(SimParamsBase)));
}

void
init_dev_mesh(const Quadmesh& mesh) {
  CudaSafeCall(
      hipMemcpyToSymbol(HIP_SYMBOL(dev_mesh), (void*)&mesh, sizeof(Quadmesh)));
}

void
init_dev_charges(const float charges[8]) {
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_charges), (void*)charges,
                                  sizeof(dev_charges)));
}

void
init_dev_masses(const float masses[8]) {
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_masses), (void*)masses,
                                  sizeof(dev_masses)));
}

void
init_dev_rank(int rank) {
  uint32_t r = rank;
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_rank), (void*)&r, sizeof(uint32_t)));
}

void
init_dev_bg_fields(vector_field<Scalar>& E,
                   vector_field<Scalar>& B) {
  FieldData data;
  data.E1 = get_pitchptr(E.data(0));
  data.E2 = get_pitchptr(E.data(1));
  data.E3 = get_pitchptr(E.data(2));
  data.B1 = get_pitchptr(B.data(0));
  data.B2 = get_pitchptr(B.data(1));
  data.B3 = get_pitchptr(B.data(2));
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_bg_fields), (void*)&data,
                                  sizeof(FieldData)));
}

void
get_dev_params(SimParams& params) {
  CudaSafeCall(hipMemcpyFromSymbol((void*)&params, dev_params,
                                    sizeof(SimParamsBase)));
}

void
get_dev_mesh(Quadmesh& mesh) {
  CudaSafeCall(
      hipMemcpyFromSymbol((void*)&mesh, dev_mesh, sizeof(Quadmesh)));
}

void
get_dev_charges(float charges[]) {
  CudaSafeCall(hipMemcpyFromSymbol((void*)charges, dev_charges,
                                    sizeof(dev_charges)));
}

void
get_dev_masses(float masses[]) {
  CudaSafeCall(hipMemcpyFromSymbol((void*)masses, dev_masses,
                                    sizeof(dev_masses)));
}

}  // namespace Aperture
