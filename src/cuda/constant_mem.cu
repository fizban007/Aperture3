#include "cuda/constant_mem.h"
#include "cuda/constant_mem_func.h"
#include "cuda/cudaUtility.h"

namespace Aperture {

__constant__ SimParamsBase dev_params;
__constant__ Quadmesh dev_mesh;
__constant__ float dev_charges[8];
__constant__ float dev_masses[8];
__constant__ FieldData dev_bg_fields;

void
init_dev_params(const SimParams& params) {
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_params), (void*)&params,
                                  sizeof(SimParamsBase)));
}

void
init_dev_mesh(const Quadmesh& mesh) {
  CudaSafeCall(
      hipMemcpyToSymbol(HIP_SYMBOL(dev_mesh), (void*)&mesh, sizeof(Quadmesh)));
}

void
init_dev_charges(const float charges[8]) {
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_charges), (void*)charges,
                                  sizeof(dev_charges)));
}

void
init_dev_masses(const float masses[8]) {
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_masses), (void*)masses,
                                  sizeof(dev_masses)));
}

void
init_dev_bg_fields(const cu_vector_field<Scalar>& E,
                   const cu_vector_field<Scalar>& B) {
  FieldData data;
  data.E1 = E.ptr(0);
  data.E2 = E.ptr(1);
  data.E3 = E.ptr(2);
  data.B1 = B.ptr(0);
  data.B2 = B.ptr(1);
  data.B3 = B.ptr(2);
  CudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(dev_bg_fields), (void*)&data,
                                  sizeof(FieldData)));
}

void
get_dev_params(SimParams& params) {
  CudaSafeCall(hipMemcpyFromSymbol((void*)&params, dev_params,
                                    sizeof(SimParamsBase)));
}

void
get_dev_mesh(Quadmesh& mesh) {
  CudaSafeCall(
      hipMemcpyFromSymbol((void*)&mesh, dev_mesh, sizeof(Quadmesh)));
}

void
get_dev_charges(float charges[]) {
  CudaSafeCall(hipMemcpyFromSymbol((void*)charges, dev_charges,
                                    sizeof(dev_charges)));
}

void
get_dev_masses(float masses[]) {
  CudaSafeCall(hipMemcpyFromSymbol((void*)masses, dev_masses,
                                    sizeof(dev_masses)));
}

}  // namespace Aperture