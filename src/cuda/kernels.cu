#include "hip/hip_runtime.h"
#include "cuda/constant_mem.h"
#include "cuda/cudaUtility.h"
#include "cuda/kernels.h"
#include "utils/util_functions.h"

namespace Aperture {

namespace Kernels {

__global__ void
init_rand_states(hiprandState* states, int seed) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, id, 0, &states[id]);
}

__global__ void
compute_tile(uint32_t* tile, const uint32_t* cell, size_t num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    if (i < num) tile[i] = dev_mesh.tile_id(cell[i]);
  }
}

__global__ void
erase_ptc_in_guard_cells(uint32_t* cell, size_t num) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    if (i < num) {
      auto c = cell[i];
      if (!dev_mesh.is_in_bulk(c)) cell[i] = MAX_CELL;
    }
  }
}

__global__ void
compute_energy_histogram(uint32_t* hist, const Scalar* E, size_t num,
                         int num_bins, Scalar E_max) {
  Scalar dlogE = std::log(E_max) / (Scalar)num_bins;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    if (i < num) {
      Scalar logE = std::log(E[i]);
      int idx = (int)floorf(logE / dlogE);
      if (idx < 0) idx = 0;
      if (idx >= num_bins) idx = num_bins - 1;

      atomicAdd(&hist[idx], 1);
    }
  }
}
__global__ void
compute_energy_histogram(uint32_t* hist, const Scalar* E, size_t num,
                         int num_bins, Scalar E_max,
                         const uint32_t* flags, ParticleFlag flag) {
  Scalar dlogE = std::log(E_max) / (Scalar)num_bins;
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    if (i < num) {
      if (!check_bit(flags[i], flag)) continue;
      Scalar logE = std::log(E[i]);
      int idx = (int)floorf(logE / dlogE);
      if (idx < 0) idx = 0;
      if (idx >= num_bins) idx = num_bins - 1;

      atomicAdd(&hist[idx], 1);
    }
  }
}

__global__ void
map_tracked_ptc(uint32_t* flags, uint32_t* cells, size_t num,
                uint32_t* tracked_map, uint32_t* num_tracked) {
  for (size_t n = threadIdx.x + blockIdx.x * blockDim.x; n < num;
       n += blockDim.x * gridDim.x) {
    if (check_bit(flags[n], ParticleFlag::tracked) &&
        cells[n] != MAX_CELL) {
      uint32_t nt = atomicAdd(num_tracked, 1u);
      if (nt < MAX_TRACKED) {
        tracked_map[nt] = n;
      }
    }
  }
}

__global__ void
adjust_cell_number(uint32_t* cells, size_t num, int shift) {
  for (size_t n = threadIdx.x + blockIdx.x * blockDim.x; n < num;
       n += blockDim.x * gridDim.x) {
    cells[n] += shift;
  }
}

}  // namespace Kernels

void
compute_tile(uint32_t* tile, const uint32_t* cell, size_t num) {
  Kernels::compute_tile<<<256, 256>>>(tile, cell, num);
  // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();
  CudaCheckError();
}

void
erase_ptc_in_guard_cells(uint32_t* cell, size_t num) {
  Kernels::erase_ptc_in_guard_cells<<<512, 512>>>(cell, num);
  // Wait for GPU to finish
  // hipDeviceSynchronize();
  CudaCheckError();
}

void
compute_energy_histogram(uint32_t* hist, const Scalar* E, size_t num,
                         int num_bins, Scalar Emax) {
  Kernels::compute_energy_histogram<<<512, 512>>>(hist, E, num,
                                                  num_bins, Emax);
  // Wait for GPU to finish
  // hipDeviceSynchronize();
  CudaCheckError();
}

void
compute_energy_histogram(uint32_t* hist, const Scalar* E, size_t num,
                         int num_bins, Scalar Emax,
                         const uint32_t* flags, ParticleFlag flag) {
  Kernels::compute_energy_histogram<<<512, 512>>>(
      hist, E, num, num_bins, Emax, flags, flag);
  // Wait for GPU to finish
  // hipDeviceSynchronize();
  CudaCheckError();
}

void
init_rand_states(hiprandState* states, int seed, int blockPerGrid,
                 int threadPerBlock) {
  Kernels::init_rand_states<<<blockPerGrid, threadPerBlock>>>(states,
                                                              seed);
  CudaCheckError();
}

void
map_tracked_ptc(uint32_t* flags, uint32_t* cells, size_t num,
                uint32_t* tracked_map, uint32_t* num_tracked) {
  int block_num = std::min(1024ul, (num + 511) / 512);
  Kernels::map_tracked_ptc<<<block_num, 512>>>(flags, cells, num, tracked_map,
                                               num_tracked);
  CudaCheckError();
  CudaSafeCall(hipDeviceSynchronize());
}

void
adjust_cell_number(uint32_t* cells, size_t num, int shift) {
  int block_num = std::min(1024ul, (num + 511) / 512);
  Kernels::adjust_cell_number<<<block_num, 512>>>(cells, num, shift);
  CudaCheckError();
  CudaSafeCall(hipDeviceSynchronize());
}

}  // namespace Aperture
