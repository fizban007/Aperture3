#include "hip/hip_runtime.h"
#include "algorithms/ptc_updater_logsph.h"
#include "cuda/algorithms/ptc_updater_helper.cu"
#include "cuda/constant_mem.h"
#include "cuda/cudaUtility.h"
#include "cuda/data_ptrs.h"
#include "cuda/grids/grid_log_sph_ptrs.h"
#include "cuda/kernels.h"
#include "cuda/ptr_util.h"
#include "cuda/utils/iterate_devices.h"
#include "sim_data.h"
#include "sim_environment.h"
#include "utils/logger.h"
#include "utils/timer.h"
#include "utils/util_functions.h"

#include "cuda/algorithms/user_push_2d_logsph.cu"

namespace Aperture {

namespace Kernels {

__device__ Scalar beta_phi(Scalar r, Scalar theta);

__device__ Scalar alpha_gr(Scalar r);

HD_INLINE void
cart2logsph(Scalar &v1, Scalar &v2, Scalar &v3, Scalar x1, Scalar x2,
            Scalar x3) {
  Scalar v1n = v1, v2n = v2, v3n = v3;
  Scalar c2 = cos(x2), s2 = sin(x2), c3 = cos(x3), s3 = sin(x3);
  v1 = v1n * s2 * c3 + v2n * s2 * s3 + v3n * c2;
  v2 = v1n * c2 * c3 + v2n * c2 * s3 - v3n * s2;
  v3 = -v1n * s3 + v2n * c3;
}

HD_INLINE void
logsph2cart(Scalar &v1, Scalar &v2, Scalar &v3, Scalar x1, Scalar x2,
            Scalar x3) {
  Scalar v1n = v1, v2n = v2, v3n = v3;
  Scalar c2 = cos(x2), s2 = sin(x2), c3 = cos(x3), s3 = sin(x3);
  v1 = v1n * s2 * c3 + v2n * c2 * c3 - v3n * s3;
  v2 = v1n * s2 * s3 + v2n * c2 * s3 + v3n * c3;
  v3 = v1n * c2 - v2n * s2;
}

__global__ void
__launch_bounds__(256, 4)
vay_push_logsph_2d(data_ptrs data, size_t num, Scalar dt,
                   hiprandState *states) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = states[tid];
  for (size_t idx = tid; idx < num; idx += blockDim.x * gridDim.x) {
    user_push_2d_logsph<1>(data, idx, dt, localState);
  }
  states[tid] = localState;
}

__global__ void
move_photons(photon_data photons, size_t num, Scalar dt, bool axis0,
             bool axis1) {
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = photons.cell[idx];
    // Skip empty particles
    if (c == MAX_CELL) continue;
    // Load particle quantities
    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    auto v1 = photons.p1[idx], v2 = photons.p2[idx],
         v3 = photons.p3[idx];
    Scalar E = std::sqrt(v1 * v1 + v2 * v2 + v3 * v3);
    v1 = v1 / E;
    v2 = v2 / E;
    v3 = v3 / E;

    auto old_x1 = photons.x1[idx], old_x2 = photons.x2[idx],
         old_x3 = photons.x3[idx];

    // Compute the actual movement
    Scalar r1 = dev_mesh.pos(0, c1, old_x1);
    Scalar exp_r1 = std::exp(r1);

    // Censor photons already outside the conversion radius
    if (exp_r1 > dev_params.r_cutoff || exp_r1 < 1.02) {
      photons.cell[idx] = MAX_CELL;
      continue;
    }

    Scalar r2 = dev_mesh.pos(1, c2, old_x2);
    Scalar x = exp_r1 * std::sin(r2) * std::cos(old_x3);
    Scalar y = exp_r1 * std::sin(r2) * std::sin(old_x3);
    Scalar z = exp_r1 * std::cos(r2);

    logsph2cart(v1, v2, v3, r1, r2, old_x3);
    x += v1 * dt;
    y += v2 * dt;
    z += v3 * dt;
    Scalar r1p = sqrt(x * x + y * y + z * z);
    Scalar r2p = acos(z / r1p);
    r1p = log(r1p);
    Scalar r3p = atan2(y, x);

    cart2logsph(v1, v2, v3, r1p, r2p, r3p);
    photons.p1[idx] = v1 * E;
    photons.p2[idx] = v2 * E;
    photons.p3[idx] = v3 * E;

    Pos_t new_x1 = old_x1 + (r1p - r1) / dev_mesh.delta[0];
    Pos_t new_x2 = old_x2 + (r2p - r2) / dev_mesh.delta[1];
    // printf("new_x1 is %f, new_x2 is %f, old_x1 is %f, old_x2 is
    // %f\n", new_x1, new_x2, old_x1, old_x2);
    int dc1 = floor(new_x1);
    int dc2 = floor(new_x2);
    photons.cell[idx] = dev_mesh.get_idx(c1 + dc1, c2 + dc2);
    new_x1 -= (Pos_t)dc1;
    new_x2 -= (Pos_t)dc2;
    // reflect around the axis
    if (dev_mesh.pos(1, c2 + dc2, new_x2) < 0.0f) {
      dc2 += 1;
      new_x2 = 1.0f - new_x2;
    } else if (dev_mesh.pos(1, c2 + dc2, new_x2) >= CONST_PI) {
      dc2 -= 1;
      new_x2 = 1.0f - new_x2;
    }
    // printf("new_x1 is %f, new_x2 is %f, dc2 = %d\n", new_x1, new_x2,
    // dc2);
    photons.x1[idx] = new_x1;
    photons.x2[idx] = new_x2;
    photons.x3[idx] = r3p;
    photons.path_left[idx] -= dt;
  }
}

template <int N>
__global__ void
__launch_bounds__(256, 4)
    deposit_and_move_2d_log_sph(data_ptrs data, size_t num,
                                mesh_ptrs_log_sph mesh_ptrs, Scalar dt,
                                uint32_t step, bool axis0, bool axis1) {
  using spline = Spline::spline_t<N>;
  auto &ptc = data.particles;
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = ptc.cell[idx];
    // Skip empty particles
    if (c == MAX_CELL || idx >= num) continue;

    // Load particle quantities
    Interpolator2D<spline> interp;
    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    auto v1 = ptc.p1[idx], v2 = ptc.p2[idx], v3 = ptc.p3[idx];
    Scalar gamma = ptc.E[idx];
    // printf("gamma is %f\n", gamma);
    // printf("velocity before is (%f, %f, %f)\n", v1, v2, v3);

    auto flag = ptc.flag[idx];
    int sp = get_ptc_type(flag);
    auto w = ptc.weight[idx];
    auto old_x1 = ptc.x1[idx], old_x2 = ptc.x2[idx],
         old_x3 = ptc.x3[idx];

    Scalar r1 = dev_mesh.pos(0, c1, old_x1);
    Scalar exp_r1 = std::exp(r1);
    Scalar r2 = dev_mesh.pos(1, c2, old_x2);

    v1 = v1 / gamma;
    v2 = v2 / gamma;
    v3 = v3 / gamma;
    Scalar v3_gr = v3 - beta_phi(exp_r1, r2);

    // step 1: Compute particle movement and update position
    Scalar x = exp_r1 * std::sin(r2) * std::cos(old_x3);
    Scalar y = exp_r1 * std::sin(r2) * std::sin(old_x3);
    Scalar z = exp_r1 * std::cos(r2);
    // printf("cart position is (%f, %f, %f)\n", x, y, z);

    logsph2cart(v1, v2, v3_gr, r1, r2, old_x3);
    // printf("cart velocity is (%f, %f, %f)\n", v1, v2, v3);
    x += alpha_gr(exp_r1) * v1 * dt;
    y += alpha_gr(exp_r1) * v2 * dt;
    // z += alpha_gr(exp_r1) * (v3 - beta_phi(exp_r1, r2)) * dt;
    z += alpha_gr(exp_r1) * v3_gr * dt;
    // printf("new cart position is (%f, %f, %f)\n", x, y, z);
    Scalar r1p = sqrt(x * x + y * y + z * z);
    Scalar r2p = acos(z / r1p);
    Scalar exp_r1p = r1p;
    r1p = log(r1p);
    Scalar r3p = atan2(y, x);
    // if (x < 0.0f) v1 *= -1.0f;

    // printf("new position is (%f, %f, %f)\n", exp(r1p), r2p, r3p);

    cart2logsph(v1, v2, v3_gr, r1p, r2p, r3p);
    ptc.p1[idx] = v1 * gamma;
    ptc.p2[idx] = v2 * gamma;
    ptc.p3[idx] = (v3_gr + beta_phi(exp_r1p, r2p)) * gamma;

    // Scalar old_pos3 =
    Pos_t new_x1 = old_x1 + (r1p - r1) / dev_mesh.delta[0];
    Pos_t new_x2 = old_x2 + (r2p - r2) / dev_mesh.delta[1];
    // printf("new_x1 is %f, new_x2 is %f, old_x1 is %f, old_x2 is
    // %f\n", new_x1, new_x2, old_x1, old_x2);
    int dc1 = floor(new_x1);
    int dc2 = floor(new_x2);
#ifndef NDEBUG
    if (dc1 > 1 || dc1 < -1 || dc2 > 1 || dc2 < -1)
      printf("----------------- Error: moved more than 1 cell!");
#endif
    new_x1 -= (Pos_t)dc1;
    new_x2 -= (Pos_t)dc2;
    // reflect around the axis
    if (dev_mesh.pos(1, c2 + dc2, new_x2) < 0.0f) {
      dc2 += 1;
      new_x2 = 1.0f - new_x2;
      ptc.p2[idx] *= -1.0;
      ptc.p3[idx] *= -1.0;
    }
    if (dev_mesh.pos(1, c2 + dc2, new_x2) >= CONST_PI) {
      dc2 -= 1;
      new_x2 = 1.0f - new_x2;
      ptc.p2[idx] *= -1.0;
      ptc.p3[idx] *= -1.0;
    }
    ptc.cell[idx] = dev_mesh.get_idx(c1 + dc1, c2 + dc2);
    // printf("new_x1 is %f, new_x2 is %f, dc2 = %d\n", new_x1, new_x2,
    // dc2);
    ptc.x1[idx] = new_x1;
    ptc.x2[idx] = new_x2;
    ptc.x3[idx] = r3p;

    // printf("c1 %d, c2 %d, x1 %f, x2 %f, v1 %f, v2 %f\n", c1, c2,
    // new_x1,
    //        new_x2, v1, v2);

    // step 2: Deposit current
    if (check_bit(flag, ParticleFlag::ignore_current)) continue;
    Scalar weight = -dev_charges[sp] * w;

    int j_0 = (dc2 == -1 ? -spline::radius - 1 : -spline::radius);
    int j_1 = (dc2 == 1 ? spline::radius : spline::radius - 1);
    int i_0 = (dc1 == -1 ? -spline::radius - 1 : -spline::radius);
    int i_1 = (dc1 == 1 ? spline::radius : spline::radius - 1);
    Scalar djy[spline::support + 1] = {0.0f};
    for (int j = j_0; j <= j_1; j++) {
      Scalar sy0 = interp.interpolate(-old_x2 + j + 1);
      Scalar sy1 = interp.interpolate(-new_x2 + (j + 1 - dc2));

      size_t j_offset = (j + c2) * data.J1.p.pitch;
      Scalar djx = 0.0f;
      for (int i = i_0; i <= i_1; i++) {
        Scalar sx0 = interp.interpolate(-old_x1 + i + 1);
        Scalar sx1 = interp.interpolate(-new_x1 + (i + 1 - dc1));

        // j1 is movement in r
        int offset = j_offset + (i + c1) * sizeof(Scalar);
        Scalar val0 = movement2d(sy0, sy1, sx0, sx1);
        djx += val0;
        atomicAdd(&data.J1[offset + sizeof(Scalar)], weight * djx);

        // j2 is movement in theta
        Scalar val1 = movement2d(sx0, sx1, sy0, sy1);
        djy[i - i_0] += val1;
        atomicAdd(&data.J2[offset + data.J2.p.pitch],
                  weight * djy[i - i_0]);

        // j3 is simply v3 times rho at volume average
        Scalar val2 = center2d(sx0, sx1, sy0, sy1);
        atomicAdd(&data.J3[offset],
                  // -weight * (v3 - beta_phi(exp_r1, r2)) * val2 /
                  // mesh_ptrs.dV[offset]);
                  -weight * v3_gr * val2 / mesh_ptrs.dV[offset]);

        // rho is deposited at the final position
        if ((step + 1) % dev_params.data_interval == 0) {
          Scalar s1 = sx1 * sy1;
          atomicAdd(&data.Rho[sp][offset], -weight * s1);
        }
      }
    }
  }
}

__global__ void
process_j(data_ptrs data, mesh_ptrs_log_sph mesh_ptrs, Scalar dt) {
  for (int j = blockIdx.y * blockDim.y + threadIdx.y;
       j < dev_mesh.dims[1]; j += blockDim.y * gridDim.y) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
      size_t offset = data.J1.compute_offset(i, j);
      Scalar w = dev_mesh.delta[0] * dev_mesh.delta[1] / dt;
      data.J1[offset] *= w / mesh_ptrs.A1_e[offset];
      data.J2[offset] *= w / mesh_ptrs.A2_e[offset];
      for (int n = 0; n < dev_params.num_species; n++) {
        data.Rho[n][offset] /= mesh_ptrs.dV[offset];
      }
    }
  }
}

__global__ void
inject_ptc(data_ptrs data, size_t num, int inj_per_cell, Scalar p1,
           Scalar p2, Scalar p3, Scalar w, Scalar *surface_e,
           Scalar *surface_p, hiprandState *states, Scalar omega) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState localState = states[id];
  auto &ptc = data.particles;
  // int inject_i = dev_mesh.guard[0] + 3;
  int inject_i = dev_mesh.guard[0] + 2;
  ParticleType p_type =
      (dev_params.inject_ions ? ParticleType::ion
                              : ParticleType::positron);
  for (int i = dev_mesh.guard[1] + 1 + id;
       // i = dev_mesh.dims[1] - dev_mesh.guard[1] - 3 + id;
       i < dev_mesh.dims[1] - dev_mesh.guard[1] - 1;
       i += blockDim.x * gridDim.x) {
    size_t offset = num + i * inj_per_cell * 2;
    Scalar r = exp(dev_mesh.pos(0, inject_i, 0.5f));
    // Scalar dens = max(-*ptrAddr(rho0, dev_mesh.guard[0] + 2, i),
    //                   *ptrAddr(rho1, dev_mesh.guard[0] + 2, i));
    Scalar dens = max(surface_e[i - dev_mesh.guard[1]],
                      surface_p[i - dev_mesh.guard[1]]);
    Scalar omega_LT = 0.4f * omega * dev_params.compactness;
    // if (i == dev_mesh.dims[1] / 2)
    //   printf("dens_e is %f, dens_p is %f, limit is %f\n",
    //          dev_params.q_e * surface_e[i - dev_mesh.guard[1]],
    //          dev_params.q_e * surface_p[i - dev_mesh.guard[1]],
    //          0.4 * square(1.0f / dev_mesh.delta[1]) *
    //              std::sin(dev_mesh.pos(1, i, 0.5f)));
    Scalar sin_theta = std::sin(dev_mesh.pos(1, i, 0.5f));
    if (dev_params.q_e * dens >
        1.5f * square(1.0f / dev_mesh.delta[1]) * sin_theta)
      continue;
    // Scalar Er = data.E1(inject_i, i);
    // Scalar n_inj =
    //     0.2 * std::abs(Er) / (dev_mesh.delta[0] * dev_params.q_e);
    for (int n = 0; n < inj_per_cell; n++) {
    // for (int n = 0; n < n_inj; n++) {
      Pos_t x2 = hiprand_uniform(&localState);
      Scalar theta = dev_mesh.pos(1, i, x2);
      // Scalar vphi = (omega - omega_LT) * r * sin(theta);
      // Scalar vphi = omega * r * sin(theta);
      Scalar vphi = 0.0f;
      // Scalar w_ptc = w * sin(theta) * std::abs(cos(theta));
      Scalar w_ptc = w * sin(theta);
      // Scalar gamma = 1.0f / std::sqrt(1.0f - vphi * vphi);
      Scalar gamma = std::sqrt(1.0 + p1 * p1 + vphi * vphi);
      float u = hiprand_uniform(&localState);
      ptc.x1[offset + n * 2] = 0.5f;
      ptc.x2[offset + n * 2] = x2;
      ptc.x3[offset + n * 2] = 0.0f;
      ptc.p1[offset + n * 2] = p1 * 2.0f * std::abs(cos(theta));
      ptc.p2[offset + n * 2] = p1 * sin(theta) * sgn(cos(theta));
      // ptc.p1[offset + n * 2] = p1;
      // ptc.p2[offset + n * 2] = p2;
      ptc.p3[offset + n * 2] = vphi;
      ptc.E[offset + n * 2] = gamma;
      // sqrt(1.0f + p1 * p1 + p2 * p2 + vphi * vphi);
      // printf("inject E is %f\n", ptc.E[offset + n * 2]);
      // ptc.p3[offset + n * 2] = p3;
      ptc.cell[offset + n * 2] = dev_mesh.get_idx(inject_i, i);
      ptc.weight[offset + n * 2] = w_ptc;
      ptc.flag[offset + n * 2] = set_ptc_type_flag(
          (u < dev_params.track_percent
               ? bit_or(ParticleFlag::primary, ParticleFlag::tracked)
               : bit_or(ParticleFlag::primary)),
          ParticleType::electron);

      ptc.x1[offset + n * 2 + 1] = 0.5f;
      ptc.x2[offset + n * 2 + 1] = x2;
      ptc.x3[offset + n * 2 + 1] = 0.0f;
      ptc.p1[offset + n * 2 + 1] = p1 * 2.0f * std::abs(cos(theta));
      ptc.p2[offset + n * 2 + 1] = p1 * sin(theta) * sgn(cos(theta));
      // ptc.p1[offset + n * 2 + 1] = p1;
      // ptc.p2[offset + n * 2 + 1] = p2;
      ptc.p3[offset + n * 2 + 1] = vphi;
      ptc.E[offset + n * 2 + 1] = gamma;
      // sqrt(1.0f + p1 * p1 + p2 * p2 + vphi * vphi);
      // printf("inject E is %f\n", ptc.E[offset + n * 2 + 1]);
      // ptc.p3[offset + n * 2 + 1] = p3;
      ptc.cell[offset + n * 2 + 1] = dev_mesh.get_idx(inject_i, i);
      ptc.weight[offset + n * 2 + 1] = w_ptc;
      ptc.flag[offset + n * 2 + 1] = set_ptc_type_flag(
          (u < dev_params.track_percent
               ? bit_or(ParticleFlag::primary, ParticleFlag::tracked)
               : bit_or(ParticleFlag::primary)),
          p_type);
      if (u < dev_params.track_percent) {
        ptc.id[offset + n * 2] = dev_rank + atomicAdd(&dev_ptc_id, 1);
        ptc.id[offset + n * 2 + 1] = dev_rank + atomicAdd(&dev_ptc_id, 1);
      }
    }
  }
  states[id] = localState;
}

__global__ void
ptc_outflow(particle_data ptc, size_t num) {
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = ptc.cell[idx];
    if (c == MAX_CELL || idx >= num) continue;

    int c1 = dev_mesh.get_c1(c);
    auto flag = ptc.flag[idx];
    if (check_bit(flag, ParticleFlag::ignore_EM)) continue;
    if (c1 > dev_mesh.dims[0] - dev_params.damping_length + 2) {
      flag |= bit_or(ParticleFlag::ignore_EM);
      ptc.flag[idx] = flag;
    }
  }
}

__global__ void
axis_rho_lower(data_ptrs data, mesh_ptrs_log_sph mesh_ptrs) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
    int j_0 = dev_mesh.guard[1];
    data.J3(i, j_0 - 1) = 0.0f;
    data.J3(i, j_0) = 0.0f;
    // fields.J2(i, j_0) -= fields.J2(i, j_0 - 1);
    // fields.J2(i, j_0 - 1) = 0.0;
  }
}

__global__ void
axis_rho_upper(data_ptrs data, mesh_ptrs_log_sph mesh_ptrs) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
    if (i >= dev_mesh.dims[0]) continue;
    int j_last = dev_mesh.dims[1] - dev_mesh.guard[1];
    // fields.J2(i, j_last - 1) -= fields.J2(i, j_last);
    // fields.J2(i, j_last) = 0.0;

    data.J3(i, j_last) = 0.0f;
    data.J3(i, j_last - 1) = 0.0f;
  }
}

__global__ void
measure_surface_density(particle_data ptc, size_t num,
                        Scalar *surface_e, Scalar *surface_p) {
  for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    auto c = ptc.cell[idx];
    if (c == MAX_CELL || idx >= num) continue;

    // Load particle quantities
    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    // Sum over 3 cells, hense the w / 3.0f in the atomicAdd
    int sum_cells = 3;
    // int inject_cell = dev_mesh.guard[0] + 3;
    int inject_cell = dev_mesh.guard[0] + 2;
    if (c1 >= inject_cell - 1 && c1 <= inject_cell - 1 + sum_cells) {
      auto flag = ptc.flag[idx];
      int sp = get_ptc_type(flag);
      auto w = ptc.weight[idx];
      if (sp == (int)ParticleType::electron) {
        atomicAdd(&surface_e[max(c2 - dev_mesh.guard[1] - 2, 0)],
                  1.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_e[max(c2 - dev_mesh.guard[1] - 1, 0)],
                  4.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_e[c2 - dev_mesh.guard[1]],
                  6.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_e[min(
                      c2 - dev_mesh.guard[1] + 1,
                      dev_mesh.dims[1] - 2 * dev_mesh.guard[1] - 1)],
                  4.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_e[min(
                      c2 - dev_mesh.guard[1] + 2,
                      dev_mesh.dims[1] - 2 * dev_mesh.guard[1] - 1)],
                  1.0f * w / float(sum_cells) / 16.0f);
      } else if (sp == (int)ParticleType::ion) {
        // } else if (sp == (int)ParticleType::positron) {
        atomicAdd(&surface_p[max(c2 - dev_mesh.guard[1] - 2, 0)],
                  1.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_p[max(c2 - dev_mesh.guard[1] - 1, 0)],
                  4.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_p[c2 - dev_mesh.guard[1]],
                  6.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_p[min(
                      c2 - dev_mesh.guard[1] + 1,
                      dev_mesh.dims[1] - 2 * dev_mesh.guard[1] - 1)],
                  4.0f * w / float(sum_cells) / 16.0f);
        atomicAdd(&surface_p[min(
                      c2 - dev_mesh.guard[1] + 2,
                      dev_mesh.dims[1] - 2 * dev_mesh.guard[1] - 1)],
                  1.0f * w / float(sum_cells) / 16.0f);
      }
    }
  }
}

__global__ void
annihilate_pairs(particle_data ptc, size_t num, pitchptr<Scalar> j1,
                 pitchptr<Scalar> j2, pitchptr<Scalar> j3) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num;
       idx += blockDim.x * gridDim.x) {
    // First do a deposit before annihilation
    auto c = ptc.cell[idx];
    auto flag = ptc.flag[idx];
    // Skip empty particles
    if (c == MAX_CELL || !check_bit(flag, ParticleFlag::annihilate))
      continue;

    // Load particle quantities
    Interpolator2D<spline_t> interp;
    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    int sp = get_ptc_type(flag);
    auto w = ptc.weight[idx];
    auto old_x1 = ptc.x1[idx], old_x2 = ptc.x2[idx];

    Pos_t new_x1 = 0.5f;
    Pos_t new_x2 = 0.5f;

    // Move the particles to be annihilated to the center of the cell
    ptc.x1[idx] = new_x1;
    ptc.x2[idx] = new_x2;

    // Deposit extra current due to this movement
    if (!check_bit(flag, ParticleFlag::ignore_current)) {
      Scalar weight = -dev_charges[sp] * w;

      Scalar djy[3] = {0.0f};
      for (int j = -1; j <= 0; j++) {
        Scalar sy0 = interp.interpolate(-old_x2 + j + 1);
        Scalar sy1 = interp.interpolate(-new_x2 + j + 1);

        // size_t j_offset = (j + c2) * j1.pitch;
        Scalar djx = 0.0f;
        for (int i = -1; i <= 0; i++) {
          Scalar sx0 = interp.interpolate(-old_x1 + i + 1);
          Scalar sx1 = interp.interpolate(-new_x1 + i + 1);

          // j1 is movement in r
          Scalar val0 = movement2d(sy0, sy1, sx0, sx1);
          djx += val0;
          atomicAdd(&j1(i + c1 + 1, j + c2), weight * djx);

          // j2 is movement in theta
          Scalar val1 = movement2d(sx0, sx1, sy0, sy1);
          djy[i + 1] += val1;
          atomicAdd(&j2(i + c1, j + c2 + 1), weight * djy[i + 1]);
        }
      }
    }

    // Actually kill the particle
    ptc.cell[idx] = MAX_CELL;
  }
}

__global__ void
flag_annihilation(particle_data data, size_t num, pitchptr<Scalar> dens,
                  pitchptr<Scalar> balance) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < num;
       i += blockDim.x * gridDim.x) {
    auto c = data.cell[i];
    // Skip empty particles
    if (c == MAX_CELL) continue;

    auto flag = data.flag[i];
    if (get_ptc_type(flag) > 1) continue;  // ignore ions
    auto w = data.weight[i];

    int c1 = dev_mesh.get_c1(c);
    int c2 = dev_mesh.get_c2(c);
    // size_t offset = c1 * sizeof(Scalar) + c2 * dens.pitch;

    Scalar n = atomicAdd(&dens(c1, c2), w);
    Scalar r = std::exp(dev_mesh.pos(0, c1, 0.5f));
    // TODO: implement the proper condition
    if (n >
        0.5 * dev_mesh.inv_delta[0] * dev_mesh.inv_delta[0] / (r * r)) {
      set_bit(flag, ParticleFlag::annihilate);
      atomicAdd(&balance(c1, c2),
                w * (get_ptc_type(flag) == (int)ParticleType::electron
                         ? -1.0f
                         : 1.0f));
    }
  }
  // After this operation, the balance array will contain how much
  // imbalance is there in the annihilated part. We will add this
  // imbalance back in as an extra particle
}

__global__ void
add_extra_particles(particle_data ptc, size_t num,
                    pitchptr<Scalar> balance) {
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  int num_offset = n2 * dev_mesh.dims[0] + n1;

  ptc.cell[num + num_offset] = num_offset;
  ptc.x1[num + num_offset] = 0.5f;
  ptc.x2[num + num_offset] = 0.5f;
  ptc.x3[num + num_offset] = 0.0f;
  ptc.p1[num + num_offset] = 0.0f;
  ptc.p2[num + num_offset] = 0.0f;
  ptc.p3[num + num_offset] = 0.0f;
  ptc.E[num + num_offset] = 1.0f;
  Scalar w = balance(n1, n2);
  ptc.weight[num + num_offset] = std::abs(w);
  if (w > 0)
    ptc.flag[num + num_offset] =
        set_ptc_type_flag(0, ParticleType::positron);
  else
    ptc.flag[num + num_offset] =
        set_ptc_type_flag(0, ParticleType::electron);
}

__global__ void
filter_current_logsph(pitchptr<Scalar> j, pitchptr<Scalar> j_tmp,
                      pitchptr<Scalar> A, bool boundary_lower0,
                      bool boundary_upper0, bool boundary_lower1,
                      bool boundary_upper1) {
  // Load position parameters
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  // size_t globalOffset = n2 * j.p.pitch + n1 * sizeof(Scalar);
  size_t globalOffset = j.compute_offset(n1, n2);

  size_t dr_plus = sizeof(Scalar);
  if (boundary_upper0 && n1 == dev_mesh.dims[0] - dev_mesh.guard[0] - 1)
    dr_plus = 0;
  // (n1 < dev_mesh.dims[0] - dev_mesh.guard[0] - 1 ? sizeof(Scalar)
  //                                                : 0);
  size_t dr_minus = sizeof(Scalar);
  if (boundary_lower0 && n1 == dev_mesh.guard[0]) dr_minus = 0;
  // (n1 > dev_mesh.guard[0] ? sizeof(Scalar) : 0);
  size_t dt_plus = j.p.pitch;
  if (boundary_upper1 && n2 == dev_mesh.dims[1] - dev_mesh.guard[1] - 1)
    dt_plus = 0;
  // (n2 < dev_mesh.dims[1] - dev_mesh.guard[1] - 1 ? j.pitch : 0);
  size_t dt_minus = j.p.pitch;
  if (boundary_lower1 && n2 == dev_mesh.guard[1]) dt_minus = 0;
  // (n2 > dev_mesh.guard[1] ? j.pitch : 0);
  // Do the actual computation here
  j_tmp[globalOffset] = 0.25f * j[globalOffset] * A[globalOffset];
  j_tmp[globalOffset] +=
      0.125f * j[globalOffset + dr_plus] * A[globalOffset + dr_plus];
  j_tmp[globalOffset] +=
      0.125f * j[globalOffset - dr_minus] * A[globalOffset - dr_minus];
  j_tmp[globalOffset] +=
      0.125f * j[globalOffset + dt_plus] * A[globalOffset + dt_plus];
  j_tmp[globalOffset] +=
      0.125f * j[globalOffset - dt_minus] * A[globalOffset - dt_minus];
  j_tmp[globalOffset] += 0.0625f * j[globalOffset + dr_plus + dt_plus] *
                         A[globalOffset + dr_plus + dt_plus];
  j_tmp[globalOffset] += 0.0625f *
                         j[globalOffset - dr_minus + dt_plus] *
                         A[globalOffset - dr_minus + dt_plus];
  j_tmp[globalOffset] += 0.0625f *
                         j[globalOffset + dr_plus - dt_minus] *
                         A[globalOffset + dr_plus - dt_minus];
  j_tmp[globalOffset] += 0.0625f *
                         j[globalOffset - dr_minus - dt_minus] *
                         A[globalOffset - dr_minus - dt_minus];
  j_tmp[globalOffset] /= A[globalOffset];
}

}  // namespace Kernels

ptc_updater_logsph::ptc_updater_logsph(sim_environment &env)
    : m_env(env),
      m_surface_e(env.params().N[1]),
      m_surface_p(env.params().N[1]),
      m_surface_tmp(env.params().N[1]) {
  m_tmp_j1 = multi_array<Scalar>(env.local_grid().extent());
  m_tmp_j2 = multi_array<Scalar>(env.local_grid().extent());
}

ptc_updater_logsph::~ptc_updater_logsph() {}

void
ptc_updater_logsph::update_particles(sim_data &data, double dt,
                                     uint32_t step) {
  timer::stamp("ptc_update");
  auto data_p = get_data_ptrs(data);

  if (m_env.grid().dim() == 2) {
    data.J.initialize();
    for (int i = 0; i < data.env.params().num_species; i++) {
      data.Rho[i].initialize();
    }
    Grid_LogSph *grid =
        dynamic_cast<Grid_LogSph *>(&m_env.local_grid());
    auto mesh_ptrs = get_mesh_ptrs(*grid);
    timer::stamp("ptc_push");
    // Skip empty particle array
    if (data.particles.number() > 0) {
      Logger::print_info(
          "Updating {} particles in log spherical coordinates",
          data.particles.number());
      Kernels::vay_push_logsph_2d<<<512, 256>>>(
          data_p, data.particles.number(), dt,
          (hiprandState *)data.d_rand_states);
      CudaCheckError();
    }
    CudaSafeCall(hipDeviceSynchronize());
    timer::show_duration_since_stamp("Pushing particles", "ms",
                                     "ptc_push");

    timer::stamp("ptc_deposit");

    if (data.particles.number() > 0) {
      // m_J1.initialize();
      // m_J2.initialize();
      Kernels::deposit_and_move_2d_log_sph<1><<<512, 256>>>(
          data_p, data.particles.number(), mesh_ptrs, dt, step,
          m_env.is_boundary(2), m_env.is_boundary(3));
      CudaSafeCall(hipDeviceSynchronize());
      CudaCheckError();
      // Kernels::convert_j<<<dim3(32, 32), dim3(32, 32)>>>(
      //     m_J1.ptr(), m_J2.ptr(), m_dev_fields);
      // CudaCheckError();
    }
    timer::show_duration_since_stamp("Depositing particles", "ms",
                                     "ptc_deposit");

    Kernels::process_j<<<dim3(32, 32), dim3(32, 32)>>>(data_p,
                                                       mesh_ptrs, dt);
    CudaCheckError();
    CudaSafeCall(hipDeviceSynchronize());

    // timer::stamp("comm");
    // m_env.send_sub_guard_cells(data.J);
    m_env.send_add_guard_cells(data.J);
    m_env.send_guard_cells(data.J);
    for (int i = 0; i < data.env.params().num_species; i++) {
      m_env.send_add_guard_cells(data.Rho[i]);
      m_env.send_guard_cells(data.Rho[i]);
    }

    Logger::print_debug("current smoothing {} times",
                        m_env.params().current_smoothing);
    for (int i = 0; i < m_env.params().current_smoothing; i++) {
      auto &mesh = grid->mesh();
      dim3 blockSize(32, 16);
      dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 16);

      Kernels::filter_current_logsph<<<gridSize, blockSize>>>(
          get_pitchptr(data.J.data(0)), get_pitchptr(m_tmp_j1),
          mesh_ptrs.A1_e, m_env.is_boundary(0), m_env.is_boundary(1),
          m_env.is_boundary(2), m_env.is_boundary(3));
      data.J.data(0).copy_from(m_tmp_j1);
      CudaCheckError();

      Kernels::filter_current_logsph<<<gridSize, blockSize>>>(
          get_pitchptr(data.J.data(1)), get_pitchptr(m_tmp_j1),
          mesh_ptrs.A2_e, m_env.is_boundary(0), m_env.is_boundary(1),
          m_env.is_boundary(2), m_env.is_boundary(3));
      data.J.data(1).copy_from(m_tmp_j1);
      CudaCheckError();

      Kernels::filter_current_logsph<<<gridSize, blockSize>>>(
          get_pitchptr(data.J.data(2)), get_pitchptr(m_tmp_j1),
          mesh_ptrs.A3_e, m_env.is_boundary(0), m_env.is_boundary(1),
          m_env.is_boundary(2), m_env.is_boundary(3));
      data.J.data(2).copy_from(m_tmp_j1);
      CudaCheckError();

      if ((step + 1) % data.env.params().data_interval == 0) {
        for (int i = 0; i < data.env.params().num_species; i++) {
          Kernels::filter_current_logsph<<<gridSize, blockSize>>>(
              get_pitchptr(data.Rho[i].data()), get_pitchptr(m_tmp_j1),
              mesh_ptrs.dV, m_env.is_boundary(0), m_env.is_boundary(1),
              m_env.is_boundary(2), m_env.is_boundary(3));
          data.Rho[i].data().copy_from(m_tmp_j1);
          CudaCheckError();
        }
      }
      CudaSafeCall(hipDeviceSynchronize());
      m_env.send_guard_cells(data.J);
      if ((step + 1) % data.env.params().data_interval == 0) {
        for (int i = 0; i < data.env.params().num_species; i++) {
          m_env.send_guard_cells(data.Rho[i]);
        }
      }
    }
    // timer::stamp("ph_update");
    // Skip empty particle array
    // timer::show_duration_since_stamp("Updating photons", "us",
    //                                  "ph_update");
  }
  m_env.send_particles(data.particles);
  if (data.photons.number() > 0) {
    Logger::print_info(
        "Updating {} photons in log spherical coordinates",
        data.photons.number());
    Kernels::move_photons<<<256, 512>>>(
        data.photons.data(), data.photons.number(), dt,
        m_env.is_boundary(2), m_env.is_boundary(3));
    CudaCheckError();
  }
  m_env.send_particles(data.photons);
  CudaSafeCall(hipDeviceSynchronize());
  // timer::show_duration_since_stamp("Sending guard cells", "us",
  // "comm");
  // data.send_particles();

  apply_boundary(data, dt, step);
  timer::show_duration_since_stamp("Ptc update", "ms", "ptc_update");
}

void
ptc_updater_logsph::apply_boundary(sim_data &data, double dt,
                                   uint32_t step) {
  auto data_p = get_data_ptrs(data);
  if (data.env.is_boundary((int)BoundaryPos::lower0)) {
  }
  data.particles.clear_guard_cells(m_env.local_grid());
  data.photons.clear_guard_cells(m_env.local_grid());
  CudaSafeCall(hipDeviceSynchronize());
  Grid_LogSph *grid = dynamic_cast<Grid_LogSph *>(&m_env.local_grid());
  auto mesh_ptrs = get_mesh_ptrs(*grid);

  if (data.env.is_boundary((int)BoundaryPos::lower1)) {
    // CudaSafeCall(hipSetDevice(n));
    // Logger::print_debug("Processing boundary {} on device {}",
    // (int)BoundaryPos::lower1, n);
    Kernels::axis_rho_lower<<<1, 512>>>(data_p, mesh_ptrs);
    CudaCheckError();
    // hipDeviceSynchronize();
  }
  if (data.env.is_boundary((int)BoundaryPos::upper1)) {
    // CudaSafeCall(hipSetDevice(n));
    // Logger::print_debug("Processing boundary {} on device {}",
    //                     (int)BoundaryPos::upper1, n);
    Kernels::axis_rho_upper<<<1, 512>>>(data_p, mesh_ptrs);
    CudaCheckError();
    // hipDeviceSynchronize();
  }
  if (data.env.is_boundary((int)BoundaryPos::upper0)) {
    Kernels::ptc_outflow<<<256, 512>>>(data.particles.data(),
                                       data.particles.number());
    CudaCheckError();
  }
  CudaSafeCall(hipDeviceSynchronize());
}

void
ptc_updater_logsph::inject_ptc(sim_data &data, int inj_per_cell,
                               Scalar p1, Scalar p2, Scalar p3,
                               Scalar w, Scalar omega) {
  if (data.env.is_boundary((int)BoundaryPos::lower0)) {
    m_surface_e.assign_dev(0.0);
    m_surface_p.assign_dev(0.0);
    m_surface_tmp.assign_dev(0.0);
    Kernels::measure_surface_density<<<256, 512>>>(
        data.particles.data(), data.particles.number(),
        m_surface_e.dev_ptr(), m_surface_p.dev_ptr());
    CudaCheckError();
    Kernels::inject_ptc<<<128, 256>>>(
        get_data_ptrs(data), data.particles.number(), inj_per_cell, p1,
        p2, p3, w, m_surface_e.dev_ptr(), m_surface_p.dev_ptr(),
        (hiprandState *)data.d_rand_states, omega);
    CudaCheckError();

    data.particles.set_num(data.particles.number() +
                           2 * inj_per_cell *
                               data.E.grid().mesh().reduced_dim(1));
  }
}

// void
// ptc_updater_logsph::annihilate_extra_pairs(sim_data &data) {
//   m_dens.data().assign_dev(0.0);
//   m_balance.data().assign_dev(0.0);

//   Kernels::flag_annihilation<<<256, 512>>>(
//       data.particles.data(), data.particles.number(), m_dens.ptr(),
//       m_balance.ptr());
//   CudaCheckError();

//   Kernels::annihilate_pairs<<<256, 512>>>(
//       data.particles.data(), data.particles.number(), data.J.ptr(0),
//       data.J.ptr(1), data.J.ptr(2));
//   CudaCheckError();

//   auto &mesh = data.E.grid().mesh();
//   dim3 blockSize(32, 16);
//   dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 16);

//   Kernels::add_extra_particles<<<gridSize, blockSize>>>(
//       data.particles.data(), data.particles.number(),
//       m_balance.ptr());
//   CudaCheckError();

//   hipDeviceSynchronize();
// }

}  // namespace Aperture
