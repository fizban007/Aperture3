#include "hip/hip_runtime.h"
#include "algorithms/field_solver_logsph.h"
#include "grids/grid_log_sph.h"
#include "cuda/constant_mem.h"
#include "cuda/cudaUtility.h"
#include "cuda/data_ptrs.h"
#include "cuda/grids/grid_log_sph_ptrs.h"
#include "cuda/ptr_util.h"
#include "cuda/utils/pitchptr.h"
#include "sim_data.h"
#include "sim_environment.h"
#include "utils/timer.h"

namespace Aperture {

namespace Kernels {

__device__ Scalar
beta_phi(Scalar r, Scalar theta) {
  return -0.4f * dev_params.compactness * dev_params.omega *
         std::sin(theta) / (r * r);
}

__device__ Scalar
alpha_gr(Scalar r) {
  // return std::sqrt(1.0f - dev_params.compactness / r);
  return 1.0f;
}

// template <int DIM1, int DIM2>
__global__ void
compute_e_update(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
                 pitchptr<Scalar> e3, pitchptr<Scalar> b1,
                 pitchptr<Scalar> b2, pitchptr<Scalar> b3,
                 pitchptr<Scalar> j1, pitchptr<Scalar> j2,
                 pitchptr<Scalar> j3, mesh_ptrs_log_sph mesh_ptrs,
                 Scalar dt) {
  // Load position parameters
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  size_t globalOffset = e1.compute_offset(n1, n2);

  Scalar r = std::exp(dev_mesh.pos(0, n1, true));
  Scalar theta = dev_mesh.pos(1, n2, true);
  Scalar theta0 = dev_mesh.pos(1, n2, false);
  // Scalar beta = 0.4f * dev_params.omega * dev_params.compactness *
  //               std::sin(theta) / (r * r);
  Scalar r1 = std::exp(dev_mesh.pos(0, n1 + 1, 0));
  Scalar r0 = std::exp(dev_mesh.pos(0, n1, 0));
  // Do the actual computation here
  // (Curl u)_1 = d2u3 - d3u2
  if (std::abs(dev_mesh.pos(1, n2, true) - CONST_PI) < 1.0e-5) {
    e1[globalOffset] += dt * (-4.0f * b3[globalOffset] * alpha_gr(r0) /
                                  (dev_mesh.delta[1] * r0) -
                              // alpha_gr(r0) * j1[globalOffset]);
                              j1[globalOffset]);
  } else {
    e1[globalOffset] +=
        // -dt * j1[globalOffset];
        dt *
        ((b3(n1, n2 + 1) * alpha_gr(r0) * mesh_ptrs.l3_b(n1, n2 + 1) -
          b3(n1, n2) * alpha_gr(r0) * mesh_ptrs.l3_b(n1, n2)) /
             mesh_ptrs.A1_e(n1, n2) -
         // alpha_gr(r0) * j1(n1, n2));
         j1(n1, n2));
  }
  // (Curl u)_2 = d3u1 - d1u3
  e2[globalOffset] +=
      // -dt * j2[globalOffset];
      dt *
      ((b3(n1, n2) * alpha_gr(r0) * mesh_ptrs.l3_b(n1, n2) -
        b3(n1 + 1, n2) * alpha_gr(r1) * mesh_ptrs.l3_b(n1 + 1, n2)) /
           mesh_ptrs.A2_e(n1, n2) -
       // alpha_gr(r) * j2(n1, n2));
       j2(n1, n2));

  // (Curl u)_3 = d1u2 - d2u1
  e3[globalOffset] +=
      // -dt * j3[globalOffset];
      dt * ((b2(n1 + 1, n2) * alpha_gr(r1) *
                 // e1(n1 + 1, n2) * beta_phi(r1, theta)) *
                 mesh_ptrs.l2_b(n1 + 1, n2) -
             b2(n1, n2) * alpha_gr(r0) *
                 // - e1(n1, n2) * beta_phi(r0, theta)) *
                 mesh_ptrs.l2_b(n1, n2) +
             b1(n1, n2) * alpha_gr(r) *
                 // + e2(n1, n2) * beta_phi(r, theta0)) *
                 mesh_ptrs.l1_b(n1, n2) -
             b1(n1, n2 + 1) * alpha_gr(r) *
                 // e2(n1, n2 + 1) * beta_phi(r, theta0 +
                 // dev_mesh.delta[1])) *
                 mesh_ptrs.l1_b(n1, n2 + 1)) /
                mesh_ptrs.A3_e(n1, n2) -
            // alpha_gr(r) * j3(n1, n2) + beta * rho);
            // j3(n1, n2) + beta * rho);
            j3(n1, n2));

  __syncthreads();
  // Extra work for the axis
  if (std::abs(dev_mesh.pos(1, n2, true) - dev_mesh.delta[1]) <
      1.0e-5) {
    n2 = dev_mesh.guard[1] - 1;
    globalOffset = e1.compute_offset(n1, n2);

    e3[globalOffset] = 0.0f;

    e1[globalOffset] += dt * (4.0f * b3(n1, n2 + 1) * alpha_gr(r0) /
                                  (dev_mesh.delta[1] * r0) -
                              // alpha_gr(r0) * j1[globalOffset]);
                              j1[globalOffset]);
  }
}

// template <int DIM1, int DIM2>
__global__ void
compute_b_update(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
                 pitchptr<Scalar> e3, pitchptr<Scalar> b1,
                 pitchptr<Scalar> b2, pitchptr<Scalar> b3,
                 mesh_ptrs_log_sph mesh_ptrs, Scalar dt) {
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  // size_t globalOffset = n2 * e1.pitch + n1 * sizeof(Scalar);
  size_t globalOffset = e1.compute_offset(n1, n2);

  Scalar r1 = std::exp(dev_mesh.pos(0, n1, 1));
  Scalar r0 = std::exp(dev_mesh.pos(0, n1 - 1, 1));
  Scalar r = std::exp(dev_mesh.pos(0, n1, 0));
  // Do the actual computation here
  // (Curl u)_1 = d2u3 - d3u2
  b1[globalOffset] +=
      -dt *
      (e3(n1, n2) * alpha_gr(r1) * mesh_ptrs.l3_e(n1, n2) -
       e3(n1, n2 - 1) * alpha_gr(r1) * mesh_ptrs.l3_e(n1, n2 - 1)) /
      mesh_ptrs.A1_b(n1, n2);

  // (Curl u)_2 = d3u1 - d1u3
  b2[globalOffset] +=
      -dt *
      (e3(n1 - 1, n2) * alpha_gr(r0) * mesh_ptrs.l3_e(n1 - 1, n2) -
       e3(n1, n2) * alpha_gr(r1) * mesh_ptrs.l3_e(n1, n2)) /
      mesh_ptrs.A2_b(n1, n2);

  // (Curl u)_3 = d1u2 - d2u1
  b3[globalOffset] +=
      -dt * (((e2(n1, n2) * alpha_gr(r1) +
               // (b1(n1, n2) + dev_bg_fields.B1(n1, n2)) * beta_phi(r1,
               // dev_mesh.pos(1, n2, 0))) *
               dev_bg_fields.B1(n1, n2) *
                   beta_phi(r1, dev_mesh.pos(1, n2, 0))) *
                  mesh_ptrs.l2_e(n1, n2) -
              (e2(n1 - 1, n2) * alpha_gr(r0) +
               // (b1(n1 - 1, n2) + dev_bg_fields.B1(n1 - 1, n2)) *
               // beta_phi(r0, dev_mesh.pos(1, n2, 0))) *
               dev_bg_fields.B1(n1 - 1, n2) *
                   beta_phi(r0, dev_mesh.pos(1, n2, 0))) *
                  mesh_ptrs.l2_e(n1 - 1, n2) +
              (e1(n1, n2 - 1) * alpha_gr(r) -
               // (b2(n1, n2 - 1) + dev_bg_fields.B2(n1, n2 - 1)) *
               // beta_phi(r, dev_mesh.pos(1, n2 - 1, 1))) *
               dev_bg_fields.B2(n1, n2 - 1) *
                   beta_phi(r, dev_mesh.pos(1, n2 - 1, 1))) *
                  mesh_ptrs.l1_e(n1, n2 - 1) -
              (e1(n1, n2) * alpha_gr(r) -
               // (b2(n1, n2) + dev_bg_fields.B2(n1, n2)) * beta_phi(r,
               // dev_mesh.pos(1, n2, 1))) *
               dev_bg_fields.B2(n1, n2) *
                   beta_phi(r, dev_mesh.pos(1, n2, 1))) *
                  mesh_ptrs.l1_e(n1, n2)) /
             mesh_ptrs.A3_b(n1, n2));

  __syncthreads();

  // Extra work for the axis at theta = 0
  if (std::abs(dev_mesh.pos(1, n2, true) - dev_mesh.delta[1]) <
      1.0e-5) {
    n2 = dev_mesh.guard[1] - 1;
    globalOffset = b2.compute_offset(n1, n2);

    b2[globalOffset] = 0.0f;
  }
}

// template <int DIM1, int DIM2>
__global__ void
compute_divs(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
             pitchptr<Scalar> e3, pitchptr<Scalar> b1,
             pitchptr<Scalar> b2, pitchptr<Scalar> b3,
             pitchptr<Scalar> divE, pitchptr<Scalar> divB,
             mesh_ptrs_log_sph mesh_ptrs) {
  int t1 = blockIdx.x, t2 = blockIdx.y;
  int c1 = threadIdx.x, c2 = threadIdx.y;
  int n1 = dev_mesh.guard[0] + t1 * blockDim.x + c1;
  int n2 = dev_mesh.guard[1] + t2 * blockDim.y + c2;
  // size_t globalOffset = n2 * divE.pitch + n1 * sizeof(Scalar);
  size_t globalOffset = divE.compute_offset(n1, n2);

  // if (n1 > dev_mesh.guard[0] + 1) {
  if (dev_mesh.pos(0, n1, 1) > dev_mesh.delta[0]) {
    divE[globalOffset] =
        (e1(n1 + 1, n2) * mesh_ptrs.A1_e(n1 + 1, n2) -
         e1(n1, n2) * mesh_ptrs.A1_e(n1, n2) +
         e2(n1, n2 + 1) * mesh_ptrs.A2_e(n1, n2 + 1) -
         e2(n1, n2) * mesh_ptrs.A2_e(n1, n2)) /
        (mesh_ptrs.dV(n1, n2) * dev_mesh.delta[0] * dev_mesh.delta[1]);

    // if (n2 == dev_mesh.dims[1] - dev_mesh.guard[1] - 1) {
    if (std::abs(dev_mesh.pos(1, n2, 1) - dev_mesh.sizes[1] +
                 dev_mesh.lower[1]) < 1.0e-5) {
      divE[globalOffset] =
          (e1(n1 + 1, n2) * mesh_ptrs.A1_e(n1 + 1, n2) -
           e1(n1, n2) * mesh_ptrs.A1_e(n1, n2) -
           // e2(n1, n2 + 1) *
           //     mesh_ptrs.A2_e(n1, n2 + 1) -
           2.0 * e2(n1, n2) * mesh_ptrs.A2_e(n1, n2)) /
          (mesh_ptrs.dV(n1, n2) * dev_mesh.delta[0] *
           dev_mesh.delta[1]);
    }
  }
  divB[globalOffset] =
      (b1(n1, n2) * mesh_ptrs.A1_b(n1, n2) -
       b1(n1 - 1, n2) * mesh_ptrs.A1_b(n1 - 1, n2) +
       b2(n1, n2) * mesh_ptrs.A2_b(n1, n2) -
       b2(n1, n2 - 1) * mesh_ptrs.A2_b(n1, n2 - 1)) /
      (mesh_ptrs.dV(n1, n2) * dev_mesh.delta[0] * dev_mesh.delta[1]);

  __syncthreads();

  if (std::abs(dev_mesh.pos(1, n2, 1)) - dev_mesh.delta[1] < 1.0e-5) {
    n2 = dev_mesh.guard[1] - 1;
    globalOffset = e1.compute_offset(n1, n2);

    divE[globalOffset] =
        (e1(n1 + 1, n2) * mesh_ptrs.A1_e(n1 + 1, n2) -
         e1(n1, n2) * mesh_ptrs.A1_e(n1, n2) +
         2.0f * e2(n1, n2 + 1) * mesh_ptrs.A2_e(n1, n2 + 1)) /
        (mesh_ptrs.dV(n1, n2) * dev_mesh.delta[0] * dev_mesh.delta[1]);
  }
}

__global__ void
stellar_boundary(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
                 pitchptr<Scalar> e3, pitchptr<Scalar> b1,
                 pitchptr<Scalar> b2, pitchptr<Scalar> b3,
                 Scalar omega) {
  for (int j = blockIdx.x * blockDim.x + threadIdx.x;
       j < dev_mesh.dims[1]; j += blockDim.x * gridDim.x) {
    Scalar theta_s = dev_mesh.pos(1, j, true);
    Scalar theta = dev_mesh.pos(1, j, false);
    // for (int i = 0; i < dev_mesh.guard[0] + 1; i++) {
    for (int i = 0; i < dev_mesh.guard[0]; i++) {
      Scalar r_s = std::exp(dev_mesh.pos(0, i, true));
      Scalar r = std::exp(dev_mesh.pos(0, i, false));
      Scalar omega_LT = 0.4f * omega * dev_params.compactness;
      b1(i, j) = 0.0f;
      e3(i, j) = 0.0f;
      e2(i, j) = -(omega - omega_LT) * std::sin(theta) *
                 dev_bg_fields.B1(i, j) / alpha_gr(r_s) / r_s / r_s;
      e1(i, j) = (omega - omega_LT) * std::sin(theta_s) *
                 dev_bg_fields.B2(i, j) / alpha_gr(r_s) / r / r;
      b2(i, j) = 0.0f;
      b3(i, j) = 0.0f;
    }
  }
}

__global__ void
axis_boundary_lower(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
                    pitchptr<Scalar> e3, pitchptr<Scalar> b1,
                    pitchptr<Scalar> b2, pitchptr<Scalar> b3) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
    e3(i, dev_mesh.guard[1] - 1) = 0.0f;
    // e3(i, dev_mesh.guard[1]) = 0.0f;
    e2(i, dev_mesh.guard[1] - 1) = -e2(i, dev_mesh.guard[1]);
    // e2(i, dev_mesh.guard[1] - 1) = e2(i, dev_mesh.guard[1]) = 0.0f;

    b3(i, dev_mesh.guard[1] - 1) = b3(i, dev_mesh.guard[1]) = 0.0f;
    b2(i, dev_mesh.guard[1] - 1) = 0.0f;
    b1(i, dev_mesh.guard[1] - 1) = b1(i, dev_mesh.guard[1]);
  }
}

__global__ void
axis_boundary_upper(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
                    pitchptr<Scalar> e3, pitchptr<Scalar> b1,
                    pitchptr<Scalar> b2, pitchptr<Scalar> b3) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x;
       i < dev_mesh.dims[0]; i += blockDim.x * gridDim.x) {
    int j_last = dev_mesh.dims[1] - dev_mesh.guard[1];
    e3(i, j_last - 1) = 0.0f;
    e2(i, j_last) = -e2(i, j_last - 1);
    // e2(i, j_last) = e2(i, j_last - 1) = 0.0f;

    b3(i, j_last) = b3(i, j_last - 1) = 0.0f;
    b2(i, j_last - 1) = 0.0f;
    b1(i, j_last) = b1(i, j_last - 1);
  }
}

__global__ void
outflow_boundary(pitchptr<Scalar> e1, pitchptr<Scalar> e2,
                 pitchptr<Scalar> e3, pitchptr<Scalar> b1,
                 pitchptr<Scalar> b2, pitchptr<Scalar> b3) {
  for (int j = blockIdx.x * blockDim.x + threadIdx.x;
       j < dev_mesh.dims[1]; j += blockDim.x * gridDim.x) {
    for (int i = 0; i < dev_params.damping_length; i++) {
      int n1 = dev_mesh.dims[0] - dev_params.damping_length + i;
      // size_t offset = j * e1.pitch + n1 * sizeof(Scalar);
      size_t offset = e1.compute_offset(n1, j);
      Scalar lambda =
          1.0f - dev_params.damping_coef *
                     square((Scalar)i / dev_params.damping_length);
      e1[offset] *= lambda;
      e2[offset] *= lambda;
      e3[offset] *= lambda;
      // b1[offset] *= lambda;
      // b2[offset] *= lambda;
      b3[offset] *= lambda;
    }
  }
}

}  // namespace Kernels

field_solver_logsph::field_solver_logsph(sim_environment &env)
    : m_env(env) {}

field_solver_logsph::~field_solver_logsph() {}

void
field_solver_logsph::update_fields(sim_data &data, double dt,
                                   double time) {
  // Only implemented 2D!
  if (data.env.grid().dim() != 2) return;
  timer::stamp("field_update");

  // First communicate to get the E field guard cells
  // data.env.get_sub_guard_cells(data.E);

  Grid_LogSph &grid = *dynamic_cast<Grid_LogSph *>(&m_env.local_grid());
  auto mesh_ptrs = get_mesh_ptrs(grid);
  auto &mesh = grid.mesh();

  dim3 blockSize(32, 16);
  dim3 gridSize(mesh.reduced_dim(0) / 32, mesh.reduced_dim(1) / 16);
  // Update B
  Kernels::compute_b_update<<<gridSize, blockSize>>>(
      get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
      get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
      get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)),
      mesh_ptrs, dt);
  CudaCheckError();

  // Communicate the new B values to guard cells
  // data.env.get_sub_guard_cells(data.B);
  // data.env.get_sub_guard_cells(data.J);

  // Update E
  Kernels::compute_e_update<<<gridSize, blockSize>>>(
      get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
      get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
      get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)),
      get_pitchptr(data.J.data(0)), get_pitchptr(data.J.data(1)),
      get_pitchptr(data.J.data(2)), mesh_ptrs, dt);
  CudaCheckError();

  // Communicate the new E values to guard cells
  // data.env.get_sub_guard_cells(data.E);

  // Update B
  Kernels::compute_divs<<<gridSize, blockSize>>>(
      get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
      get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
      get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)),
      get_pitchptr(data.divE.data()), get_pitchptr(data.divB.data()),
      mesh_ptrs);
  CudaCheckError();
  data.compute_edotb();

  CudaSafeCall(hipDeviceSynchronize());
  timer::show_duration_since_stamp("Field update", "us",
                                   "field_update");
}

// void
// field_solver_logsph::set_background_j(const vfield_t &J) {}

void
field_solver_logsph::apply_boundary(sim_data &data, double omega,
                                    double time) {
  // int dev_id = data.dev_id;
  // CudaSafeCall(hipSetDevice(dev_id));
  if (data.env.is_boundary(BoundaryPos::lower0)) {
    Kernels::stellar_boundary<<<32, 256>>>(
        get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
        get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
        get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)),
        omega);
    CudaCheckError();
  }

  if (data.env.is_boundary(BoundaryPos::upper0)) {
    Kernels::outflow_boundary<<<32, 256>>>(
        get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
        get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
        get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)));
    CudaCheckError();
  }

  if (data.env.is_boundary(BoundaryPos::lower1)) {
    Kernels::axis_boundary_lower<<<32, 256>>>(
        get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
        get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
        get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)));
    CudaCheckError();
  }

  if (data.env.is_boundary(BoundaryPos::upper1)) {
    Kernels::axis_boundary_upper<<<32, 256>>>(
        get_pitchptr(data.E.data(0)), get_pitchptr(data.E.data(1)),
        get_pitchptr(data.E.data(2)), get_pitchptr(data.B.data(0)),
        get_pitchptr(data.B.data(1)), get_pitchptr(data.B.data(2)));
    CudaCheckError();
  }
  // Logger::print_info("omega is {}", omega);
}

}  // namespace Aperture
