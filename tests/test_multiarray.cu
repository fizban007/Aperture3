#include "hip/hip_runtime.h"
#include "data/multi_array.h"
#include "data/detail/multi_array_utils.hpp"
#include <hip/hip_runtime.h>
#include <iostream>
#include "catch.hpp"
#include "cuda/cudaUtility.h"

using namespace Aperture;

__global__
void add(const float* a, const float* b, float* c);

__global__
void add2D(const Extent ext, const float* a, const float* b, float* c) {

  for (int j = blockIdx.y * blockDim.y + threadIdx.y;
       j < ext.y;
       j += blockDim.y * gridDim.y) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
        i < ext.x;
        i += blockDim.x * gridDim.x) {
      size_t idx = i + j * ext.x;
      c[idx] = a[idx] + b[idx];
    }
  }
  
}

struct Data {
  MultiArray<float> a, b, c;
  size_t size, memSize;

  Data(int x, int y = 1, int z = 1) :
      a(x, y, z), b(x, y, z), c(x, y, z) {
    size = x * y * z;
    memSize = size * sizeof(float);
  }

  void prefetch(int deviceId) {
    // hipMemPrefetchAsync(a.data(), memSize, deviceId);
    // hipMemPrefetchAsync(b.data(), memSize, deviceId);
    // hipMemPrefetchAsync(c.data(), memSize, deviceId);
    a.sync_to_device(deviceId);
    b.sync_to_device(deviceId);
    c.sync_to_device(deviceId);
  }

  void prefetch() {
    // hipMemPrefetchAsync(c.data(), memSize, hipCpuDeviceId);
    c.sync_to_host();
  }
};

TEST_CASE("Initialize multi_array", "[MultiArray]") {
  hipFree(0);
  Data data(256, 256);

  data.a.assign(1.0);
  data.b.assign(2.0);

  add<<<256, 256>>>(data.a.data(), data.b.data(), data.c.data());

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  size_t N = data.a.size();
  for (size_t i = 0; i < N; i++) {
    CHECK(data.c[i] == 3.0f);
  }

}

TEST_CASE("Add 2D multi_array", "[MultiArray]") {
  Data data(1500, 1500);

  data.a.assign(1.0);
  data.b.assign(2.0);

  hipDeviceProp_t p;
  int deviceId;
  hipGetDevice(&deviceId);
  hipGetDeviceProperties(&p, deviceId);
  data.prefetch(deviceId);

  dim3 blockSize(32, 32);
  dim3 gridSize(32, 32);
  add2D<<<gridSize, blockSize>>>(data.a.extent(), data.a.data(), data.b.data(), data.c.data());

  data.prefetch();
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  size_t N = data.a.size();
  for (size_t i = 0; i < N; i++) {
    CHECK(data.c[i] == 3.0f);
  }
}

TEST_CASE("Map Array Multiply", "[MultiArray]")  {
  using namespace Aperture::detail;
  Data data(150, 150, 100);

  data.a.assign(2.0);
  data.b.assign(1.5);
  std::cout << data.a.extent() << std::endl;

  int deviceId;
  hipGetDevice(&deviceId);
  data.prefetch(deviceId);

  // dim3 blockSize(32, 32);
  // dim3 gridSize(32, 32);
  dim3 blockSize(8, 8, 8);
  dim3 gridSize(16, 16, 8);
  knl_map_array_binary_op<<<gridSize, blockSize>>>(data.a.data(), data.b.data(), data.c.data(), data.a.extent(), Op_Multiply<float>());
  CudaCheckError();

  data.prefetch();
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  size_t N = data.c.size();
  for (size_t i = 0; i < N; i++) {
    INFO("i, j, k are " << i % 150 << ", " << (i / 150) % 150 << ", " << i / (150*150));
    REQUIRE(data.c[i] == 3.0f);
  }
}
