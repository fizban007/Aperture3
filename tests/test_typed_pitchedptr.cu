#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "cuda/cudaUtility.h"
#include "cuda/utils/typed_pitchedptr.cuh"
#include <vector>

using namespace Aperture;

template <typename T>
__global__ void assign_values(typed_pitchedptr<T> ptr) {
  ptr(threadIdx.x, blockIdx.x) = threadIdx.x * blockIdx.x;
}

TEST_CASE("Simple usage", "[pitchedptr]") {
  hipPitchedPtr p2f, p3f;
  hipPitchedPtr p2d, p3d;

  CudaSafeCall(hipMalloc3D(&p2f, make_hipExtent(100 * sizeof(float), 100, 1)));
  CudaSafeCall(hipMalloc3D(&p2d, make_hipExtent(100 * sizeof(double), 100, 1)));
  CudaSafeCall(hipMalloc3D(&p3f, make_hipExtent(100 * sizeof(float), 100, 100)));
  CudaSafeCall(hipMalloc3D(&p3d, make_hipExtent(100 * sizeof(double), 100, 100)));

  std::vector<float> v2f(100*100), v3f(100*100*100);
  std::vector<double> v2d(100*100), v3d(100*100*100);

  assign_values<<<100, 100>>>(typed_pitchedptr<float>(p2f));
  assign_values<<<100, 100>>>(typed_pitchedptr<double>(p2d));
  CudaSafeCall(hipDeviceSynchronize());

  hipMemcpy3DParms pv2f = {0};
  hipMemcpy3DParms pv2d = {0};
  pv2f.srcPtr = p2f;
  pv2f.dstPtr = make_hipPitchedPtr(v2f.data(), 100*sizeof(float), 100, 100);
  pv2f.srcPos = make_hipPos(0, 0, 0);
  pv2f.dstPos = make_hipPos(0, 0, 0);
  pv2f.extent = make_hipExtent(100*sizeof(float), 100, 1);
  pv2f.kind = hipMemcpyDeviceToHost;
  pv2d.srcPtr = p2d;
  pv2d.dstPtr = make_hipPitchedPtr(v2d.data(), 100*sizeof(double), 100, 100);
  pv2d.srcPos = make_hipPos(0, 0, 0);
  pv2d.dstPos = make_hipPos(0, 0, 0);
  pv2d.extent = make_hipExtent(100*sizeof(double), 100, 1);
  pv2d.kind = hipMemcpyDeviceToHost;
  CudaSafeCall(hipMemcpy3D(&pv2f));
  CudaSafeCall(hipMemcpy3D(&pv2d));

  for(int i = 0; i < 100; i++) {
    for (int j = 0; j < 100; j++) {
      CHECK(v2f[i + j * 100] == Approx(i * j));
      CHECK(v2d[i + j * 100] == Approx(i * j));
    }
  }

  CudaSafeCall(hipFree(p2f.ptr));
  CudaSafeCall(hipFree(p2d.ptr));
  CudaSafeCall(hipFree(p3f.ptr));
  CudaSafeCall(hipFree(p3d.ptr));
}
