
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__
void set_value(float* array, int size, float value) {
  for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < size;
       i += blockDim.x * gridDim.x) {
    array[i] = value;
  }
}

__global__
void get_value(float* array, int pos) {
  printf("value is %f\n", array[pos]);
}

int main(int argc, char *argv[]) {
  // Poll the system to detect how many GPUs are on the node
  int n_devices;
  hipGetDeviceCount(&n_devices);
  if (n_devices <= 0) {
    exit(1);
  }

  std::vector<int> dev_map(n_devices);
  // Logger::print_info("Found {} Cuda devices:", n_devices);
  for (int i = 0; i < n_devices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    // Logger::print_info("    Device Number: {}", i);
    // Logger::print_info("    Device Name: {}", prop.name);
    // Logger::print_info("    Device Total Memory: {}MiB",
    //                    prop.totalGlobalMem / (1024 * 1024));
    dev_map[i] = i;
  }

  float *v1, *v2;
  int N = 10000;
  hipSetDevice(dev_map[0]);
  hipMallocManaged(&v1, N*sizeof(float));
  set_value<<<64, 128>>>(v1, N, 1.0);
  hipSetDevice(dev_map[1]);
  hipMallocManaged(&v2, N*sizeof(float));
  set_value<<<64, 128>>>(v2, N, 2.0);
  hipDeviceSynchronize();
  get_value<<<1,1>>>(v1, 4000);
  hipDeviceSynchronize();

  return 0;
}