#include "hip/hip_runtime.h"
#include "catch.hpp"
#include "cuda/constant_mem.h"
#include "cu_sim_environment.h"
#include <cstring>

using namespace Aperture;
using namespace std;

// Defined in test_config.cpp
// void check_test_params(const SimParams& params);

__global__ void
print_dev_mesh(int a) {
  printf("Test!!! %d\n", a);
  printf("mesh dims are %d %d %d\n", dev_mesh.dims[0], dev_mesh.dims[1],
         dev_mesh.dims[2]);
}

TEST_CASE("Loading environment", "[Env]") {
  int my_argc = 7;
  char** my_argv = new char*[my_argc + 1];
  for (int i = 0; i < my_argc; i++) my_argv[i] = new char[100];
  strcpy(my_argv[0], "aperture");
  strcpy(my_argv[1], "-c");
  strcpy(my_argv[2], "test.toml");
  strcpy(my_argv[3], "-s");
  strcpy(my_argv[4], "100000");
  strcpy(my_argv[5], "-d");
  strcpy(my_argv[6], "20");
  my_argv[7] = nullptr;
  // Aperture::cu_sim_environment &env =
  // Aperture::cu_sim_environment::get_instance().initialize(&my_argc,
  // &my_argv); env =
  // std::make_unique<Aperture::cu_sim_environment>(&my_argc, &my_argv);
  cu_sim_environment env(&my_argc, &my_argv);

  print_dev_mesh<<<1, 1>>>(10);

  Quadmesh mesh;
  env.check_dev_mesh(mesh);
  CHECK(mesh.dims[0] == 10006);
  CHECK(mesh.dims[1] == 5);
  CHECK(mesh.dims[2] == 11);

  // SimParams p;
  // p.data_dir = env.params().data_dir;

  // env.check_dev_params(p);
  // check_test_params(p);

  for (int i = 0; i < my_argc; i++) delete[] my_argv[i];
  delete[] my_argv;
}
